
#include <hip/hip_runtime.h>
extern "C" {
typedef struct {
    int e0;
    char* e1;
} struct_Buffer_5426;
typedef struct {
    struct_Buffer_5426 e0;
    struct_Buffer_5426 e1;
    int e2;
    int e3;
} struct_image_5425;
typedef struct {
    struct_Buffer_5426 e0;
    int e1;
    int e2;
} struct_filter_5429;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_20630(struct_image_5425, struct_filter_5429, struct_Buffer_5426);
__global__ void lambda_20762(struct_filter_5429, struct_image_5425, struct_Buffer_5426, double*, struct_Buffer_5426);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_20630(struct_image_5425 _20633_22969, struct_filter_5429 _20634_22970, struct_Buffer_5426 _20635_22971) {
    int  _22977;
    int p_22977;
    int  _22983;
    int p_22983;
    int  _22989;
    int p_22989;
    int  _22995;
    int p_22995;
    int  _23001;
    int p_23001;
    int  _23007;
    int p_23007;
    int  _23030;
    int p_23030;
    double  sum_23032;
    double psum_23032;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _22977 = blockIdx_x();
    p_22977 = _22977;
    l22975: ;
        _22977 = p_22977;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22983 = blockDim_x();
        p_22983 = _22983;
    l22981: ;
        _22983 = p_22983;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22989 = threadIdx_x();
        p_22989 = _22989;
    l22987: ;
        _22989 = p_22989;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22995 = blockIdx_y();
        p_22995 = _22995;
    l22993: ;
        _22995 = p_22995;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23001 = blockDim_y();
        p_23001 = _23001;
    l22999: ;
        _23001 = p_23001;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23007 = threadIdx_y();
        p_23007 = _23007;
    l23005: ;
        _23007 = p_23007;
        #line 155 "gpu_device.impala"
        int _23008;
        _23008 = _22977 * _22983;
        #line 11 "main.impala"
        int _23011;
        _23011 = _20633_22969.e2;
        #line 155 "gpu_device.impala"
        int gid_x_23009;
        gid_x_23009 = _23008 + _22989;
        #line 160 "gpu_device.impala"
        bool _23012;
        _23012 = gid_x_23009 < _23011;
        #line 160 "gpu_device.impala"
        if (_23012) goto l23013; else goto l23078;
    l23078: ;
        #line 163 "gpu_device.impala"
        goto l23077;
    l23013: ;
        #line 11 "main.impala"
        int _23017;
        _23017 = _20633_22969.e3;
        #line 157 "gpu_device.impala"
        int _23014;
        _23014 = _22995 * _23001;
        #line 157 "gpu_device.impala"
        int gid_y_23015;
        gid_y_23015 = _23014 + _23007;
        #line 160 "gpu_device.impala"
        bool _23018;
        _23018 = gid_y_23015 < _23017;
        #line 160 "gpu_device.impala"
        if (_23018) goto l23019; else goto l23076;
    l23076: ;
        #line 163 "gpu_device.impala"
        goto l23077;
    l23077: ;
        return ;
    l23019: ;
        #line 45 "gpu_device.impala"
        char* _23060;
        _23060 = _20635_22971.e1;
        #line 50 "gpu_device.impala"
        int _23049;
        _23049 = gid_y_23015 * _23011;
        #line 50 "gpu_device.impala"
        int _23062;
        _23062 = _23049 + gid_x_23009;
        #line 4 "gaussian.impala"
        int _23021;
        _23021 = _20634_22970.e1;
        #line 50 "gpu_device.impala"
        struct_Buffer_5426 _23046;
        _23046 = _20633_22969.e1;
        #line 45 "gpu_device.impala"
        double* _23061;
        union { double* dst; char* src; } u_23061;
        u_23061.src = _23060;
        _23061 = u_23061.dst;
        #line 45 "gpu_device.impala"
        double* _23063;
        _23063 = _23061 + _23062;
        #line 4 "gaussian.impala"
        int h_anchor_23023;
        h_anchor_23023 = _23021 / 2;
        #line 50 "gpu_device.impala"
        char* _23047;
        _23047 = _23046.e1;
        #line 17 "gaussian.impala"
        bool _23024;
        _23024 = h_anchor_23023 <= gid_x_23009;
        #line 50 "gpu_device.impala"
        double* _23048;
        union { double* dst; char* src; } u_23048;
        u_23048.src = _23047;
        _23048 = u_23048.dst;
        #line 17 "gaussian.impala"
        if (_23024) goto l23025; else goto l23075;
    l23075: ;
        #line 27 "gaussian.impala"
        goto l23069;
    l23025: ;
        #line 17 "gaussian.impala"
        int _23026;
        _23026 = _23011 - h_anchor_23023;
        #line 17 "gaussian.impala"
        bool _23027;
        _23027 = gid_x_23009 < _23026;
        #line 17 "gaussian.impala"
        if (_23027) goto l23028; else goto l23068;
    l23068: ;
        #line 27 "gaussian.impala"
        goto l23069;
    l23069: ;
        #line 50 "gpu_device.impala"
        double* _23070;
        _23070 = _23048 + _23062;
        #line 50 "gpu_device.impala"
        double _23071;
        _23071 = *_23070;
        #line 50 "gpu_device.impala"
        double _23073;
        _23073 = _23071;
        #line 45 "gpu_device.impala"
        *_23063 = _23073;
        return ;
    l23028: ;
        #line 55 "gpu_device.impala"
        struct_Buffer_5426 _23039;
        _23039 = _20634_22970.e0;
        #line 19 "gaussian.impala"
        int _23066;
        _23066 = 0 - h_anchor_23023;
        #line 19 "gaussian.impala"
        int _23034;
        _23034 = 1 + h_anchor_23023;
        #line 55 "gpu_device.impala"
        char* _23040;
        _23040 = _23039.e1;
        #line 55 "gpu_device.impala"
        double* _23041;
        union { double* dst; char* src; } u_23041;
        u_23041.src = _23040;
        _23041 = u_23041.dst;
        #line 19 "gpu_device.impala"
        p_23030 = _23066;
        psum_23032 = 0.000000e+00;
        goto l23029;
    l23029: ;
        _23030 = p_23030;
        sum_23032 = psum_23032;
        #line 19 "gpu_device.impala"
        bool _23035;
        _23035 = _23030 < _23034;
        #line 19 "gpu_device.impala"
        if (_23035) goto l23036; else goto l23059;
    l23059: ;
        #line 45 "gpu_device.impala"
        *_23063 = sum_23032;
        return ;
    l23036: ;
        #line 23 "gpu_device.impala"
        int _23037;
        _23037 = 1 + _23030;
        #line 21 "gaussian.impala"
        int _23042;
        _23042 = _23030 + h_anchor_23023;
        #line 21 "gaussian.impala"
        int _23050;
        _23050 = gid_x_23009 + _23030;
        #line 54 "gpu_device.impala"
        double* i_23043;
        i_23043 = _23041 + _23042;
        #line 50 "gpu_device.impala"
        int _23051;
        _23051 = _23049 + _23050;
        #line 55 "gpu_device.impala"
        double _23044;
        _23044 = *i_23043;
        #line 50 "gpu_device.impala"
        double* _23052;
        _23052 = _23048 + _23051;
        #line 55 "gpu_device.impala"
        double _23055;
        _23055 = _23044;
        #line 50 "gpu_device.impala"
        double _23053;
        _23053 = *_23052;
        #line 50 "gpu_device.impala"
        double _23056;
        _23056 = _23053;
        #line 21 "gaussian.impala"
        double _23057;
        _23057 = _23055 * _23056;
        #line 21 "gaussian.impala"
        double _23058;
        _23058 = sum_23032 + _23057;
        #line 19 "gpu_device.impala"
        p_23030 = _23037;
        psum_23032 = _23058;
        goto l23029;
}

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_20762(struct_filter_5429 _20765_23082, struct_image_5425 _20766_23083, struct_Buffer_5426 _20767_23084, double* _20768_23085, struct_Buffer_5426 _20769_23086) {
    int  _23089;
    int p_23089;
    int  _23092;
    int p_23092;
    int  _23095;
    int p_23095;
    int  _23098;
    int p_23098;
    int  _23101;
    int p_23101;
    int  _23104;
    int p_23104;
    int  _23123;
    int p_23123;
    double  sum_23125;
    double psum_23125;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _23089 = blockIdx_x();
    p_23089 = _23089;
    l23087: ;
        _23089 = p_23089;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23092 = blockDim_x();
        p_23092 = _23092;
    l23090: ;
        _23092 = p_23092;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23095 = threadIdx_x();
        p_23095 = _23095;
    l23093: ;
        _23095 = p_23095;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23098 = blockIdx_y();
        p_23098 = _23098;
    l23096: ;
        _23098 = p_23098;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23101 = blockDim_y();
        p_23101 = _23101;
    l23099: ;
        _23101 = p_23101;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23104 = threadIdx_y();
        p_23104 = _23104;
    l23102: ;
        _23104 = p_23104;
        #line 155 "gpu_device.impala"
        int _23105;
        _23105 = _23089 * _23092;
        #line 11 "main.impala"
        int _23107;
        _23107 = _20766_23083.e2;
        #line 155 "gpu_device.impala"
        int gid_x_23106;
        gid_x_23106 = _23105 + _23095;
        #line 160 "gpu_device.impala"
        bool _23108;
        _23108 = gid_x_23106 < _23107;
        #line 160 "gpu_device.impala"
        if (_23108) goto l23109; else goto l23167;
    l23167: ;
        #line 163 "gpu_device.impala"
        goto l23166;
    l23109: ;
        #line 157 "gpu_device.impala"
        int _23110;
        _23110 = _23098 * _23101;
        #line 157 "gpu_device.impala"
        int gid_y_23111;
        gid_y_23111 = _23110 + _23104;
        #line 11 "main.impala"
        int _23112;
        _23112 = _20766_23083.e3;
        #line 160 "gpu_device.impala"
        bool _23113;
        _23113 = gid_y_23111 < _23112;
        #line 160 "gpu_device.impala"
        if (_23113) goto l23114; else goto l23165;
    l23165: ;
        #line 163 "gpu_device.impala"
        goto l23166;
    l23166: ;
        return ;
    l23114: ;
        #line 6 "gaussian.impala"
        int _23115;
        _23115 = _20765_23082.e2;
        #line 50 "gpu_device.impala"
        char* _23137;
        _23137 = _20769_23086.e1;
        #line 45 "gpu_device.impala"
        char* _23150;
        _23150 = _20767_23084.e1;
        #line 50 "gpu_device.impala"
        int _23152;
        _23152 = gid_y_23111 * _23107;
        #line 50 "gpu_device.impala"
        double* _23138;
        union { double* dst; char* src; } u_23138;
        u_23138.src = _23137;
        _23138 = u_23138.dst;
        #line 6 "gaussian.impala"
        int v_anchor_23116;
        v_anchor_23116 = _23115 / 2;
        #line 50 "gpu_device.impala"
        int _23153;
        _23153 = _23152 + gid_x_23106;
        #line 45 "gpu_device.impala"
        double* _23151;
        union { double* dst; char* src; } u_23151;
        u_23151.src = _23150;
        _23151 = u_23151.dst;
        #line 39 "gaussian.impala"
        bool _23117;
        _23117 = v_anchor_23116 <= gid_y_23111;
        #line 45 "gpu_device.impala"
        double* _23154;
        _23154 = _23151 + _23153;
        #line 39 "gaussian.impala"
        if (_23117) goto l23118; else goto l23164;
    l23164: ;
        #line 49 "gaussian.impala"
        goto l23158;
    l23118: ;
        #line 39 "gaussian.impala"
        int _23119;
        _23119 = _23112 - v_anchor_23116;
        #line 39 "gaussian.impala"
        bool _23120;
        _23120 = gid_y_23111 < _23119;
        #line 39 "gaussian.impala"
        if (_23120) goto l23121; else goto l23157;
    l23157: ;
        #line 49 "gaussian.impala"
        goto l23158;
    l23158: ;
        #line 50 "gpu_device.impala"
        double* _23159;
        _23159 = _23138 + _23153;
        #line 50 "gpu_device.impala"
        double _23160;
        _23160 = *_23159;
        #line 50 "gpu_device.impala"
        double _23162;
        _23162 = _23160;
        #line 45 "gpu_device.impala"
        *_23154 = _23162;
        return ;
    l23121: ;
        #line 41 "gaussian.impala"
        int _23156;
        _23156 = 0 - v_anchor_23116;
        #line 55 "gpu_device.impala"
        struct_Buffer_5426 _23130;
        _23130 = _20765_23082.e0;
        #line 41 "gaussian.impala"
        int _23126;
        _23126 = 1 + v_anchor_23116;
        #line 55 "gpu_device.impala"
        char* _23131;
        _23131 = _23130.e1;
        #line 55 "gpu_device.impala"
        double* _23132;
        union { double* dst; char* src; } u_23132;
        u_23132.src = _23131;
        _23132 = u_23132.dst;
        #line 19 "gpu_device.impala"
        p_23123 = _23156;
        psum_23125 = 0.000000e+00;
        goto l23122;
    l23122: ;
        _23123 = p_23123;
        sum_23125 = psum_23125;
        #line 19 "gpu_device.impala"
        bool _23127;
        _23127 = _23123 < _23126;
        #line 19 "gpu_device.impala"
        if (_23127) goto l23128; else goto l23149;
    l23149: ;
        #line 45 "gpu_device.impala"
        *_23154 = sum_23125;
        return ;
    l23128: ;
        #line 43 "gaussian.impala"
        int _23133;
        _23133 = _23123 + v_anchor_23116;
        #line 43 "gaussian.impala"
        int _23139;
        _23139 = gid_y_23111 + _23123;
        #line 54 "gpu_device.impala"
        double* i_23134;
        i_23134 = _23132 + _23133;
        #line 50 "gpu_device.impala"
        int _23140;
        _23140 = _23139 * _23107;
        #line 23 "gpu_device.impala"
        int _23129;
        _23129 = 1 + _23123;
        #line 55 "gpu_device.impala"
        double _23135;
        _23135 = *i_23134;
        #line 50 "gpu_device.impala"
        int _23141;
        _23141 = _23140 + gid_x_23106;
        #line 55 "gpu_device.impala"
        double _23145;
        _23145 = _23135;
        #line 50 "gpu_device.impala"
        double* _23142;
        _23142 = _23138 + _23141;
        #line 50 "gpu_device.impala"
        double _23143;
        _23143 = *_23142;
        #line 50 "gpu_device.impala"
        double _23146;
        _23146 = _23143;
        #line 43 "gaussian.impala"
        double _23147;
        _23147 = _23145 * _23146;
        #line 43 "gaussian.impala"
        double _23148;
        _23148 = sum_23125 + _23147;
        #line 19 "gpu_device.impala"
        p_23123 = _23129;
        psum_23125 = _23148;
        goto l23122;
}

}