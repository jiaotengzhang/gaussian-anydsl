
#include <hip/hip_runtime.h>
extern "C" {
typedef struct {
    int e0;
    char* e1;
} struct_Buffer_5540;
typedef struct {
    struct_Buffer_5540 e0;
    int e1;
    int e2;
} struct_filter_5543;
typedef struct {
    struct_Buffer_5540 e0;
    struct_Buffer_5540 e1;
    int e2;
    int e3;
} struct_image_5544;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_21082(struct_Buffer_5540, struct_filter_5543, struct_image_5544);
__global__ void lambda_21214(struct_Buffer_5540, struct_filter_5543, double*, struct_image_5544, struct_Buffer_5540);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_21082(struct_Buffer_5540 _21085_23421, struct_filter_5543 _21086_23422, struct_image_5544 _21087_23423) {
    int  _23429;
    int p_23429;
    int  _23435;
    int p_23435;
    int  _23441;
    int p_23441;
    int  _23447;
    int p_23447;
    int  _23453;
    int p_23453;
    int  _23459;
    int p_23459;
    int  _23482;
    int p_23482;
    double  sum_23484;
    double psum_23484;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _23429 = blockIdx_x();
    p_23429 = _23429;
    l23427: ;
        _23429 = p_23429;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23435 = blockDim_x();
        p_23435 = _23435;
    l23433: ;
        _23435 = p_23435;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23441 = threadIdx_x();
        p_23441 = _23441;
    l23439: ;
        _23441 = p_23441;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23447 = blockIdx_y();
        p_23447 = _23447;
    l23445: ;
        _23447 = p_23447;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23453 = blockDim_y();
        p_23453 = _23453;
    l23451: ;
        _23453 = p_23453;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23459 = threadIdx_y();
        p_23459 = _23459;
    l23457: ;
        _23459 = p_23459;
        #line 11 "main.impala"
        int _23463;
        _23463 = _21087_23423.e2;
        #line 144 "gpu_device.impala"
        int _23460;
        _23460 = _23429 * _23435;
        #line 144 "gpu_device.impala"
        int gid_x_23461;
        gid_x_23461 = _23460 + _23441;
        #line 149 "gpu_device.impala"
        bool _23464;
        _23464 = gid_x_23461 < _23463;
        #line 149 "gpu_device.impala"
        if (_23464) goto l23465; else goto l23530;
    l23530: ;
        #line 152 "gpu_device.impala"
        goto l23529;
    l23465: ;
        #line 146 "gpu_device.impala"
        int _23466;
        _23466 = _23447 * _23453;
        #line 11 "main.impala"
        int _23469;
        _23469 = _21087_23423.e3;
        #line 146 "gpu_device.impala"
        int gid_y_23467;
        gid_y_23467 = _23466 + _23459;
        #line 149 "gpu_device.impala"
        bool _23470;
        _23470 = gid_y_23467 < _23469;
        #line 149 "gpu_device.impala"
        if (_23470) goto l23471; else goto l23528;
    l23528: ;
        #line 152 "gpu_device.impala"
        goto l23529;
    l23529: ;
        return ;
    l23471: ;
        #line 35 "gpu_device.impala"
        char* _23512;
        _23512 = _21085_23421.e1;
        #line 25 "gaussian.impala"
        int _23501;
        _23501 = gid_y_23467 * _23463;
        #line 35 "gpu_device.impala"
        double* _23513;
        union { double* dst; char* src; } u_23513;
        u_23513.src = _23512;
        _23513 = u_23513.dst;
        #line 50 "gpu_device.impala"
        struct_Buffer_5540 _23498;
        _23498 = _21087_23423.e1;
        #line 25 "gaussian.impala"
        int _23514;
        _23514 = _23501 + gid_x_23461;
        #line 4 "gaussian.impala"
        int _23473;
        _23473 = _21086_23422.e1;
        #line 50 "gpu_device.impala"
        char* _23499;
        _23499 = _23498.e1;
        #line 34 "gpu_device.impala"
        double* index_23515;
        index_23515 = _23513 + _23514;
        #line 4 "gaussian.impala"
        int h_anchor_23475;
        h_anchor_23475 = _23473 / 2;
        #line 50 "gpu_device.impala"
        double* _23500;
        union { double* dst; char* src; } u_23500;
        u_23500.src = _23499;
        _23500 = u_23500.dst;
        #line 17 "gaussian.impala"
        bool _23476;
        _23476 = h_anchor_23475 <= gid_x_23461;
        #line 17 "gaussian.impala"
        if (_23476) goto l23477; else goto l23527;
    l23527: ;
        #line 27 "gaussian.impala"
        goto l23521;
    l23477: ;
        #line 17 "gaussian.impala"
        int _23478;
        _23478 = _23463 - h_anchor_23475;
        #line 17 "gaussian.impala"
        bool _23479;
        _23479 = gid_x_23461 < _23478;
        #line 17 "gaussian.impala"
        if (_23479) goto l23480; else goto l23520;
    l23520: ;
        #line 27 "gaussian.impala"
        goto l23521;
    l23521: ;
        #line 50 "gpu_device.impala"
        double* _23522;
        _23522 = _23500 + _23514;
        #line 50 "gpu_device.impala"
        double _23523;
        _23523 = *_23522;
        #line 50 "gpu_device.impala"
        double _23525;
        _23525 = _23523;
        #line 35 "gpu_device.impala"
        *index_23515 = _23525;
        return ;
    l23480: ;
        #line 19 "gaussian.impala"
        int _23486;
        _23486 = 1 + h_anchor_23475;
        #line 19 "gaussian.impala"
        int _23518;
        _23518 = 0 - h_anchor_23475;
        #line 55 "gpu_device.impala"
        struct_Buffer_5540 _23491;
        _23491 = _21086_23422.e0;
        #line 55 "gpu_device.impala"
        char* _23492;
        _23492 = _23491.e1;
        #line 55 "gpu_device.impala"
        double* _23493;
        union { double* dst; char* src; } u_23493;
        u_23493.src = _23492;
        _23493 = u_23493.dst;
        #line 19 "gpu_device.impala"
        p_23482 = _23518;
        psum_23484 = 0.000000e+00;
        goto l23481;
    l23481: ;
        _23482 = p_23482;
        sum_23484 = psum_23484;
        #line 19 "gpu_device.impala"
        bool _23487;
        _23487 = _23482 < _23486;
        #line 19 "gpu_device.impala"
        if (_23487) goto l23488; else goto l23511;
    l23511: ;
        #line 35 "gpu_device.impala"
        *index_23515 = sum_23484;
        return ;
    l23488: ;
        #line 21 "gaussian.impala"
        int _23502;
        _23502 = gid_x_23461 + _23482;
        #line 23 "gpu_device.impala"
        int _23489;
        _23489 = 1 + _23482;
        #line 21 "gaussian.impala"
        int _23494;
        _23494 = _23482 + h_anchor_23475;
        #line 50 "gpu_device.impala"
        int _23503;
        _23503 = _23501 + _23502;
        #line 54 "gpu_device.impala"
        double* i_23495;
        i_23495 = _23493 + _23494;
        #line 50 "gpu_device.impala"
        double* _23504;
        _23504 = _23500 + _23503;
        #line 55 "gpu_device.impala"
        double _23496;
        _23496 = *i_23495;
        #line 55 "gpu_device.impala"
        double _23507;
        _23507 = _23496;
        #line 50 "gpu_device.impala"
        double _23505;
        _23505 = *_23504;
        #line 50 "gpu_device.impala"
        double _23508;
        _23508 = _23505;
        #line 21 "gaussian.impala"
        double _23509;
        _23509 = _23507 * _23508;
        #line 21 "gaussian.impala"
        double _23510;
        _23510 = sum_23484 + _23509;
        #line 19 "gpu_device.impala"
        p_23482 = _23489;
        psum_23484 = _23510;
        goto l23481;
}

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_21214(struct_Buffer_5540 _21217_23534, struct_filter_5543 _21218_23535, double* _21219_23536, struct_image_5544 _21220_23537, struct_Buffer_5540 _21221_23538) {
    int  _23541;
    int p_23541;
    int  _23544;
    int p_23544;
    int  _23547;
    int p_23547;
    int  _23550;
    int p_23550;
    int  _23553;
    int p_23553;
    int  _23556;
    int p_23556;
    int  _23575;
    int p_23575;
    double  sum_23577;
    double psum_23577;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _23541 = blockIdx_x();
    p_23541 = _23541;
    l23539: ;
        _23541 = p_23541;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23544 = blockDim_x();
        p_23544 = _23544;
    l23542: ;
        _23544 = p_23544;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23547 = threadIdx_x();
        p_23547 = _23547;
    l23545: ;
        _23547 = p_23547;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23550 = blockIdx_y();
        p_23550 = _23550;
    l23548: ;
        _23550 = p_23550;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23553 = blockDim_y();
        p_23553 = _23553;
    l23551: ;
        _23553 = p_23553;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23556 = threadIdx_y();
        p_23556 = _23556;
    l23554: ;
        _23556 = p_23556;
        #line 11 "main.impala"
        int _23559;
        _23559 = _21220_23537.e2;
        #line 144 "gpu_device.impala"
        int _23557;
        _23557 = _23541 * _23544;
        #line 144 "gpu_device.impala"
        int gid_x_23558;
        gid_x_23558 = _23557 + _23547;
        #line 149 "gpu_device.impala"
        bool _23560;
        _23560 = gid_x_23558 < _23559;
        #line 149 "gpu_device.impala"
        if (_23560) goto l23561; else goto l23619;
    l23619: ;
        #line 152 "gpu_device.impala"
        goto l23618;
    l23561: ;
        #line 11 "main.impala"
        int _23564;
        _23564 = _21220_23537.e3;
        #line 146 "gpu_device.impala"
        int _23562;
        _23562 = _23550 * _23553;
        #line 146 "gpu_device.impala"
        int gid_y_23563;
        gid_y_23563 = _23562 + _23556;
        #line 149 "gpu_device.impala"
        bool _23565;
        _23565 = gid_y_23563 < _23564;
        #line 149 "gpu_device.impala"
        if (_23565) goto l23566; else goto l23617;
    l23617: ;
        #line 152 "gpu_device.impala"
        goto l23618;
    l23618: ;
        return ;
    l23566: ;
        #line 45 "gpu_device.impala"
        char* _23602;
        _23602 = _21221_23538.e1;
        #line 6 "gaussian.impala"
        int _23567;
        _23567 = _21218_23535.e2;
        #line 40 "gpu_device.impala"
        char* _23589;
        _23589 = _21217_23534.e1;
        #line 51 "gaussian.impala"
        int _23604;
        _23604 = gid_y_23563 * _23559;
        #line 45 "gpu_device.impala"
        double* _23603;
        union { double* dst; char* src; } u_23603;
        u_23603.src = _23602;
        _23603 = u_23603.dst;
        #line 6 "gaussian.impala"
        int v_anchor_23568;
        v_anchor_23568 = _23567 / 2;
        #line 40 "gpu_device.impala"
        double* _23590;
        union { double* dst; char* src; } u_23590;
        u_23590.src = _23589;
        _23590 = u_23590.dst;
        #line 51 "gaussian.impala"
        int _23605;
        _23605 = _23604 + gid_x_23558;
        #line 45 "gpu_device.impala"
        double* _23606;
        _23606 = _23603 + _23605;
        #line 39 "gaussian.impala"
        bool _23569;
        _23569 = v_anchor_23568 <= gid_y_23563;
        #line 39 "gaussian.impala"
        if (_23569) goto l23570; else goto l23616;
    l23616: ;
        #line 49 "gaussian.impala"
        goto l23610;
    l23570: ;
        #line 39 "gaussian.impala"
        int _23571;
        _23571 = _23564 - v_anchor_23568;
        #line 39 "gaussian.impala"
        bool _23572;
        _23572 = gid_y_23563 < _23571;
        #line 39 "gaussian.impala"
        if (_23572) goto l23573; else goto l23609;
    l23609: ;
        #line 49 "gaussian.impala"
        goto l23610;
    l23610: ;
        #line 39 "gpu_device.impala"
        double* index_23611;
        index_23611 = _23590 + _23605;
        #line 40 "gpu_device.impala"
        double _23612;
        _23612 = *index_23611;
        #line 40 "gpu_device.impala"
        double _23614;
        _23614 = _23612;
        #line 45 "gpu_device.impala"
        *_23606 = _23614;
        return ;
    l23573: ;
        #line 41 "gaussian.impala"
        int _23608;
        _23608 = 0 - v_anchor_23568;
        #line 41 "gaussian.impala"
        int _23578;
        _23578 = 1 + v_anchor_23568;
        #line 55 "gpu_device.impala"
        struct_Buffer_5540 _23582;
        _23582 = _21218_23535.e0;
        #line 55 "gpu_device.impala"
        char* _23583;
        _23583 = _23582.e1;
        #line 55 "gpu_device.impala"
        double* _23584;
        union { double* dst; char* src; } u_23584;
        u_23584.src = _23583;
        _23584 = u_23584.dst;
        #line 19 "gpu_device.impala"
        p_23575 = _23608;
        psum_23577 = 0.000000e+00;
        goto l23574;
    l23574: ;
        _23575 = p_23575;
        sum_23577 = psum_23577;
        #line 19 "gpu_device.impala"
        bool _23579;
        _23579 = _23575 < _23578;
        #line 19 "gpu_device.impala"
        if (_23579) goto l23580; else goto l23601;
    l23601: ;
        #line 45 "gpu_device.impala"
        *_23606 = sum_23577;
        return ;
    l23580: ;
        #line 23 "gpu_device.impala"
        int _23581;
        _23581 = 1 + _23575;
        #line 43 "gaussian.impala"
        int _23591;
        _23591 = gid_y_23563 + _23575;
        #line 43 "gaussian.impala"
        int _23585;
        _23585 = _23575 + v_anchor_23568;
        #line 43 "gaussian.impala"
        int _23592;
        _23592 = _23591 * _23559;
        #line 54 "gpu_device.impala"
        double* i_23586;
        i_23586 = _23584 + _23585;
        #line 43 "gaussian.impala"
        int _23593;
        _23593 = _23592 + gid_x_23558;
        #line 55 "gpu_device.impala"
        double _23587;
        _23587 = *i_23586;
        #line 39 "gpu_device.impala"
        double* index_23594;
        index_23594 = _23590 + _23593;
        #line 55 "gpu_device.impala"
        double _23597;
        _23597 = _23587;
        #line 40 "gpu_device.impala"
        double _23595;
        _23595 = *index_23594;
        #line 40 "gpu_device.impala"
        double _23598;
        _23598 = _23595;
        #line 43 "gaussian.impala"
        double _23599;
        _23599 = _23597 * _23598;
        #line 43 "gaussian.impala"
        double _23600;
        _23600 = sum_23577 + _23599;
        #line 19 "gpu_device.impala"
        p_23575 = _23581;
        psum_23577 = _23600;
        goto l23574;
}

}