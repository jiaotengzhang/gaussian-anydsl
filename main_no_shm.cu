
#include <hip/hip_runtime.h>
extern "C" {
typedef struct {
    int e0;
    char* e1;
} struct_Buffer_5426;
typedef struct {
    struct_Buffer_5426 e0;
    int e1;
    int e2;
} struct_filter_5425;
typedef struct {
    struct_Buffer_5426 e0;
    struct_Buffer_5426 e1;
    int e2;
    int e3;
} struct_image_5429;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_20628(struct_filter_5425, struct_image_5429, struct_Buffer_5426);
__global__ void lambda_20760(struct_image_5429, struct_filter_5425, double*, struct_Buffer_5426, struct_Buffer_5426);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_20628(struct_filter_5425 _20631_22967, struct_image_5429 _20632_22968, struct_Buffer_5426 _20633_22969) {
    int  _22975;
    int p_22975;
    int  _22981;
    int p_22981;
    int  _22987;
    int p_22987;
    int  _22993;
    int p_22993;
    int  _22999;
    int p_22999;
    int  _23005;
    int p_23005;
    int  _23028;
    int p_23028;
    double  sum_23030;
    double psum_23030;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _22975 = blockIdx_x();
    p_22975 = _22975;
    l22973: ;
        _22975 = p_22975;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22981 = blockDim_x();
        p_22981 = _22981;
    l22979: ;
        _22981 = p_22981;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22987 = threadIdx_x();
        p_22987 = _22987;
    l22985: ;
        _22987 = p_22987;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22993 = blockIdx_y();
        p_22993 = _22993;
    l22991: ;
        _22993 = p_22993;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22999 = blockDim_y();
        p_22999 = _22999;
    l22997: ;
        _22999 = p_22999;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23005 = threadIdx_y();
        p_23005 = _23005;
    l23003: ;
        _23005 = p_23005;
        #line 11 "main.impala"
        int _23009;
        _23009 = _20632_22968.e2;
        #line 155 "gpu_device.impala"
        int _23006;
        _23006 = _22975 * _22981;
        #line 155 "gpu_device.impala"
        int gid_x_23007;
        gid_x_23007 = _23006 + _22987;
        #line 160 "gpu_device.impala"
        bool _23010;
        _23010 = gid_x_23007 < _23009;
        #line 160 "gpu_device.impala"
        if (_23010) goto l23011; else goto l23076;
    l23076: ;
        #line 163 "gpu_device.impala"
        goto l23075;
    l23011: ;
        #line 11 "main.impala"
        int _23015;
        _23015 = _20632_22968.e3;
        #line 157 "gpu_device.impala"
        int _23012;
        _23012 = _22993 * _22999;
        #line 157 "gpu_device.impala"
        int gid_y_23013;
        gid_y_23013 = _23012 + _23005;
        #line 160 "gpu_device.impala"
        bool _23016;
        _23016 = gid_y_23013 < _23015;
        #line 160 "gpu_device.impala"
        if (_23016) goto l23017; else goto l23074;
    l23074: ;
        #line 163 "gpu_device.impala"
        goto l23075;
    l23075: ;
        return ;
    l23017: ;
        #line 50 "gpu_device.impala"
        int _23047;
        _23047 = gid_y_23013 * _23009;
        #line 45 "gpu_device.impala"
        char* _23058;
        _23058 = _20633_22969.e1;
        #line 4 "gaussian.impala"
        int _23019;
        _23019 = _20631_22967.e1;
        #line 45 "gpu_device.impala"
        double* _23059;
        union { double* dst; char* src; } u_23059;
        u_23059.src = _23058;
        _23059 = u_23059.dst;
        #line 50 "gpu_device.impala"
        struct_Buffer_5426 _23044;
        _23044 = _20632_22968.e1;
        #line 4 "gaussian.impala"
        int h_anchor_23021;
        h_anchor_23021 = _23019 / 2;
        #line 17 "gaussian.impala"
        bool _23022;
        _23022 = h_anchor_23021 <= gid_x_23007;
        #line 50 "gpu_device.impala"
        int _23060;
        _23060 = _23047 + gid_x_23007;
        #line 45 "gpu_device.impala"
        double* _23061;
        _23061 = _23059 + _23060;
        #line 50 "gpu_device.impala"
        char* _23045;
        _23045 = _23044.e1;
        #line 50 "gpu_device.impala"
        double* _23046;
        union { double* dst; char* src; } u_23046;
        u_23046.src = _23045;
        _23046 = u_23046.dst;
        #line 17 "gaussian.impala"
        if (_23022) goto l23023; else goto l23073;
    l23073: ;
        #line 27 "gaussian.impala"
        goto l23067;
    l23023: ;
        #line 17 "gaussian.impala"
        int _23024;
        _23024 = _23009 - h_anchor_23021;
        #line 17 "gaussian.impala"
        bool _23025;
        _23025 = gid_x_23007 < _23024;
        #line 17 "gaussian.impala"
        if (_23025) goto l23026; else goto l23066;
    l23066: ;
        #line 27 "gaussian.impala"
        goto l23067;
    l23067: ;
        #line 50 "gpu_device.impala"
        double* _23068;
        _23068 = _23046 + _23060;
        #line 50 "gpu_device.impala"
        double _23069;
        _23069 = *_23068;
        #line 50 "gpu_device.impala"
        double _23071;
        _23071 = _23069;
        #line 45 "gpu_device.impala"
        *_23061 = _23071;
        return ;
    l23026: ;
        #line 55 "gpu_device.impala"
        struct_Buffer_5426 _23037;
        _23037 = _20631_22967.e0;
        #line 19 "gaussian.impala"
        int _23032;
        _23032 = 1 + h_anchor_23021;
        #line 19 "gaussian.impala"
        int _23064;
        _23064 = 0 - h_anchor_23021;
        #line 55 "gpu_device.impala"
        char* _23038;
        _23038 = _23037.e1;
        #line 55 "gpu_device.impala"
        double* _23039;
        union { double* dst; char* src; } u_23039;
        u_23039.src = _23038;
        _23039 = u_23039.dst;
        #line 19 "gpu_device.impala"
        p_23028 = _23064;
        psum_23030 = 0.000000e+00;
        goto l23027;
    l23027: ;
        _23028 = p_23028;
        sum_23030 = psum_23030;
        #line 19 "gpu_device.impala"
        bool _23033;
        _23033 = _23028 < _23032;
        #line 19 "gpu_device.impala"
        if (_23033) goto l23034; else goto l23057;
    l23057: ;
        #line 45 "gpu_device.impala"
        *_23061 = sum_23030;
        return ;
    l23034: ;
        #line 23 "gpu_device.impala"
        int _23035;
        _23035 = 1 + _23028;
        #line 21 "gaussian.impala"
        int _23048;
        _23048 = gid_x_23007 + _23028;
        #line 21 "gaussian.impala"
        int _23040;
        _23040 = _23028 + h_anchor_23021;
        #line 50 "gpu_device.impala"
        int _23049;
        _23049 = _23047 + _23048;
        #line 54 "gpu_device.impala"
        double* i_23041;
        i_23041 = _23039 + _23040;
        #line 50 "gpu_device.impala"
        double* _23050;
        _23050 = _23046 + _23049;
        #line 55 "gpu_device.impala"
        double _23042;
        _23042 = *i_23041;
        #line 55 "gpu_device.impala"
        double _23053;
        _23053 = _23042;
        #line 50 "gpu_device.impala"
        double _23051;
        _23051 = *_23050;
        #line 50 "gpu_device.impala"
        double _23054;
        _23054 = _23051;
        #line 21 "gaussian.impala"
        double _23055;
        _23055 = _23053 * _23054;
        #line 21 "gaussian.impala"
        double _23056;
        _23056 = sum_23030 + _23055;
        #line 19 "gpu_device.impala"
        p_23028 = _23035;
        psum_23030 = _23056;
        goto l23027;
}

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_20760(struct_image_5429 _20763_23080, struct_filter_5425 _20764_23081, double* _20765_23082, struct_Buffer_5426 _20766_23083, struct_Buffer_5426 _20767_23084) {
    int  _23087;
    int p_23087;
    int  _23090;
    int p_23090;
    int  _23093;
    int p_23093;
    int  _23096;
    int p_23096;
    int  _23099;
    int p_23099;
    int  _23102;
    int p_23102;
    int  _23121;
    int p_23121;
    double  sum_23123;
    double psum_23123;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _23087 = blockIdx_x();
    p_23087 = _23087;
    l23085: ;
        _23087 = p_23087;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23090 = blockDim_x();
        p_23090 = _23090;
    l23088: ;
        _23090 = p_23090;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23093 = threadIdx_x();
        p_23093 = _23093;
    l23091: ;
        _23093 = p_23093;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23096 = blockIdx_y();
        p_23096 = _23096;
    l23094: ;
        _23096 = p_23096;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23099 = blockDim_y();
        p_23099 = _23099;
    l23097: ;
        _23099 = p_23099;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23102 = threadIdx_y();
        p_23102 = _23102;
    l23100: ;
        _23102 = p_23102;
        #line 155 "gpu_device.impala"
        int _23103;
        _23103 = _23087 * _23090;
        #line 11 "main.impala"
        int _23105;
        _23105 = _20763_23080.e2;
        #line 155 "gpu_device.impala"
        int gid_x_23104;
        gid_x_23104 = _23103 + _23093;
        #line 160 "gpu_device.impala"
        bool _23106;
        _23106 = gid_x_23104 < _23105;
        #line 160 "gpu_device.impala"
        if (_23106) goto l23107; else goto l23165;
    l23165: ;
        #line 163 "gpu_device.impala"
        goto l23164;
    l23107: ;
        #line 11 "main.impala"
        int _23110;
        _23110 = _20763_23080.e3;
        #line 157 "gpu_device.impala"
        int _23108;
        _23108 = _23096 * _23099;
        #line 157 "gpu_device.impala"
        int gid_y_23109;
        gid_y_23109 = _23108 + _23102;
        #line 160 "gpu_device.impala"
        bool _23111;
        _23111 = gid_y_23109 < _23110;
        #line 160 "gpu_device.impala"
        if (_23111) goto l23112; else goto l23163;
    l23163: ;
        #line 163 "gpu_device.impala"
        goto l23164;
    l23164: ;
        return ;
    l23112: ;
        #line 50 "gpu_device.impala"
        int _23150;
        _23150 = gid_y_23109 * _23105;
        #line 6 "gaussian.impala"
        int _23113;
        _23113 = _20764_23081.e2;
        #line 45 "gpu_device.impala"
        char* _23148;
        _23148 = _20766_23083.e1;
        #line 6 "gaussian.impala"
        int v_anchor_23114;
        v_anchor_23114 = _23113 / 2;
        #line 45 "gpu_device.impala"
        double* _23149;
        union { double* dst; char* src; } u_23149;
        u_23149.src = _23148;
        _23149 = u_23149.dst;
        #line 50 "gpu_device.impala"
        char* _23135;
        _23135 = _20767_23084.e1;
        #line 50 "gpu_device.impala"
        int _23151;
        _23151 = _23150 + gid_x_23104;
        #line 39 "gaussian.impala"
        bool _23115;
        _23115 = v_anchor_23114 <= gid_y_23109;
        #line 45 "gpu_device.impala"
        double* _23152;
        _23152 = _23149 + _23151;
        #line 50 "gpu_device.impala"
        double* _23136;
        union { double* dst; char* src; } u_23136;
        u_23136.src = _23135;
        _23136 = u_23136.dst;
        #line 39 "gaussian.impala"
        if (_23115) goto l23116; else goto l23162;
    l23162: ;
        #line 49 "gaussian.impala"
        goto l23156;
    l23116: ;
        #line 39 "gaussian.impala"
        int _23117;
        _23117 = _23110 - v_anchor_23114;
        #line 39 "gaussian.impala"
        bool _23118;
        _23118 = gid_y_23109 < _23117;
        #line 39 "gaussian.impala"
        if (_23118) goto l23119; else goto l23155;
    l23155: ;
        #line 49 "gaussian.impala"
        goto l23156;
    l23156: ;
        #line 50 "gpu_device.impala"
        double* _23157;
        _23157 = _23136 + _23151;
        #line 50 "gpu_device.impala"
        double _23158;
        _23158 = *_23157;
        #line 50 "gpu_device.impala"
        double _23160;
        _23160 = _23158;
        #line 45 "gpu_device.impala"
        *_23152 = _23160;
        return ;
    l23119: ;
        #line 41 "gaussian.impala"
        int _23124;
        _23124 = 1 + v_anchor_23114;
        #line 41 "gaussian.impala"
        int _23154;
        _23154 = 0 - v_anchor_23114;
        #line 55 "gpu_device.impala"
        struct_Buffer_5426 _23128;
        _23128 = _20764_23081.e0;
        #line 55 "gpu_device.impala"
        char* _23129;
        _23129 = _23128.e1;
        #line 55 "gpu_device.impala"
        double* _23130;
        union { double* dst; char* src; } u_23130;
        u_23130.src = _23129;
        _23130 = u_23130.dst;
        #line 19 "gpu_device.impala"
        p_23121 = _23154;
        psum_23123 = 0.000000e+00;
        goto l23120;
    l23120: ;
        _23121 = p_23121;
        sum_23123 = psum_23123;
        #line 19 "gpu_device.impala"
        bool _23125;
        _23125 = _23121 < _23124;
        #line 19 "gpu_device.impala"
        if (_23125) goto l23126; else goto l23147;
    l23147: ;
        #line 45 "gpu_device.impala"
        *_23152 = sum_23123;
        return ;
    l23126: ;
        #line 43 "gaussian.impala"
        int _23137;
        _23137 = gid_y_23109 + _23121;
        #line 50 "gpu_device.impala"
        int _23138;
        _23138 = _23137 * _23105;
        #line 23 "gpu_device.impala"
        int _23127;
        _23127 = 1 + _23121;
        #line 43 "gaussian.impala"
        int _23131;
        _23131 = _23121 + v_anchor_23114;
        #line 50 "gpu_device.impala"
        int _23139;
        _23139 = _23138 + gid_x_23104;
        #line 54 "gpu_device.impala"
        double* i_23132;
        i_23132 = _23130 + _23131;
        #line 50 "gpu_device.impala"
        double* _23140;
        _23140 = _23136 + _23139;
        #line 55 "gpu_device.impala"
        double _23133;
        _23133 = *i_23132;
        #line 55 "gpu_device.impala"
        double _23143;
        _23143 = _23133;
        #line 50 "gpu_device.impala"
        double _23141;
        _23141 = *_23140;
        #line 50 "gpu_device.impala"
        double _23144;
        _23144 = _23141;
        #line 43 "gaussian.impala"
        double _23145;
        _23145 = _23143 * _23144;
        #line 43 "gaussian.impala"
        double _23146;
        _23146 = sum_23123 + _23145;
        #line 19 "gpu_device.impala"
        p_23121 = _23127;
        psum_23123 = _23146;
        goto l23120;
}

}