
#include <hip/hip_runtime.h>
extern "C" {
typedef struct {
    int e0;
    char* e1;
} struct_Buffer_6429;
typedef struct {
    struct_Buffer_6429 e0;
    struct_Buffer_6429 e1;
    int e2;
    int e3;
} struct_image_6428;
typedef struct {
    struct_Buffer_6429 e0;
    int e1;
    int e2;
} struct_filter_6434;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_26808(struct_image_6428, double*, struct_Buffer_6429, struct_filter_6434);
__global__ void lambda_27103(struct_image_6428, double*, struct_Buffer_6429, double*, struct_filter_6434);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_26808(struct_image_6428 _26811_32339, double* _26812_32340, struct_Buffer_6429 _26813_32341, struct_filter_6434 _26814_32342) {
    int  _32348;
    int p_32348;
    int  _32354;
    int p_32354;
    int  _32360;
    int p_32360;
    int  _32366;
    int p_32366;
    int  _32372;
    int p_32372;
    int  _32378;
    int p_32378;
    int  _32381;
    int p_32381;
    int  _32384;
    int p_32384;
    double*  reserve_shared_32392;
    double* preserve_shared_32392;
    int  _32395;
    int p_32395;
    int  _32398;
    int p_32398;
    int  _32401;
    int p_32401;
    int  _32404;
    int p_32404;
    int  _32407;
    int p_32407;
    int  _32419;
    int p_32419;
    int  _32421;
    int p_32421;
    int  _32483;
    int p_32483;
    int  _32558;
    int p_32558;
    double  sum_32560;
    double psum_32560;
    int  _32489;
    int p_32489;
    int  _32491;
    int p_32491;
    int  _32497;
    int p_32497;
    int  _32500;
    int p_32500;
    int  _32432;
    int p_32432;
    int  _32435;
    int p_32435;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _32348 = blockIdx_x();
    p_32348 = _32348;
    l32346: ;
        _32348 = p_32348;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32354 = blockDim_x();
        p_32354 = _32354;
    l32352: ;
        _32354 = p_32354;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32360 = threadIdx_x();
        p_32360 = _32360;
    l32358: ;
        _32360 = p_32360;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32366 = blockIdx_y();
        p_32366 = _32366;
    l32364: ;
        _32366 = p_32366;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32372 = blockDim_y();
        p_32372 = _32372;
    l32370: ;
        _32372 = p_32372;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32378 = threadIdx_y();
        p_32378 = _32378;
    l32376: ;
        _32378 = p_32378;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32381 = blockDim_x();
        p_32381 = _32381;
    l32379: ;
        _32381 = p_32381;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32384 = blockDim_y();
        p_32384 = _32384;
    l32382: ;
        _32384 = p_32384;
        #line 191 "gpu_device_shm.impala"
        __shared__ double reserver_reserve_shared_32392[938];
        preserve_shared_32392 = reserver_reserve_shared_32392;
    l32390: ;
        reserve_shared_32392 = preserve_shared_32392;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32395 = blockIdx_x();
        p_32395 = _32395;
    l32393: ;
        _32395 = p_32395;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32398 = blockDim_x();
        p_32398 = _32398;
    l32396: ;
        _32398 = p_32398;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32401 = blockIdx_y();
        p_32401 = _32401;
    l32399: ;
        _32401 = p_32401;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32404 = blockDim_y();
        p_32404 = _32404;
    l32402: ;
        _32404 = p_32404;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32407 = blockDim_y();
        p_32407 = _32407;
    l32405: ;
        _32407 = p_32407;
        #line 177 "gpu_device_shm.impala"
        int _32441;
        _32441 = _32348 * _32354;
        #line 11 "main.impala"
        int _32456;
        _32456 = _26811_32339.e3;
        #line 183 "gpu_device_shm.impala"
        int _32411;
        _32411 = _26814_32342.e2;
        #line 179 "gpu_device_shm.impala"
        int _32450;
        _32450 = _32366 * _32372;
        #line 11 "main.impala"
        int _32447;
        _32447 = _26811_32339.e2;
        #line 4 "gaussian.impala"
        int _32424;
        _32424 = _26814_32342.e1;
        #line 177 "gpu_device_shm.impala"
        int gid_x_32442;
        gid_x_32442 = _32441 + _32360;
        #line 183 "gpu_device_shm.impala"
        int extend_height_32412;
        extend_height_32412 = _32411 / 2;
        #line 179 "gpu_device_shm.impala"
        int gid_y_32451;
        gid_y_32451 = _32450 + _32378;
        #line 4 "gaussian.impala"
        int h_anchor_32425;
        h_anchor_32425 = _32424 / 2;
        #line 187 "gpu_device_shm.impala"
        int _32413;
        _32413 = 2 * extend_height_32412;
        #line 185 "gpu_device_shm.impala"
        int _32426;
        _32426 = 2 * h_anchor_32425;
        #line 187 "gpu_device_shm.impala"
        int shm_dimy_32414;
        shm_dimy_32414 = _32384 + _32413;
        #line 185 "gpu_device_shm.impala"
        int shm_dimx_32427;
        shm_dimx_32427 = _32381 + _32426;
        #line 38 "gpu_device_shm.impala"
        bool _32415;
        _32415 = 0 < shm_dimy_32414;
        #line 38 "gpu_device_shm.impala"
        if (_32415) goto l32416; else goto l32611;
    l32611: ;
        #line 221 "gpu_device_shm.impala"
        goto l32541;
    l32416: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32419 = blockDim_x();
        p_32419 = _32419;
    l32417: ;
        _32419 = p_32419;
        #line 211 "gpu_device_shm.impala"
        int _32452;
        _32452 = gid_y_32451 - extend_height_32412;
        #line 209 "gpu_device_shm.impala"
        int _32443;
        _32443 = gid_x_32442 - h_anchor_32425;
        #line 214 "gpu_device_shm.impala"
        bool _32457;
        _32457 = _32452 < _32456;
        #line 214 "gpu_device_shm.impala"
        bool _32453;
        _32453 = 0 <= _32452;
        #line 217 "gpu_device_shm.impala"
        int _32462;
        _32462 = _32452 * _32447;
        #line 38 "gpu_device_shm.impala"
        p_32421 = 0;
        goto l32420;
    l32420: ;
        _32421 = p_32421;
        #line 38 "gpu_device_shm.impala"
        bool _32428;
        _32428 = _32421 < shm_dimx_32427;
        #line 38 "gpu_device_shm.impala"
        if (_32428) goto l32429; else goto l32481;
    l32481: ;
        #line 38 "gpu_device_shm.impala"
        p_32483 = _32407;
        goto l32482;
    l32482: ;
        _32483 = p_32483;
        #line 38 "gpu_device_shm.impala"
        bool _32485;
        _32485 = _32483 < shm_dimy_32414;
        #line 38 "gpu_device_shm.impala"
        if (_32485) goto l32486; else goto l32540;
    l32540: ;
        #line 221 "gpu_device_shm.impala"
        goto l32541;
    l32541: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        __syncthreads();
    l32546: ;
        #line 227 "gpu_device_shm.impala"
        bool _32548;
        _32548 = gid_x_32442 < _32447;
        #line 227 "gpu_device_shm.impala"
        if (_32548) goto l32549; else goto l32610;
    l32610: ;
        #line 230 "gpu_device_shm.impala"
        goto l32609;
    l32549: ;
        #line 227 "gpu_device_shm.impala"
        bool _32550;
        _32550 = gid_y_32451 < _32456;
        #line 227 "gpu_device_shm.impala"
        if (_32550) goto l32551; else goto l32608;
    l32608: ;
        #line 230 "gpu_device_shm.impala"
        goto l32609;
    l32609: ;
        return ;
    l32551: ;
        #line 17 "gaussian.impala"
        bool _32552;
        _32552 = h_anchor_32425 <= gid_x_32442;
        #line 194 "gpu_device_shm.impala"
        int _32573;
        _32573 = _32401 * _32404;
        #line 194 "gpu_device_shm.impala"
        int _32574;
        _32574 = extend_height_32412 - _32573;
        #line 64 "gpu_device_shm.impala"
        int _32592;
        _32592 = gid_y_32451 * _32447;
        #line 64 "gpu_device_shm.impala"
        int _32593;
        _32593 = _32592 + gid_x_32442;
        #line 193 "gpu_device_shm.impala"
        int _32579;
        _32579 = _32395 * _32398;
        #line 193 "gpu_device_shm.impala"
        int _32580;
        _32580 = h_anchor_32425 - _32579;
        #line 64 "gpu_device_shm.impala"
        char* _32590;
        _32590 = _26813_32341.e1;
        #line 72 "gpu_device_shm.impala"
        int _32575;
        _32575 = gid_y_32451 + _32574;
        #line 64 "gpu_device_shm.impala"
        double* _32591;
        union { double* dst; char* src; } u_32591;
        u_32591.src = _32590;
        _32591 = u_32591.dst;
        #line 72 "gpu_device_shm.impala"
        int _32576;
        _32576 = _32575 * shm_dimx_32427;
        #line 64 "gpu_device_shm.impala"
        double* _32594;
        _32594 = _32591 + _32593;
        #line 17 "gaussian.impala"
        if (_32552) goto l32553; else goto l32607;
    l32607: ;
        #line 27 "gaussian.impala"
        goto l32599;
    l32553: ;
        #line 17 "gaussian.impala"
        int _32554;
        _32554 = _32447 - h_anchor_32425;
        #line 17 "gaussian.impala"
        bool _32555;
        _32555 = gid_x_32442 < _32554;
        #line 17 "gaussian.impala"
        if (_32555) goto l32556; else goto l32598;
    l32598: ;
        #line 27 "gaussian.impala"
        goto l32599;
    l32599: ;
        #line 72 "gpu_device_shm.impala"
        int _32600;
        _32600 = _32576 + gid_x_32442;
        #line 72 "gpu_device_shm.impala"
        int _32601;
        _32601 = _32600 + _32580;
        #line 72 "gpu_device_shm.impala"
        double* _32602;
        _32602 = reserve_shared_32392 + _32601;
        #line 72 "gpu_device_shm.impala"
        double _32603;
        _32603 = *_32602;
        #line 72 "gpu_device_shm.impala"
        double _32605;
        _32605 = _32603;
        #line 64 "gpu_device_shm.impala"
        *_32594 = _32605;
        return ;
    l32556: ;
        #line 19 "gaussian.impala"
        int _32562;
        _32562 = 1 + h_anchor_32425;
        #line 77 "gpu_device_shm.impala"
        struct_Buffer_6429 _32566;
        _32566 = _26814_32342.e0;
        #line 19 "gaussian.impala"
        int _32596;
        _32596 = 0 - h_anchor_32425;
        #line 77 "gpu_device_shm.impala"
        char* _32567;
        _32567 = _32566.e1;
        #line 77 "gpu_device_shm.impala"
        double* _32568;
        union { double* dst; char* src; } u_32568;
        u_32568.src = _32567;
        _32568 = u_32568.dst;
        #line 27 "gpu_device_shm.impala"
        p_32558 = _32596;
        psum_32560 = 0.000000e+00;
        goto l32557;
    l32557: ;
        _32558 = p_32558;
        sum_32560 = psum_32560;
        #line 27 "gpu_device_shm.impala"
        bool _32563;
        _32563 = _32558 < _32562;
        #line 27 "gpu_device_shm.impala"
        if (_32563) goto l32564; else goto l32589;
    l32589: ;
        #line 64 "gpu_device_shm.impala"
        *_32594 = sum_32560;
        return ;
    l32564: ;
        #line 21 "gaussian.impala"
        int _32577;
        _32577 = gid_x_32442 + _32558;
        #line 72 "gpu_device_shm.impala"
        int _32578;
        _32578 = _32576 + _32577;
        #line 21 "gaussian.impala"
        int _32569;
        _32569 = _32558 + h_anchor_32425;
        #line 31 "gpu_device_shm.impala"
        int _32565;
        _32565 = 1 + _32558;
        #line 72 "gpu_device_shm.impala"
        int _32581;
        _32581 = _32578 + _32580;
        #line 76 "gpu_device_shm.impala"
        double* i_32570;
        i_32570 = _32568 + _32569;
        #line 72 "gpu_device_shm.impala"
        double* _32582;
        _32582 = reserve_shared_32392 + _32581;
        #line 77 "gpu_device_shm.impala"
        double _32571;
        _32571 = *i_32570;
        #line 77 "gpu_device_shm.impala"
        double _32585;
        _32585 = _32571;
        #line 72 "gpu_device_shm.impala"
        double _32583;
        _32583 = *_32582;
        #line 72 "gpu_device_shm.impala"
        double _32586;
        _32586 = _32583;
        #line 21 "gaussian.impala"
        double _32587;
        _32587 = _32585 * _32586;
        #line 21 "gaussian.impala"
        double _32588;
        _32588 = sum_32560 + _32587;
        #line 27 "gpu_device_shm.impala"
        p_32558 = _32565;
        psum_32560 = _32588;
        goto l32557;
    l32486: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32489 = blockDim_x();
        p_32489 = _32489;
    l32487: ;
        _32489 = p_32489;
        #line 211 "gpu_device_shm.impala"
        int img_index_y_32512;
        img_index_y_32512 = _32452 + _32483;
        #line 214 "gpu_device_shm.impala"
        bool _32513;
        _32513 = 0 <= img_index_y_32512;
        #line 217 "gpu_device_shm.impala"
        int _32520;
        _32520 = img_index_y_32512 * _32447;
        #line 214 "gpu_device_shm.impala"
        bool _32515;
        _32515 = img_index_y_32512 < _32456;
        #line 38 "gpu_device_shm.impala"
        p_32491 = 0;
        goto l32490;
    l32490: ;
        _32491 = p_32491;
        #line 38 "gpu_device_shm.impala"
        bool _32493;
        _32493 = _32491 < shm_dimx_32427;
        #line 38 "gpu_device_shm.impala"
        if (_32493) goto l32494; else goto l32538;
    l32538: ;
        #line 42 "gpu_device_shm.impala"
        int _32539;
        _32539 = _32483 + _32407;
        #line 38 "gpu_device_shm.impala"
        p_32483 = _32539;
        goto l32482;
    l32494: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32497 = threadIdx_x();
        p_32497 = _32497;
    l32495: ;
        _32497 = p_32497;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32500 = threadIdx_y();
        p_32500 = _32500;
    l32498: ;
        _32500 = p_32500;
        #line 202 "gpu_device_shm.impala"
        int shm_index_x_32501;
        shm_index_x_32501 = _32497 + _32491;
        #line 207 "gpu_device_shm.impala"
        bool _32502;
        _32502 = shm_index_x_32501 < shm_dimx_32427;
        #line 207 "gpu_device_shm.impala"
        if (_32502) goto l32503; else goto l32537;
    l32537: ;
        #line 219 "gpu_device_shm.impala"
        goto l32536;
    l32503: ;
        #line 204 "gpu_device_shm.impala"
        int shm_index_y_32504;
        shm_index_y_32504 = _32500 + _32483;
        #line 207 "gpu_device_shm.impala"
        bool _32505;
        _32505 = shm_index_y_32504 < shm_dimy_32414;
        #line 207 "gpu_device_shm.impala"
        if (_32505) goto l32506; else goto l32535;
    l32535: ;
        #line 219 "gpu_device_shm.impala"
        goto l32536;
    l32536: ;
        #line 40 "gpu_device_shm.impala"
        goto l32517;
    l32506: ;
        #line 209 "gpu_device_shm.impala"
        int img_index_x_32507;
        img_index_x_32507 = _32443 + _32491;
        #line 214 "gpu_device_shm.impala"
        bool _32508;
        _32508 = 0 <= img_index_x_32507;
        #line 214 "gpu_device_shm.impala"
        if (_32508) goto l32509; else goto l32534;
    l32534: ;
        #line 218 "gpu_device_shm.impala"
        goto l32531;
    l32509: ;
        #line 214 "gpu_device_shm.impala"
        bool _32510;
        _32510 = img_index_x_32507 < _32447;
        #line 214 "gpu_device_shm.impala"
        if (_32510) goto l32511; else goto l32533;
    l32533: ;
        #line 218 "gpu_device_shm.impala"
        goto l32531;
    l32511: ;
        #line 214 "gpu_device_shm.impala"
        if (_32513) goto l32514; else goto l32532;
    l32532: ;
        #line 218 "gpu_device_shm.impala"
        goto l32531;
    l32514: ;
        #line 214 "gpu_device_shm.impala"
        if (_32515) goto l32516; else goto l32530;
    l32530: ;
        #line 218 "gpu_device_shm.impala"
        goto l32531;
    l32531: ;
        #line 40 "gpu_device_shm.impala"
        goto l32517;
    l32516: ;
        #line 216 "gpu_device_shm.impala"
        int _32525;
        _32525 = shm_index_y_32504 * shm_dimx_32427;
        #line 217 "gpu_device_shm.impala"
        int _32521;
        _32521 = _32520 + img_index_x_32507;
        #line 216 "gpu_device_shm.impala"
        int _32526;
        _32526 = _32525 + shm_index_x_32501;
        #line 217 "gpu_device_shm.impala"
        double* _32522;
        _32522 = _26812_32340 + _32521;
        #line 216 "gpu_device_shm.impala"
        double* _32527;
        _32527 = reserve_shared_32392 + _32526;
        #line 217 "gpu_device_shm.impala"
        double _32523;
        _32523 = *_32522;
        #line 217 "gpu_device_shm.impala"
        double _32528;
        _32528 = _32523;
        #line 216 "gpu_device_shm.impala"
        *_32527 = _32528;
        #line 40 "gpu_device_shm.impala"
        goto l32517;
    l32517: ;
        #line 42 "gpu_device_shm.impala"
        int _32519;
        _32519 = _32491 + _32489;
        #line 38 "gpu_device_shm.impala"
        p_32491 = _32519;
        goto l32490;
    l32429: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32432 = threadIdx_x();
        p_32432 = _32432;
    l32430: ;
        _32432 = p_32432;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32435 = threadIdx_y();
        p_32435 = _32435;
    l32433: ;
        _32435 = p_32435;
        #line 202 "gpu_device_shm.impala"
        int shm_index_x_32436;
        shm_index_x_32436 = _32432 + _32421;
        #line 207 "gpu_device_shm.impala"
        bool _32437;
        _32437 = shm_index_x_32436 < shm_dimx_32427;
        #line 207 "gpu_device_shm.impala"
        if (_32437) goto l32438; else goto l32480;
    l32480: ;
        #line 219 "gpu_device_shm.impala"
        goto l32479;
    l32438: ;
        #line 207 "gpu_device_shm.impala"
        bool _32439;
        _32439 = _32435 < shm_dimy_32414;
        #line 207 "gpu_device_shm.impala"
        if (_32439) goto l32440; else goto l32478;
    l32478: ;
        #line 219 "gpu_device_shm.impala"
        goto l32479;
    l32479: ;
        #line 40 "gpu_device_shm.impala"
        goto l32459;
    l32440: ;
        #line 209 "gpu_device_shm.impala"
        int img_index_x_32444;
        img_index_x_32444 = _32443 + _32421;
        #line 214 "gpu_device_shm.impala"
        bool _32445;
        _32445 = 0 <= img_index_x_32444;
        #line 214 "gpu_device_shm.impala"
        if (_32445) goto l32446; else goto l32477;
    l32477: ;
        #line 218 "gpu_device_shm.impala"
        goto l32474;
    l32446: ;
        #line 214 "gpu_device_shm.impala"
        bool _32448;
        _32448 = img_index_x_32444 < _32447;
        #line 214 "gpu_device_shm.impala"
        if (_32448) goto l32449; else goto l32476;
    l32476: ;
        #line 218 "gpu_device_shm.impala"
        goto l32474;
    l32449: ;
        #line 214 "gpu_device_shm.impala"
        if (_32453) goto l32454; else goto l32475;
    l32475: ;
        #line 218 "gpu_device_shm.impala"
        goto l32474;
    l32454: ;
        #line 214 "gpu_device_shm.impala"
        if (_32457) goto l32458; else goto l32473;
    l32473: ;
        #line 218 "gpu_device_shm.impala"
        goto l32474;
    l32474: ;
        #line 40 "gpu_device_shm.impala"
        goto l32459;
    l32458: ;
        #line 216 "gpu_device_shm.impala"
        int _32468;
        _32468 = _32435 * shm_dimx_32427;
        #line 217 "gpu_device_shm.impala"
        int _32463;
        _32463 = _32462 + img_index_x_32444;
        #line 216 "gpu_device_shm.impala"
        int _32469;
        _32469 = _32468 + shm_index_x_32436;
        #line 217 "gpu_device_shm.impala"
        double* _32464;
        _32464 = _26812_32340 + _32463;
        #line 217 "gpu_device_shm.impala"
        double _32465;
        _32465 = *_32464;
        #line 216 "gpu_device_shm.impala"
        double* _32470;
        _32470 = reserve_shared_32392 + _32469;
        #line 217 "gpu_device_shm.impala"
        double _32471;
        _32471 = _32465;
        #line 216 "gpu_device_shm.impala"
        *_32470 = _32471;
        #line 40 "gpu_device_shm.impala"
        goto l32459;
    l32459: ;
        #line 42 "gpu_device_shm.impala"
        int _32461;
        _32461 = _32421 + _32419;
        #line 38 "gpu_device_shm.impala"
        p_32421 = _32461;
        goto l32420;
}

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_27103(struct_image_6428 _27106_32615, double* _27107_32616, struct_Buffer_6429 _27108_32617, double* _27109_32618, struct_filter_6434 _27110_32619) {
    int  _32622;
    int p_32622;
    int  _32625;
    int p_32625;
    int  _32628;
    int p_32628;
    int  _32631;
    int p_32631;
    int  _32634;
    int p_32634;
    int  _32637;
    int p_32637;
    int  _32640;
    int p_32640;
    int  _32643;
    int p_32643;
    double*  reserve_shared_32646;
    double* preserve_shared_32646;
    int  _32649;
    int p_32649;
    int  _32652;
    int p_32652;
    int  _32655;
    int p_32655;
    int  _32658;
    int p_32658;
    int  _32661;
    int p_32661;
    int  _32670;
    int p_32670;
    int  _32672;
    int p_32672;
    int  _32731;
    int p_32731;
    int  _32803;
    int p_32803;
    double  sum_32805;
    double psum_32805;
    int  _32737;
    int p_32737;
    int  _32739;
    int p_32739;
    int  _32745;
    int p_32745;
    int  _32748;
    int p_32748;
    int  _32682;
    int p_32682;
    int  _32685;
    int p_32685;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _32622 = blockIdx_x();
    p_32622 = _32622;
    l32620: ;
        _32622 = p_32622;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32625 = blockDim_x();
        p_32625 = _32625;
    l32623: ;
        _32625 = p_32625;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32628 = threadIdx_x();
        p_32628 = _32628;
    l32626: ;
        _32628 = p_32628;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32631 = blockIdx_y();
        p_32631 = _32631;
    l32629: ;
        _32631 = p_32631;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32634 = blockDim_y();
        p_32634 = _32634;
    l32632: ;
        _32634 = p_32634;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32637 = threadIdx_y();
        p_32637 = _32637;
    l32635: ;
        _32637 = p_32637;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32640 = blockDim_x();
        p_32640 = _32640;
    l32638: ;
        _32640 = p_32640;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32643 = blockDim_y();
        p_32643 = _32643;
    l32641: ;
        _32643 = p_32643;
        #line 191 "gpu_device_shm.impala"
        __shared__ double reserver_reserve_shared_32646[938];
        preserve_shared_32646 = reserver_reserve_shared_32646;
    l32644: ;
        reserve_shared_32646 = preserve_shared_32646;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32649 = blockIdx_x();
        p_32649 = _32649;
    l32647: ;
        _32649 = p_32649;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32652 = blockDim_x();
        p_32652 = _32652;
    l32650: ;
        _32652 = p_32652;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32655 = blockIdx_y();
        p_32655 = _32655;
    l32653: ;
        _32655 = p_32655;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32658 = blockDim_y();
        p_32658 = _32658;
    l32656: ;
        _32658 = p_32658;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32661 = blockDim_y();
        p_32661 = _32661;
    l32659: ;
        _32661 = p_32661;
        #line 179 "gpu_device_shm.impala"
        int _32700;
        _32700 = _32631 * _32634;
        #line 11 "main.impala"
        int _32697;
        _32697 = _27106_32615.e2;
        #line 179 "gpu_device_shm.impala"
        int gid_y_32701;
        gid_y_32701 = _32700 + _32637;
        #line 6 "gaussian.impala"
        int _32662;
        _32662 = _27110_32619.e2;
        #line 11 "main.impala"
        int _32705;
        _32705 = _27106_32615.e3;
        #line 6 "gaussian.impala"
        int v_anchor_32663;
        v_anchor_32663 = _32662 / 2;
        #line 177 "gpu_device_shm.impala"
        int _32691;
        _32691 = _32622 * _32625;
        #line 181 "gpu_device_shm.impala"
        int _32674;
        _32674 = _27110_32619.e1;
        #line 187 "gpu_device_shm.impala"
        int _32664;
        _32664 = 2 * v_anchor_32663;
        #line 177 "gpu_device_shm.impala"
        int gid_x_32692;
        gid_x_32692 = _32691 + _32628;
        #line 181 "gpu_device_shm.impala"
        int extend_width_32675;
        extend_width_32675 = _32674 / 2;
        #line 187 "gpu_device_shm.impala"
        int shm_dimy_32665;
        shm_dimy_32665 = _32643 + _32664;
        #line 185 "gpu_device_shm.impala"
        int _32676;
        _32676 = 2 * extend_width_32675;
        #line 38 "gpu_device_shm.impala"
        bool _32666;
        _32666 = 0 < shm_dimy_32665;
        #line 185 "gpu_device_shm.impala"
        int shm_dimx_32677;
        shm_dimx_32677 = _32640 + _32676;
        #line 38 "gpu_device_shm.impala"
        if (_32666) goto l32667; else goto l32856;
    l32856: ;
        #line 221 "gpu_device_shm.impala"
        goto l32789;
    l32667: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32670 = blockDim_x();
        p_32670 = _32670;
    l32668: ;
        _32670 = p_32670;
        #line 211 "gpu_device_shm.impala"
        int _32702;
        _32702 = gid_y_32701 - v_anchor_32663;
        #line 214 "gpu_device_shm.impala"
        bool _32703;
        _32703 = 0 <= _32702;
        #line 209 "gpu_device_shm.impala"
        int _32693;
        _32693 = gid_x_32692 - extend_width_32675;
        #line 214 "gpu_device_shm.impala"
        bool _32706;
        _32706 = _32702 < _32705;
        #line 217 "gpu_device_shm.impala"
        int _32711;
        _32711 = _32702 * _32697;
        #line 38 "gpu_device_shm.impala"
        p_32672 = 0;
        goto l32671;
    l32671: ;
        _32672 = p_32672;
        #line 38 "gpu_device_shm.impala"
        bool _32678;
        _32678 = _32672 < shm_dimx_32677;
        #line 38 "gpu_device_shm.impala"
        if (_32678) goto l32679; else goto l32729;
    l32729: ;
        #line 38 "gpu_device_shm.impala"
        p_32731 = _32661;
        goto l32730;
    l32730: ;
        _32731 = p_32731;
        #line 38 "gpu_device_shm.impala"
        bool _32733;
        _32733 = _32731 < shm_dimy_32665;
        #line 38 "gpu_device_shm.impala"
        if (_32733) goto l32734; else goto l32788;
    l32788: ;
        #line 221 "gpu_device_shm.impala"
        goto l32789;
    l32789: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        __syncthreads();
    l32791: ;
        #line 227 "gpu_device_shm.impala"
        bool _32793;
        _32793 = gid_x_32692 < _32697;
        #line 227 "gpu_device_shm.impala"
        if (_32793) goto l32794; else goto l32855;
    l32855: ;
        #line 230 "gpu_device_shm.impala"
        goto l32854;
    l32794: ;
        #line 227 "gpu_device_shm.impala"
        bool _32795;
        _32795 = gid_y_32701 < _32705;
        #line 227 "gpu_device_shm.impala"
        if (_32795) goto l32796; else goto l32853;
    l32853: ;
        #line 230 "gpu_device_shm.impala"
        goto l32854;
    l32854: ;
        return ;
    l32796: ;
        #line 64 "gpu_device_shm.impala"
        int _32836;
        _32836 = gid_y_32701 * _32697;
        #line 64 "gpu_device_shm.impala"
        char* _32834;
        _32834 = _27108_32617.e1;
        #line 39 "gaussian.impala"
        bool _32797;
        _32797 = v_anchor_32663 <= gid_y_32701;
        #line 193 "gpu_device_shm.impala"
        int _32823;
        _32823 = _32649 * _32652;
        #line 194 "gpu_device_shm.impala"
        int _32818;
        _32818 = _32655 * _32658;
        #line 64 "gpu_device_shm.impala"
        int _32837;
        _32837 = _32836 + gid_x_32692;
        #line 64 "gpu_device_shm.impala"
        double* _32835;
        union { double* dst; char* src; } u_32835;
        u_32835.src = _32834;
        _32835 = u_32835.dst;
        #line 193 "gpu_device_shm.impala"
        int _32824;
        _32824 = extend_width_32675 - _32823;
        #line 194 "gpu_device_shm.impala"
        int _32819;
        _32819 = v_anchor_32663 - _32818;
        #line 64 "gpu_device_shm.impala"
        double* _32838;
        _32838 = _32835 + _32837;
        #line 39 "gaussian.impala"
        if (_32797) goto l32798; else goto l32852;
    l32852: ;
        #line 49 "gaussian.impala"
        goto l32842;
    l32798: ;
        #line 39 "gaussian.impala"
        int _32799;
        _32799 = _32705 - v_anchor_32663;
        #line 39 "gaussian.impala"
        bool _32800;
        _32800 = gid_y_32701 < _32799;
        #line 39 "gaussian.impala"
        if (_32800) goto l32801; else goto l32841;
    l32841: ;
        #line 49 "gaussian.impala"
        goto l32842;
    l32842: ;
        #line 72 "gpu_device_shm.impala"
        int _32843;
        _32843 = gid_y_32701 + _32819;
        #line 72 "gpu_device_shm.impala"
        int _32844;
        _32844 = _32843 * shm_dimx_32677;
        #line 72 "gpu_device_shm.impala"
        int _32845;
        _32845 = _32844 + gid_x_32692;
        #line 72 "gpu_device_shm.impala"
        int _32846;
        _32846 = _32845 + _32824;
        #line 72 "gpu_device_shm.impala"
        double* _32847;
        _32847 = reserve_shared_32646 + _32846;
        #line 72 "gpu_device_shm.impala"
        double _32848;
        _32848 = *_32847;
        #line 72 "gpu_device_shm.impala"
        double _32850;
        _32850 = _32848;
        #line 64 "gpu_device_shm.impala"
        *_32838 = _32850;
        return ;
    l32801: ;
        #line 77 "gpu_device_shm.impala"
        struct_Buffer_6429 _32810;
        _32810 = _27110_32619.e0;
        #line 41 "gaussian.impala"
        int _32806;
        _32806 = 1 + v_anchor_32663;
        #line 41 "gaussian.impala"
        int _32840;
        _32840 = 0 - v_anchor_32663;
        #line 77 "gpu_device_shm.impala"
        char* _32811;
        _32811 = _32810.e1;
        #line 77 "gpu_device_shm.impala"
        double* _32812;
        union { double* dst; char* src; } u_32812;
        u_32812.src = _32811;
        _32812 = u_32812.dst;
        #line 27 "gpu_device_shm.impala"
        p_32803 = _32840;
        psum_32805 = 0.000000e+00;
        goto l32802;
    l32802: ;
        _32803 = p_32803;
        sum_32805 = psum_32805;
        #line 27 "gpu_device_shm.impala"
        bool _32807;
        _32807 = _32803 < _32806;
        #line 27 "gpu_device_shm.impala"
        if (_32807) goto l32808; else goto l32833;
    l32833: ;
        #line 64 "gpu_device_shm.impala"
        *_32838 = sum_32805;
        return ;
    l32808: ;
        #line 43 "gaussian.impala"
        int _32813;
        _32813 = _32803 + v_anchor_32663;
        #line 43 "gaussian.impala"
        int _32817;
        _32817 = gid_y_32701 + _32803;
        #line 31 "gpu_device_shm.impala"
        int _32809;
        _32809 = 1 + _32803;
        #line 76 "gpu_device_shm.impala"
        double* i_32814;
        i_32814 = _32812 + _32813;
        #line 72 "gpu_device_shm.impala"
        int _32820;
        _32820 = _32817 + _32819;
        #line 77 "gpu_device_shm.impala"
        double _32815;
        _32815 = *i_32814;
        #line 72 "gpu_device_shm.impala"
        int _32821;
        _32821 = _32820 * shm_dimx_32677;
        #line 77 "gpu_device_shm.impala"
        double _32829;
        _32829 = _32815;
        #line 72 "gpu_device_shm.impala"
        int _32822;
        _32822 = _32821 + gid_x_32692;
        #line 72 "gpu_device_shm.impala"
        int _32825;
        _32825 = _32822 + _32824;
        #line 72 "gpu_device_shm.impala"
        double* _32826;
        _32826 = reserve_shared_32646 + _32825;
        #line 72 "gpu_device_shm.impala"
        double _32827;
        _32827 = *_32826;
        #line 72 "gpu_device_shm.impala"
        double _32830;
        _32830 = _32827;
        #line 43 "gaussian.impala"
        double _32831;
        _32831 = _32829 * _32830;
        #line 43 "gaussian.impala"
        double _32832;
        _32832 = sum_32805 + _32831;
        #line 27 "gpu_device_shm.impala"
        p_32803 = _32809;
        psum_32805 = _32832;
        goto l32802;
    l32734: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32737 = blockDim_x();
        p_32737 = _32737;
    l32735: ;
        _32737 = p_32737;
        #line 211 "gpu_device_shm.impala"
        int img_index_y_32760;
        img_index_y_32760 = _32702 + _32731;
        #line 214 "gpu_device_shm.impala"
        bool _32761;
        _32761 = 0 <= img_index_y_32760;
        #line 217 "gpu_device_shm.impala"
        int _32768;
        _32768 = img_index_y_32760 * _32697;
        #line 214 "gpu_device_shm.impala"
        bool _32763;
        _32763 = img_index_y_32760 < _32705;
        #line 38 "gpu_device_shm.impala"
        p_32739 = 0;
        goto l32738;
    l32738: ;
        _32739 = p_32739;
        #line 38 "gpu_device_shm.impala"
        bool _32741;
        _32741 = _32739 < shm_dimx_32677;
        #line 38 "gpu_device_shm.impala"
        if (_32741) goto l32742; else goto l32786;
    l32786: ;
        #line 42 "gpu_device_shm.impala"
        int _32787;
        _32787 = _32731 + _32661;
        #line 38 "gpu_device_shm.impala"
        p_32731 = _32787;
        goto l32730;
    l32742: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32745 = threadIdx_x();
        p_32745 = _32745;
    l32743: ;
        _32745 = p_32745;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32748 = threadIdx_y();
        p_32748 = _32748;
    l32746: ;
        _32748 = p_32748;
        #line 202 "gpu_device_shm.impala"
        int shm_index_x_32749;
        shm_index_x_32749 = _32745 + _32739;
        #line 207 "gpu_device_shm.impala"
        bool _32750;
        _32750 = shm_index_x_32749 < shm_dimx_32677;
        #line 207 "gpu_device_shm.impala"
        if (_32750) goto l32751; else goto l32785;
    l32785: ;
        #line 219 "gpu_device_shm.impala"
        goto l32784;
    l32751: ;
        #line 204 "gpu_device_shm.impala"
        int shm_index_y_32752;
        shm_index_y_32752 = _32748 + _32731;
        #line 207 "gpu_device_shm.impala"
        bool _32753;
        _32753 = shm_index_y_32752 < shm_dimy_32665;
        #line 207 "gpu_device_shm.impala"
        if (_32753) goto l32754; else goto l32783;
    l32783: ;
        #line 219 "gpu_device_shm.impala"
        goto l32784;
    l32784: ;
        #line 40 "gpu_device_shm.impala"
        goto l32765;
    l32754: ;
        #line 209 "gpu_device_shm.impala"
        int img_index_x_32755;
        img_index_x_32755 = _32693 + _32739;
        #line 214 "gpu_device_shm.impala"
        bool _32756;
        _32756 = 0 <= img_index_x_32755;
        #line 214 "gpu_device_shm.impala"
        if (_32756) goto l32757; else goto l32782;
    l32782: ;
        #line 218 "gpu_device_shm.impala"
        goto l32779;
    l32757: ;
        #line 214 "gpu_device_shm.impala"
        bool _32758;
        _32758 = img_index_x_32755 < _32697;
        #line 214 "gpu_device_shm.impala"
        if (_32758) goto l32759; else goto l32781;
    l32781: ;
        #line 218 "gpu_device_shm.impala"
        goto l32779;
    l32759: ;
        #line 214 "gpu_device_shm.impala"
        if (_32761) goto l32762; else goto l32780;
    l32780: ;
        #line 218 "gpu_device_shm.impala"
        goto l32779;
    l32762: ;
        #line 214 "gpu_device_shm.impala"
        if (_32763) goto l32764; else goto l32778;
    l32778: ;
        #line 218 "gpu_device_shm.impala"
        goto l32779;
    l32779: ;
        #line 40 "gpu_device_shm.impala"
        goto l32765;
    l32764: ;
        #line 216 "gpu_device_shm.impala"
        int _32773;
        _32773 = shm_index_y_32752 * shm_dimx_32677;
        #line 217 "gpu_device_shm.impala"
        int _32769;
        _32769 = _32768 + img_index_x_32755;
        #line 216 "gpu_device_shm.impala"
        int _32774;
        _32774 = _32773 + shm_index_x_32749;
        #line 217 "gpu_device_shm.impala"
        double* _32770;
        _32770 = _27109_32618 + _32769;
        #line 216 "gpu_device_shm.impala"
        double* _32775;
        _32775 = reserve_shared_32646 + _32774;
        #line 217 "gpu_device_shm.impala"
        double _32771;
        _32771 = *_32770;
        #line 217 "gpu_device_shm.impala"
        double _32776;
        _32776 = _32771;
        #line 216 "gpu_device_shm.impala"
        *_32775 = _32776;
        #line 40 "gpu_device_shm.impala"
        goto l32765;
    l32765: ;
        #line 42 "gpu_device_shm.impala"
        int _32767;
        _32767 = _32739 + _32737;
        #line 38 "gpu_device_shm.impala"
        p_32739 = _32767;
        goto l32738;
    l32679: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32682 = threadIdx_x();
        p_32682 = _32682;
    l32680: ;
        _32682 = p_32682;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _32685 = threadIdx_y();
        p_32685 = _32685;
    l32683: ;
        _32685 = p_32685;
        #line 202 "gpu_device_shm.impala"
        int shm_index_x_32686;
        shm_index_x_32686 = _32682 + _32672;
        #line 207 "gpu_device_shm.impala"
        bool _32687;
        _32687 = shm_index_x_32686 < shm_dimx_32677;
        #line 207 "gpu_device_shm.impala"
        if (_32687) goto l32688; else goto l32728;
    l32728: ;
        #line 219 "gpu_device_shm.impala"
        goto l32727;
    l32688: ;
        #line 207 "gpu_device_shm.impala"
        bool _32689;
        _32689 = _32685 < shm_dimy_32665;
        #line 207 "gpu_device_shm.impala"
        if (_32689) goto l32690; else goto l32726;
    l32726: ;
        #line 219 "gpu_device_shm.impala"
        goto l32727;
    l32727: ;
        #line 40 "gpu_device_shm.impala"
        goto l32708;
    l32690: ;
        #line 209 "gpu_device_shm.impala"
        int img_index_x_32694;
        img_index_x_32694 = _32693 + _32672;
        #line 214 "gpu_device_shm.impala"
        bool _32695;
        _32695 = 0 <= img_index_x_32694;
        #line 214 "gpu_device_shm.impala"
        if (_32695) goto l32696; else goto l32725;
    l32725: ;
        #line 218 "gpu_device_shm.impala"
        goto l32722;
    l32696: ;
        #line 214 "gpu_device_shm.impala"
        bool _32698;
        _32698 = img_index_x_32694 < _32697;
        #line 214 "gpu_device_shm.impala"
        if (_32698) goto l32699; else goto l32724;
    l32724: ;
        #line 218 "gpu_device_shm.impala"
        goto l32722;
    l32699: ;
        #line 214 "gpu_device_shm.impala"
        if (_32703) goto l32704; else goto l32723;
    l32723: ;
        #line 218 "gpu_device_shm.impala"
        goto l32722;
    l32704: ;
        #line 214 "gpu_device_shm.impala"
        if (_32706) goto l32707; else goto l32721;
    l32721: ;
        #line 218 "gpu_device_shm.impala"
        goto l32722;
    l32722: ;
        #line 40 "gpu_device_shm.impala"
        goto l32708;
    l32707: ;
        #line 217 "gpu_device_shm.impala"
        int _32712;
        _32712 = _32711 + img_index_x_32694;
        #line 216 "gpu_device_shm.impala"
        int _32716;
        _32716 = _32685 * shm_dimx_32677;
        #line 217 "gpu_device_shm.impala"
        double* _32713;
        _32713 = _27109_32618 + _32712;
        #line 216 "gpu_device_shm.impala"
        int _32717;
        _32717 = _32716 + shm_index_x_32686;
        #line 217 "gpu_device_shm.impala"
        double _32714;
        _32714 = *_32713;
        #line 216 "gpu_device_shm.impala"
        double* _32718;
        _32718 = reserve_shared_32646 + _32717;
        #line 217 "gpu_device_shm.impala"
        double _32719;
        _32719 = _32714;
        #line 216 "gpu_device_shm.impala"
        *_32718 = _32719;
        #line 40 "gpu_device_shm.impala"
        goto l32708;
    l32708: ;
        #line 42 "gpu_device_shm.impala"
        int _32710;
        _32710 = _32672 + _32670;
        #line 38 "gpu_device_shm.impala"
        p_32672 = _32710;
        goto l32671;
}

}