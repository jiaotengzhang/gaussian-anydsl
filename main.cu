
#include <hip/hip_runtime.h>
extern "C" {
typedef struct {
    int e0;
    char* e1;
} struct_Buffer_5292;
typedef struct {
    struct_Buffer_5292 e0;
    struct_Buffer_5292 e1;
    int e2;
    int e3;
} struct_image_5291;
typedef struct {
    struct_Buffer_5292 e0;
    int e1;
} struct_filter_5295;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_19779(double*, struct_image_5291, struct_filter_5295, struct_Buffer_5292);
__global__ void lambda_19916(double*, struct_image_5291, struct_Buffer_5292, struct_filter_5295);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_19779(double* _19782_22128, struct_image_5291 _19783_22129, struct_filter_5295 _19784_22130, struct_Buffer_5292 _19785_22131) {
    int  _22137;
    int p_22137;
    int  _22143;
    int p_22143;
    int  _22149;
    int p_22149;
    int  _22155;
    int p_22155;
    int  _22161;
    int p_22161;
    int  _22167;
    int p_22167;
    int  _22190;
    int p_22190;
    double  sum_22192;
    double psum_22192;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _22137 = blockIdx_x();
    p_22137 = _22137;
    l22135: ;
        _22137 = p_22137;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22143 = blockDim_x();
        p_22143 = _22143;
    l22141: ;
        _22143 = p_22143;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22149 = threadIdx_x();
        p_22149 = _22149;
    l22147: ;
        _22149 = p_22149;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22155 = blockIdx_y();
        p_22155 = _22155;
    l22153: ;
        _22155 = p_22155;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22161 = blockDim_y();
        p_22161 = _22161;
    l22159: ;
        _22161 = p_22161;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22167 = threadIdx_y();
        p_22167 = _22167;
    l22165: ;
        _22167 = p_22167;
        #line 66 "gpu_device.impala"
        int _22168;
        _22168 = _22137 * _22143;
        #line 66 "gpu_device.impala"
        int gid_x_22169;
        gid_x_22169 = _22168 + _22149;
        #line 6 "main.impala"
        int _22171;
        _22171 = _19783_22129.e2;
        #line 69 "gpu_device.impala"
        bool _22172;
        _22172 = gid_x_22169 < _22171;
        #line 69 "gpu_device.impala"
        if (_22172) goto l22173; else goto l22238;
    l22238: ;
        #line 71 "gpu_device.impala"
        goto l22237;
    l22173: ;
        #line 6 "main.impala"
        int _22177;
        _22177 = _19783_22129.e3;
        #line 67 "gpu_device.impala"
        int _22174;
        _22174 = _22155 * _22161;
        #line 67 "gpu_device.impala"
        int gid_y_22175;
        gid_y_22175 = _22174 + _22167;
        #line 69 "gpu_device.impala"
        bool _22178;
        _22178 = gid_y_22175 < _22177;
        #line 69 "gpu_device.impala"
        if (_22178) goto l22179; else goto l22236;
    l22236: ;
        #line 71 "gpu_device.impala"
        goto l22237;
    l22237: ;
        return ;
    l22179: ;
        #line 3 "gaussian.impala"
        int _22181;
        _22181 = _19784_22130.e1;
        #line 26 "gpu_device.impala"
        char* _22220;
        _22220 = _19785_22131.e1;
        #line 31 "gpu_device.impala"
        struct_Buffer_5292 _22206;
        _22206 = _19783_22129.e1;
        #line 31 "gpu_device.impala"
        char* _22207;
        _22207 = _22206.e1;
        #line 31 "gpu_device.impala"
        int _22209;
        _22209 = gid_y_22175 * _22171;
        #line 3 "gaussian.impala"
        int anchor_22183;
        anchor_22183 = _22181 / 2;
        #line 26 "gpu_device.impala"
        double* _22221;
        union { double* dst; char* src; } u_22221;
        u_22221.src = _22220;
        _22221 = u_22221.dst;
        #line 31 "gpu_device.impala"
        double* _22208;
        union { double* dst; char* src; } u_22208;
        u_22208.src = _22207;
        _22208 = u_22208.dst;
        #line 31 "gpu_device.impala"
        int _22222;
        _22222 = _22209 + gid_x_22169;
        #line 10 "gaussian.impala"
        bool _22184;
        _22184 = anchor_22183 <= gid_x_22169;
        #line 26 "gpu_device.impala"
        double* _22223;
        _22223 = _22221 + _22222;
        #line 10 "gaussian.impala"
        if (_22184) goto l22185; else goto l22235;
    l22235: ;
        #line 16 "gaussian.impala"
        goto l22229;
    l22185: ;
        #line 10 "gaussian.impala"
        int _22186;
        _22186 = _22171 - anchor_22183;
        #line 10 "gaussian.impala"
        bool _22187;
        _22187 = gid_x_22169 < _22186;
        #line 10 "gaussian.impala"
        if (_22187) goto l22188; else goto l22228;
    l22228: ;
        #line 16 "gaussian.impala"
        goto l22229;
    l22229: ;
        #line 31 "gpu_device.impala"
        double* _22230;
        _22230 = _22208 + _22222;
        #line 31 "gpu_device.impala"
        double _22231;
        _22231 = *_22230;
        #line 31 "gpu_device.impala"
        double _22233;
        _22233 = _22231;
        #line 26 "gpu_device.impala"
        *_22223 = _22233;
        return ;
    l22188: ;
        #line 11 "gaussian.impala"
        int _22194;
        _22194 = 1 + anchor_22183;
        #line 36 "gpu_device.impala"
        struct_Buffer_5292 _22199;
        _22199 = _19784_22130.e0;
        #line 11 "gaussian.impala"
        int _22226;
        _22226 = 0 - anchor_22183;
        #line 36 "gpu_device.impala"
        char* _22200;
        _22200 = _22199.e1;
        #line 36 "gpu_device.impala"
        double* _22201;
        union { double* dst; char* src; } u_22201;
        u_22201.src = _22200;
        _22201 = u_22201.dst;
        #line 14 "gpu_device.impala"
        p_22190 = _22226;
        psum_22192 = 0.000000e+00;
        goto l22189;
    l22189: ;
        _22190 = p_22190;
        sum_22192 = psum_22192;
        #line 14 "gpu_device.impala"
        bool _22195;
        _22195 = _22190 < _22194;
        #line 14 "gpu_device.impala"
        if (_22195) goto l22196; else goto l22219;
    l22219: ;
        #line 26 "gpu_device.impala"
        *_22223 = sum_22192;
        return ;
    l22196: ;
        #line 12 "gaussian.impala"
        int _22202;
        _22202 = _22190 + anchor_22183;
        #line 16 "gpu_device.impala"
        int _22197;
        _22197 = 1 + _22190;
        #line 35 "gpu_device.impala"
        double* i_22203;
        i_22203 = _22201 + _22202;
        #line 12 "gaussian.impala"
        int _22210;
        _22210 = gid_x_22169 + _22190;
        #line 36 "gpu_device.impala"
        double _22204;
        _22204 = *i_22203;
        #line 31 "gpu_device.impala"
        int _22211;
        _22211 = _22209 + _22210;
        #line 36 "gpu_device.impala"
        double _22215;
        _22215 = _22204;
        #line 31 "gpu_device.impala"
        double* _22212;
        _22212 = _22208 + _22211;
        #line 31 "gpu_device.impala"
        double _22213;
        _22213 = *_22212;
        #line 31 "gpu_device.impala"
        double _22216;
        _22216 = _22213;
        #line 12 "gaussian.impala"
        double _22217;
        _22217 = _22215 * _22216;
        #line 12 "gaussian.impala"
        double _22218;
        _22218 = sum_22192 + _22217;
        #line 14 "gpu_device.impala"
        p_22190 = _22197;
        psum_22192 = _22218;
        goto l22189;
}

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_19916(double* _19919_22242, struct_image_5291 _19920_22243, struct_Buffer_5292 _19921_22244, struct_filter_5295 _19922_22245) {
    int  _22248;
    int p_22248;
    int  _22251;
    int p_22251;
    int  _22254;
    int p_22254;
    int  _22257;
    int p_22257;
    int  _22260;
    int p_22260;
    int  _22263;
    int p_22263;
    int  _22282;
    int p_22282;
    double  sum_22284;
    double psum_22284;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _22248 = blockIdx_x();
    p_22248 = _22248;
    l22246: ;
        _22248 = p_22248;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22251 = blockDim_x();
        p_22251 = _22251;
    l22249: ;
        _22251 = p_22251;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22254 = threadIdx_x();
        p_22254 = _22254;
    l22252: ;
        _22254 = p_22254;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22257 = blockIdx_y();
        p_22257 = _22257;
    l22255: ;
        _22257 = p_22257;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22260 = blockDim_y();
        p_22260 = _22260;
    l22258: ;
        _22260 = p_22260;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22263 = threadIdx_y();
        p_22263 = _22263;
    l22261: ;
        _22263 = p_22263;
        #line 66 "gpu_device.impala"
        int _22264;
        _22264 = _22248 * _22251;
        #line 66 "gpu_device.impala"
        int gid_x_22265;
        gid_x_22265 = _22264 + _22254;
        #line 6 "main.impala"
        int _22266;
        _22266 = _19920_22243.e2;
        #line 69 "gpu_device.impala"
        bool _22267;
        _22267 = gid_x_22265 < _22266;
        #line 69 "gpu_device.impala"
        if (_22267) goto l22268; else goto l22327;
    l22327: ;
        #line 71 "gpu_device.impala"
        goto l22326;
    l22268: ;
        #line 6 "main.impala"
        int _22271;
        _22271 = _19920_22243.e3;
        #line 67 "gpu_device.impala"
        int _22269;
        _22269 = _22257 * _22260;
        #line 67 "gpu_device.impala"
        int gid_y_22270;
        gid_y_22270 = _22269 + _22263;
        #line 69 "gpu_device.impala"
        bool _22272;
        _22272 = gid_y_22270 < _22271;
        #line 69 "gpu_device.impala"
        if (_22272) goto l22273; else goto l22325;
    l22325: ;
        #line 71 "gpu_device.impala"
        goto l22326;
    l22326: ;
        return ;
    l22273: ;
        #line 26 "gpu_device.impala"
        char* _22310;
        _22310 = _19921_22244.e1;
        #line 24 "gaussian.impala"
        int _22274;
        _22274 = _19922_22245.e1;
        #line 31 "gpu_device.impala"
        struct_Buffer_5292 _22296;
        _22296 = _19920_22243.e1;
        #line 31 "gpu_device.impala"
        int _22312;
        _22312 = gid_y_22270 * _22266;
        #line 26 "gpu_device.impala"
        double* _22311;
        union { double* dst; char* src; } u_22311;
        u_22311.src = _22310;
        _22311 = u_22311.dst;
        #line 24 "gaussian.impala"
        int anchor_22275;
        anchor_22275 = _22274 / 2;
        #line 31 "gpu_device.impala"
        char* _22297;
        _22297 = _22296.e1;
        #line 31 "gpu_device.impala"
        int _22313;
        _22313 = _22312 + gid_x_22265;
        #line 26 "gpu_device.impala"
        double* _22314;
        _22314 = _22311 + _22313;
        #line 31 "gaussian.impala"
        bool _22276;
        _22276 = anchor_22275 <= gid_y_22270;
        #line 31 "gpu_device.impala"
        double* _22298;
        union { double* dst; char* src; } u_22298;
        u_22298.src = _22297;
        _22298 = u_22298.dst;
        #line 31 "gaussian.impala"
        if (_22276) goto l22277; else goto l22324;
    l22324: ;
        #line 37 "gaussian.impala"
        goto l22318;
    l22277: ;
        #line 31 "gaussian.impala"
        int _22278;
        _22278 = _22271 - anchor_22275;
        #line 31 "gaussian.impala"
        bool _22279;
        _22279 = gid_y_22270 < _22278;
        #line 31 "gaussian.impala"
        if (_22279) goto l22280; else goto l22317;
    l22317: ;
        #line 37 "gaussian.impala"
        goto l22318;
    l22318: ;
        #line 31 "gpu_device.impala"
        double* _22319;
        _22319 = _22298 + _22313;
        #line 31 "gpu_device.impala"
        double _22320;
        _22320 = *_22319;
        #line 31 "gpu_device.impala"
        double _22322;
        _22322 = _22320;
        #line 26 "gpu_device.impala"
        *_22314 = _22322;
        return ;
    l22280: ;
        #line 36 "gpu_device.impala"
        struct_Buffer_5292 _22289;
        _22289 = _19922_22245.e0;
        #line 36 "gpu_device.impala"
        char* _22290;
        _22290 = _22289.e1;
        #line 36 "gpu_device.impala"
        double* _22291;
        union { double* dst; char* src; } u_22291;
        u_22291.src = _22290;
        _22291 = u_22291.dst;
        #line 32 "gaussian.impala"
        int _22285;
        _22285 = 1 + anchor_22275;
        #line 32 "gaussian.impala"
        int _22316;
        _22316 = 0 - anchor_22275;
        #line 14 "gpu_device.impala"
        p_22282 = _22316;
        psum_22284 = 0.000000e+00;
        goto l22281;
    l22281: ;
        _22282 = p_22282;
        sum_22284 = psum_22284;
        #line 14 "gpu_device.impala"
        bool _22286;
        _22286 = _22282 < _22285;
        #line 14 "gpu_device.impala"
        if (_22286) goto l22287; else goto l22309;
    l22309: ;
        #line 26 "gpu_device.impala"
        *_22314 = sum_22284;
        return ;
    l22287: ;
        #line 16 "gpu_device.impala"
        int _22288;
        _22288 = 1 + _22282;
        #line 33 "gaussian.impala"
        int _22292;
        _22292 = _22282 + anchor_22275;
        #line 33 "gaussian.impala"
        int _22299;
        _22299 = gid_y_22270 + _22282;
        #line 35 "gpu_device.impala"
        double* i_22293;
        i_22293 = _22291 + _22292;
        #line 31 "gpu_device.impala"
        int _22300;
        _22300 = _22299 * _22266;
        #line 36 "gpu_device.impala"
        double _22294;
        _22294 = *i_22293;
        #line 31 "gpu_device.impala"
        int _22301;
        _22301 = _22300 + gid_x_22265;
        #line 36 "gpu_device.impala"
        double _22305;
        _22305 = _22294;
        #line 31 "gpu_device.impala"
        double* _22302;
        _22302 = _22298 + _22301;
        #line 31 "gpu_device.impala"
        double _22303;
        _22303 = *_22302;
        #line 31 "gpu_device.impala"
        double _22306;
        _22306 = _22303;
        #line 33 "gaussian.impala"
        double _22307;
        _22307 = _22305 * _22306;
        #line 33 "gaussian.impala"
        double _22308;
        _22308 = sum_22284 + _22307;
        #line 14 "gpu_device.impala"
        p_22282 = _22288;
        psum_22284 = _22308;
        goto l22281;
}

}