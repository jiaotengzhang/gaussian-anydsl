
#include <hip/hip_runtime.h>
extern "C" {
typedef struct {
    int e0;
    char* e1;
} struct_Buffer_6445;
typedef struct {
    struct_Buffer_6445 e0;
    struct_Buffer_6445 e1;
    struct_Buffer_6445 e2;
    int e3;
    int e4;
} struct_image_6444;
typedef struct {
    struct_Buffer_6445 e0;
    int e1;
    int e2;
} struct_filter_6450;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_26231(struct_image_6444, double*, struct_filter_6450, struct_Buffer_6445);
__global__ void lambda_26506(struct_image_6444, double*, struct_filter_6450, struct_Buffer_6445, double*, struct_Buffer_6445);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_26231(struct_image_6444 _26234_31341, double* _26235_31342, struct_filter_6450 _26236_31343, struct_Buffer_6445 _26237_31344) {
    int  _31350;
    int p_31350;
    int  _31356;
    int p_31356;
    int  _31362;
    int p_31362;
    int  _31368;
    int p_31368;
    int  _31374;
    int p_31374;
    int  _31380;
    int p_31380;
    double*  shared_31388;
    double* pshared_31388;
    int  _31391;
    int p_31391;
    int  _31394;
    int p_31394;
    int  _31405;
    int p_31405;
    int  _31408;
    int p_31408;
    int  _31410;
    int p_31410;
    int  _31469;
    int p_31469;
    int  _31546;
    int p_31546;
    double  sum_31548;
    double psum_31548;
    int  _31475;
    int p_31475;
    int  _31478;
    int p_31478;
    int  _31480;
    int p_31480;
    int  _31497;
    int p_31497;
    int  _31500;
    int p_31500;
    int  _31503;
    int p_31503;
    int  _31439;
    int p_31439;
    int  _31442;
    int p_31442;
    int  _31445;
    int p_31445;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _31350 = blockIdx_x();
    p_31350 = _31350;
    l31348: ;
        _31350 = p_31350;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31356 = blockDim_x();
        p_31356 = _31356;
    l31354: ;
        _31356 = p_31356;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31362 = threadIdx_x();
        p_31362 = _31362;
    l31360: ;
        _31362 = p_31362;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31368 = blockIdx_y();
        p_31368 = _31368;
    l31366: ;
        _31368 = p_31368;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31374 = blockDim_y();
        p_31374 = _31374;
    l31372: ;
        _31374 = p_31374;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31380 = threadIdx_y();
        p_31380 = _31380;
    l31378: ;
        _31380 = p_31380;
        #line 160 "gpu_device_shm.impala"
        __shared__ double reserver_shared_31388[2130];
        pshared_31388 = reserver_shared_31388;
    l31386: ;
        shared_31388 = pshared_31388;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31391 = blockDim_y();
        p_31391 = _31391;
    l31389: ;
        _31391 = p_31391;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31394 = blockDim_y();
        p_31394 = _31394;
    l31392: ;
        _31394 = p_31394;
        #line 158 "gpu_device_shm.impala"
        int _31428;
        _31428 = _31368 * _31374;
        #line 163 "gpu_device_shm.impala"
        int _31398;
        _31398 = _26236_31343.e2;
        #line 11 "main.impala"
        int _31425;
        _31425 = _26234_31341.e3;
        #line 158 "gpu_device_shm.impala"
        int gid_y_31429;
        gid_y_31429 = _31428 + _31380;
        #line 156 "gpu_device_shm.impala"
        int _31418;
        _31418 = _31350 * _31356;
        #line 156 "gpu_device_shm.impala"
        int gid_x_31419;
        gid_x_31419 = _31418 + _31362;
        #line 11 "main.impala"
        int _31434;
        _31434 = _26234_31341.e4;
        #line 4 "gaussian.impala"
        int _31413;
        _31413 = _26236_31343.e1;
        #line 163 "gpu_device_shm.impala"
        int _31399;
        _31399 = 2 * _31398;
        #line 163 "gpu_device_shm.impala"
        int _31400;
        _31400 = _31391 + _31399;
        #line 31 "gpu_device_shm.impala"
        bool _31401;
        _31401 = 0 < _31400;
        #line 31 "gpu_device_shm.impala"
        if (_31401) goto l31402; else goto l31593;
    l31593: ;
        #line 172 "gpu_device_shm.impala"
        goto l31528;
    l31402: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31405 = blockDim_x();
        p_31405 = _31405;
    l31403: ;
        _31405 = p_31405;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31408 = blockDim_x();
        p_31408 = _31408;
    l31406: ;
        _31408 = p_31408;
        #line 165 "gpu_device_shm.impala"
        int _31420;
        _31420 = gid_x_31419 - _31413;
        #line 166 "gpu_device_shm.impala"
        int _31430;
        _31430 = gid_y_31429 - _31398;
        #line 164 "gpu_device_shm.impala"
        int _31414;
        _31414 = 2 * _31413;
        #line 168 "gpu_device_shm.impala"
        bool _31431;
        _31431 = 0 <= _31430;
        #line 168 "gpu_device_shm.impala"
        bool _31435;
        _31435 = _31430 < _31434;
        #line 164 "gpu_device_shm.impala"
        int _31415;
        _31415 = _31405 + _31414;
        #line 169 "gpu_device_shm.impala"
        int _31449;
        _31449 = _31430 * _31425;
        #line 31 "gpu_device_shm.impala"
        p_31410 = 0;
        goto l31409;
    l31409: ;
        _31410 = p_31410;
        #line 31 "gpu_device_shm.impala"
        bool _31416;
        _31416 = _31410 < _31415;
        #line 31 "gpu_device_shm.impala"
        if (_31416) goto l31417; else goto l31467;
    l31467: ;
        #line 31 "gpu_device_shm.impala"
        p_31469 = _31394;
        goto l31468;
    l31468: ;
        _31469 = p_31469;
        #line 31 "gpu_device_shm.impala"
        bool _31471;
        _31471 = _31469 < _31400;
        #line 31 "gpu_device_shm.impala"
        if (_31471) goto l31472; else goto l31527;
    l31527: ;
        #line 172 "gpu_device_shm.impala"
        goto l31528;
    l31528: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        __syncthreads();
    l31533: ;
        #line 178 "gpu_device_shm.impala"
        bool _31535;
        _31535 = gid_x_31419 < _31425;
        #line 178 "gpu_device_shm.impala"
        if (_31535) goto l31536; else goto l31592;
    l31592: ;
        #line 181 "gpu_device_shm.impala"
        goto l31591;
    l31536: ;
        #line 178 "gpu_device_shm.impala"
        bool _31537;
        _31537 = gid_y_31429 < _31434;
        #line 178 "gpu_device_shm.impala"
        if (_31537) goto l31538; else goto l31590;
    l31590: ;
        #line 181 "gpu_device_shm.impala"
        goto l31591;
    l31591: ;
        return ;
    l31538: ;
        #line 4 "gaussian.impala"
        int h_anchor_31539;
        h_anchor_31539 = _31413 / 2;
        #line 62 "gpu_device_shm.impala"
        struct_Buffer_6445 _31561;
        _31561 = _26234_31341.e1;
        #line 47 "gpu_device_shm.impala"
        char* _31575;
        _31575 = _26237_31344.e1;
        #line 25 "gaussian.impala"
        int _31564;
        _31564 = gid_y_31429 * _31425;
        #line 17 "gaussian.impala"
        bool _31540;
        _31540 = h_anchor_31539 <= gid_x_31419;
        #line 47 "gpu_device_shm.impala"
        double* _31576;
        union { double* dst; char* src; } u_31576;
        u_31576.src = _31575;
        _31576 = u_31576.dst;
        #line 62 "gpu_device_shm.impala"
        char* _31562;
        _31562 = _31561.e1;
        #line 25 "gaussian.impala"
        int _31577;
        _31577 = _31564 + gid_x_31419;
        #line 46 "gpu_device_shm.impala"
        double* index_31578;
        index_31578 = _31576 + _31577;
        #line 62 "gpu_device_shm.impala"
        double* _31563;
        union { double* dst; char* src; } u_31563;
        u_31563.src = _31562;
        _31563 = u_31563.dst;
        #line 17 "gaussian.impala"
        if (_31540) goto l31541; else goto l31589;
    l31589: ;
        #line 27 "gaussian.impala"
        goto l31583;
    l31541: ;
        #line 17 "gaussian.impala"
        int _31542;
        _31542 = _31425 - h_anchor_31539;
        #line 17 "gaussian.impala"
        bool _31543;
        _31543 = gid_x_31419 < _31542;
        #line 17 "gaussian.impala"
        if (_31543) goto l31544; else goto l31582;
    l31582: ;
        #line 27 "gaussian.impala"
        goto l31583;
    l31583: ;
        #line 62 "gpu_device_shm.impala"
        double* _31584;
        _31584 = _31563 + _31577;
        #line 62 "gpu_device_shm.impala"
        double _31585;
        _31585 = *_31584;
        #line 62 "gpu_device_shm.impala"
        double _31587;
        _31587 = _31585;
        #line 47 "gpu_device_shm.impala"
        *index_31578 = _31587;
        return ;
    l31544: ;
        #line 19 "gaussian.impala"
        int _31580;
        _31580 = 0 - h_anchor_31539;
        #line 67 "gpu_device_shm.impala"
        struct_Buffer_6445 _31554;
        _31554 = _26236_31343.e0;
        #line 67 "gpu_device_shm.impala"
        char* _31555;
        _31555 = _31554.e1;
        #line 19 "gaussian.impala"
        int _31550;
        _31550 = 1 + h_anchor_31539;
        #line 67 "gpu_device_shm.impala"
        double* _31556;
        union { double* dst; char* src; } u_31556;
        u_31556.src = _31555;
        _31556 = u_31556.dst;
        #line 20 "gpu_device_shm.impala"
        p_31546 = _31580;
        psum_31548 = 0.000000e+00;
        goto l31545;
    l31545: ;
        _31546 = p_31546;
        sum_31548 = psum_31548;
        #line 20 "gpu_device_shm.impala"
        bool _31551;
        _31551 = _31546 < _31550;
        #line 20 "gpu_device_shm.impala"
        if (_31551) goto l31552; else goto l31574;
    l31574: ;
        #line 47 "gpu_device_shm.impala"
        *index_31578 = sum_31548;
        return ;
    l31552: ;
        #line 21 "gaussian.impala"
        int _31557;
        _31557 = _31546 + h_anchor_31539;
        #line 24 "gpu_device_shm.impala"
        int _31553;
        _31553 = 1 + _31546;
        #line 66 "gpu_device_shm.impala"
        double* i_31558;
        i_31558 = _31556 + _31557;
        #line 21 "gaussian.impala"
        int _31565;
        _31565 = gid_x_31419 + _31546;
        #line 67 "gpu_device_shm.impala"
        double _31559;
        _31559 = *i_31558;
        #line 62 "gpu_device_shm.impala"
        int _31566;
        _31566 = _31564 + _31565;
        #line 67 "gpu_device_shm.impala"
        double _31570;
        _31570 = _31559;
        #line 62 "gpu_device_shm.impala"
        double* _31567;
        _31567 = _31563 + _31566;
        #line 62 "gpu_device_shm.impala"
        double _31568;
        _31568 = *_31567;
        #line 62 "gpu_device_shm.impala"
        double _31571;
        _31571 = _31568;
        #line 21 "gaussian.impala"
        double _31572;
        _31572 = _31570 * _31571;
        #line 21 "gaussian.impala"
        double _31573;
        _31573 = sum_31548 + _31572;
        #line 20 "gpu_device_shm.impala"
        p_31546 = _31553;
        psum_31548 = _31573;
        goto l31545;
    l31472: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31475 = blockDim_x();
        p_31475 = _31475;
    l31473: ;
        _31475 = p_31475;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31478 = blockDim_x();
        p_31478 = _31478;
    l31476: ;
        _31478 = p_31478;
        #line 166 "gpu_device_shm.impala"
        int img_index_y_31490;
        img_index_y_31490 = _31430 + _31469;
        #line 164 "gpu_device_shm.impala"
        int _31482;
        _31482 = _31475 + _31414;
        #line 169 "gpu_device_shm.impala"
        int _31507;
        _31507 = img_index_y_31490 * _31425;
        #line 168 "gpu_device_shm.impala"
        bool _31491;
        _31491 = 0 <= img_index_y_31490;
        #line 168 "gpu_device_shm.impala"
        bool _31493;
        _31493 = img_index_y_31490 < _31434;
        #line 31 "gpu_device_shm.impala"
        p_31480 = 0;
        goto l31479;
    l31479: ;
        _31480 = p_31480;
        #line 31 "gpu_device_shm.impala"
        bool _31483;
        _31483 = _31480 < _31482;
        #line 31 "gpu_device_shm.impala"
        if (_31483) goto l31484; else goto l31525;
    l31525: ;
        #line 35 "gpu_device_shm.impala"
        int _31526;
        _31526 = _31469 + _31394;
        #line 31 "gpu_device_shm.impala"
        p_31469 = _31526;
        goto l31468;
    l31484: ;
        #line 165 "gpu_device_shm.impala"
        int img_index_x_31485;
        img_index_x_31485 = _31420 + _31480;
        #line 168 "gpu_device_shm.impala"
        bool _31486;
        _31486 = 0 <= img_index_x_31485;
        #line 168 "gpu_device_shm.impala"
        if (_31486) goto l31487; else goto l31524;
    l31524: ;
        #line 170 "gpu_device_shm.impala"
        goto l31521;
    l31487: ;
        #line 168 "gpu_device_shm.impala"
        bool _31488;
        _31488 = img_index_x_31485 < _31425;
        #line 168 "gpu_device_shm.impala"
        if (_31488) goto l31489; else goto l31523;
    l31523: ;
        #line 170 "gpu_device_shm.impala"
        goto l31521;
    l31489: ;
        #line 168 "gpu_device_shm.impala"
        if (_31491) goto l31492; else goto l31522;
    l31522: ;
        #line 170 "gpu_device_shm.impala"
        goto l31521;
    l31492: ;
        #line 168 "gpu_device_shm.impala"
        if (_31493) goto l31494; else goto l31520;
    l31520: ;
        #line 170 "gpu_device_shm.impala"
        goto l31521;
    l31521: ;
        #line 33 "gpu_device_shm.impala"
        goto l31504;
    l31494: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31497 = threadIdx_y();
        p_31497 = _31497;
    l31495: ;
        _31497 = p_31497;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31500 = blockDim_y();
        p_31500 = _31500;
    l31498: ;
        _31500 = p_31500;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31503 = threadIdx_x();
        p_31503 = _31503;
    l31501: ;
        _31503 = p_31503;
        #line 169 "gpu_device_shm.impala"
        int _31513;
        _31513 = _31500 + _31399;
        #line 169 "gpu_device_shm.impala"
        int _31508;
        _31508 = _31507 + img_index_x_31485;
        #line 169 "gpu_device_shm.impala"
        int _31512;
        _31512 = _31497 + _31469;
        #line 169 "gpu_device_shm.impala"
        int _31514;
        _31514 = _31512 * _31513;
        #line 169 "gpu_device_shm.impala"
        double* _31509;
        _31509 = _26235_31342 + _31508;
        #line 169 "gpu_device_shm.impala"
        int _31515;
        _31515 = _31514 + _31503;
        #line 169 "gpu_device_shm.impala"
        double _31510;
        _31510 = *_31509;
        #line 169 "gpu_device_shm.impala"
        int _31516;
        _31516 = _31515 + _31480;
        #line 169 "gpu_device_shm.impala"
        double _31518;
        _31518 = _31510;
        #line 169 "gpu_device_shm.impala"
        double* _31517;
        _31517 = shared_31388 + _31516;
        #line 169 "gpu_device_shm.impala"
        *_31517 = _31518;
        #line 33 "gpu_device_shm.impala"
        goto l31504;
    l31504: ;
        #line 35 "gpu_device_shm.impala"
        int _31506;
        _31506 = _31480 + _31478;
        #line 31 "gpu_device_shm.impala"
        p_31480 = _31506;
        goto l31479;
    l31417: ;
        #line 165 "gpu_device_shm.impala"
        int img_index_x_31421;
        img_index_x_31421 = _31420 + _31410;
        #line 168 "gpu_device_shm.impala"
        bool _31422;
        _31422 = 0 <= img_index_x_31421;
        #line 168 "gpu_device_shm.impala"
        if (_31422) goto l31423; else goto l31466;
    l31466: ;
        #line 170 "gpu_device_shm.impala"
        goto l31463;
    l31423: ;
        #line 168 "gpu_device_shm.impala"
        bool _31426;
        _31426 = img_index_x_31421 < _31425;
        #line 168 "gpu_device_shm.impala"
        if (_31426) goto l31427; else goto l31465;
    l31465: ;
        #line 170 "gpu_device_shm.impala"
        goto l31463;
    l31427: ;
        #line 168 "gpu_device_shm.impala"
        if (_31431) goto l31432; else goto l31464;
    l31464: ;
        #line 170 "gpu_device_shm.impala"
        goto l31463;
    l31432: ;
        #line 168 "gpu_device_shm.impala"
        if (_31435) goto l31436; else goto l31462;
    l31462: ;
        #line 170 "gpu_device_shm.impala"
        goto l31463;
    l31463: ;
        #line 33 "gpu_device_shm.impala"
        goto l31446;
    l31436: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31439 = threadIdx_y();
        p_31439 = _31439;
    l31437: ;
        _31439 = p_31439;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31442 = blockDim_y();
        p_31442 = _31442;
    l31440: ;
        _31442 = p_31442;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31445 = threadIdx_x();
        p_31445 = _31445;
    l31443: ;
        _31445 = p_31445;
        #line 169 "gpu_device_shm.impala"
        int _31450;
        _31450 = _31449 + img_index_x_31421;
        #line 169 "gpu_device_shm.impala"
        int _31455;
        _31455 = _31442 + _31399;
        #line 169 "gpu_device_shm.impala"
        int _31456;
        _31456 = _31439 * _31455;
        #line 169 "gpu_device_shm.impala"
        double* _31451;
        _31451 = _26235_31342 + _31450;
        #line 169 "gpu_device_shm.impala"
        int _31457;
        _31457 = _31456 + _31445;
        #line 169 "gpu_device_shm.impala"
        double _31452;
        _31452 = *_31451;
        #line 169 "gpu_device_shm.impala"
        int _31458;
        _31458 = _31457 + _31410;
        #line 169 "gpu_device_shm.impala"
        double _31460;
        _31460 = _31452;
        #line 169 "gpu_device_shm.impala"
        double* _31459;
        _31459 = shared_31388 + _31458;
        #line 169 "gpu_device_shm.impala"
        *_31459 = _31460;
        #line 33 "gpu_device_shm.impala"
        goto l31446;
    l31446: ;
        #line 35 "gpu_device_shm.impala"
        int _31448;
        _31448 = _31410 + _31408;
        #line 31 "gpu_device_shm.impala"
        p_31410 = _31448;
        goto l31409;
}

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_26506(struct_image_6444 _26509_31597, double* _26510_31598, struct_filter_6450 _26511_31599, struct_Buffer_6445 _26512_31600, double* _26513_31601, struct_Buffer_6445 _26514_31602) {
    int  _31605;
    int p_31605;
    int  _31608;
    int p_31608;
    int  _31611;
    int p_31611;
    int  _31614;
    int p_31614;
    int  _31617;
    int p_31617;
    int  _31620;
    int p_31620;
    double*  shared_31623;
    double* pshared_31623;
    int  _31626;
    int p_31626;
    int  _31629;
    int p_31629;
    int  _31637;
    int p_31637;
    int  _31640;
    int p_31640;
    int  _31642;
    int p_31642;
    int  _31697;
    int p_31697;
    int  _31771;
    int p_31771;
    double  sum_31773;
    double psum_31773;
    int  _31703;
    int p_31703;
    int  _31706;
    int p_31706;
    int  _31708;
    int p_31708;
    int  _31725;
    int p_31725;
    int  _31728;
    int p_31728;
    int  _31731;
    int p_31731;
    int  _31668;
    int p_31668;
    int  _31671;
    int p_31671;
    int  _31674;
    int p_31674;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _31605 = blockIdx_x();
    p_31605 = _31605;
    l31603: ;
        _31605 = p_31605;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31608 = blockDim_x();
        p_31608 = _31608;
    l31606: ;
        _31608 = p_31608;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31611 = threadIdx_x();
        p_31611 = _31611;
    l31609: ;
        _31611 = p_31611;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31614 = blockIdx_y();
        p_31614 = _31614;
    l31612: ;
        _31614 = p_31614;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31617 = blockDim_y();
        p_31617 = _31617;
    l31615: ;
        _31617 = p_31617;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31620 = threadIdx_y();
        p_31620 = _31620;
    l31618: ;
        _31620 = p_31620;
        #line 160 "gpu_device_shm.impala"
        __shared__ double reserver_shared_31623[2130];
        pshared_31623 = reserver_shared_31623;
    l31621: ;
        shared_31623 = pshared_31623;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31626 = blockDim_y();
        p_31626 = _31626;
    l31624: ;
        _31626 = p_31626;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31629 = blockDim_y();
        p_31629 = _31629;
    l31627: ;
        _31629 = p_31629;
        #line 11 "main.impala"
        int _31663;
        _31663 = _26509_31597.e4;
        #line 158 "gpu_device_shm.impala"
        int _31658;
        _31658 = _31614 * _31617;
        #line 11 "main.impala"
        int _31655;
        _31655 = _26509_31597.e3;
        #line 156 "gpu_device_shm.impala"
        int _31649;
        _31649 = _31605 * _31608;
        #line 6 "gaussian.impala"
        int _31630;
        _31630 = _26511_31599.e2;
        #line 158 "gpu_device_shm.impala"
        int gid_y_31659;
        gid_y_31659 = _31658 + _31620;
        #line 156 "gpu_device_shm.impala"
        int gid_x_31650;
        gid_x_31650 = _31649 + _31611;
        #line 163 "gpu_device_shm.impala"
        int _31631;
        _31631 = 2 * _31630;
        #line 163 "gpu_device_shm.impala"
        int _31632;
        _31632 = _31626 + _31631;
        #line 31 "gpu_device_shm.impala"
        bool _31633;
        _31633 = 0 < _31632;
        #line 31 "gpu_device_shm.impala"
        if (_31633) goto l31634; else goto l31816;
    l31816: ;
        #line 172 "gpu_device_shm.impala"
        goto l31756;
    l31634: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31637 = blockDim_x();
        p_31637 = _31637;
    l31635: ;
        _31637 = p_31637;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31640 = blockDim_x();
        p_31640 = _31640;
    l31638: ;
        _31640 = p_31640;
        #line 166 "gpu_device_shm.impala"
        int _31660;
        _31660 = gid_y_31659 - _31630;
        #line 168 "gpu_device_shm.impala"
        bool _31664;
        _31664 = _31660 < _31663;
        #line 164 "gpu_device_shm.impala"
        int _31644;
        _31644 = _26511_31599.e1;
        #line 165 "gpu_device_shm.impala"
        int _31651;
        _31651 = gid_x_31650 - _31644;
        #line 169 "gpu_device_shm.impala"
        int _31678;
        _31678 = _31660 * _31655;
        #line 164 "gpu_device_shm.impala"
        int _31645;
        _31645 = 2 * _31644;
        #line 168 "gpu_device_shm.impala"
        bool _31661;
        _31661 = 0 <= _31660;
        #line 164 "gpu_device_shm.impala"
        int _31646;
        _31646 = _31637 + _31645;
        #line 31 "gpu_device_shm.impala"
        p_31642 = 0;
        goto l31641;
    l31641: ;
        _31642 = p_31642;
        #line 31 "gpu_device_shm.impala"
        bool _31647;
        _31647 = _31642 < _31646;
        #line 31 "gpu_device_shm.impala"
        if (_31647) goto l31648; else goto l31695;
    l31695: ;
        #line 31 "gpu_device_shm.impala"
        p_31697 = _31629;
        goto l31696;
    l31696: ;
        _31697 = p_31697;
        #line 31 "gpu_device_shm.impala"
        bool _31699;
        _31699 = _31697 < _31632;
        #line 31 "gpu_device_shm.impala"
        if (_31699) goto l31700; else goto l31755;
    l31755: ;
        #line 172 "gpu_device_shm.impala"
        goto l31756;
    l31756: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        __syncthreads();
    l31758: ;
        #line 178 "gpu_device_shm.impala"
        bool _31760;
        _31760 = gid_x_31650 < _31655;
        #line 178 "gpu_device_shm.impala"
        if (_31760) goto l31761; else goto l31815;
    l31815: ;
        #line 181 "gpu_device_shm.impala"
        goto l31814;
    l31761: ;
        #line 178 "gpu_device_shm.impala"
        bool _31762;
        _31762 = gid_y_31659 < _31663;
        #line 178 "gpu_device_shm.impala"
        if (_31762) goto l31763; else goto l31813;
    l31813: ;
        #line 181 "gpu_device_shm.impala"
        goto l31814;
    l31814: ;
        return ;
    l31763: ;
        #line 57 "gpu_device_shm.impala"
        char* _31798;
        _31798 = _26514_31602.e1;
        #line 6 "gaussian.impala"
        int v_anchor_31764;
        v_anchor_31764 = _31630 / 2;
        #line 51 "gaussian.impala"
        int _31800;
        _31800 = gid_y_31659 * _31655;
        #line 39 "gaussian.impala"
        bool _31765;
        _31765 = v_anchor_31764 <= gid_y_31659;
        #line 52 "gpu_device_shm.impala"
        char* _31785;
        _31785 = _26512_31600.e1;
        #line 57 "gpu_device_shm.impala"
        double* _31799;
        union { double* dst; char* src; } u_31799;
        u_31799.src = _31798;
        _31799 = u_31799.dst;
        #line 51 "gaussian.impala"
        int _31801;
        _31801 = _31800 + gid_x_31650;
        #line 52 "gpu_device_shm.impala"
        double* _31786;
        union { double* dst; char* src; } u_31786;
        u_31786.src = _31785;
        _31786 = u_31786.dst;
        #line 57 "gpu_device_shm.impala"
        double* _31802;
        _31802 = _31799 + _31801;
        #line 39 "gaussian.impala"
        if (_31765) goto l31766; else goto l31812;
    l31812: ;
        #line 49 "gaussian.impala"
        goto l31806;
    l31766: ;
        #line 39 "gaussian.impala"
        int _31767;
        _31767 = _31663 - v_anchor_31764;
        #line 39 "gaussian.impala"
        bool _31768;
        _31768 = gid_y_31659 < _31767;
        #line 39 "gaussian.impala"
        if (_31768) goto l31769; else goto l31805;
    l31805: ;
        #line 49 "gaussian.impala"
        goto l31806;
    l31806: ;
        #line 51 "gpu_device_shm.impala"
        double* index_31807;
        index_31807 = _31786 + _31801;
        #line 52 "gpu_device_shm.impala"
        double _31808;
        _31808 = *index_31807;
        #line 52 "gpu_device_shm.impala"
        double _31810;
        _31810 = _31808;
        #line 57 "gpu_device_shm.impala"
        *_31802 = _31810;
        return ;
    l31769: ;
        #line 67 "gpu_device_shm.impala"
        struct_Buffer_6445 _31778;
        _31778 = _26511_31599.e0;
        #line 41 "gaussian.impala"
        int _31774;
        _31774 = 1 + v_anchor_31764;
        #line 41 "gaussian.impala"
        int _31804;
        _31804 = 0 - v_anchor_31764;
        #line 67 "gpu_device_shm.impala"
        char* _31779;
        _31779 = _31778.e1;
        #line 67 "gpu_device_shm.impala"
        double* _31780;
        union { double* dst; char* src; } u_31780;
        u_31780.src = _31779;
        _31780 = u_31780.dst;
        #line 20 "gpu_device_shm.impala"
        p_31771 = _31804;
        psum_31773 = 0.000000e+00;
        goto l31770;
    l31770: ;
        _31771 = p_31771;
        sum_31773 = psum_31773;
        #line 20 "gpu_device_shm.impala"
        bool _31775;
        _31775 = _31771 < _31774;
        #line 20 "gpu_device_shm.impala"
        if (_31775) goto l31776; else goto l31797;
    l31797: ;
        #line 57 "gpu_device_shm.impala"
        *_31802 = sum_31773;
        return ;
    l31776: ;
        #line 24 "gpu_device_shm.impala"
        int _31777;
        _31777 = 1 + _31771;
        #line 43 "gaussian.impala"
        int _31787;
        _31787 = gid_y_31659 + _31771;
        #line 43 "gaussian.impala"
        int _31781;
        _31781 = _31771 + v_anchor_31764;
        #line 66 "gpu_device_shm.impala"
        double* i_31782;
        i_31782 = _31780 + _31781;
        #line 43 "gaussian.impala"
        int _31788;
        _31788 = _31787 * _31655;
        #line 67 "gpu_device_shm.impala"
        double _31783;
        _31783 = *i_31782;
        #line 43 "gaussian.impala"
        int _31789;
        _31789 = _31788 + gid_x_31650;
        #line 67 "gpu_device_shm.impala"
        double _31793;
        _31793 = _31783;
        #line 51 "gpu_device_shm.impala"
        double* index_31790;
        index_31790 = _31786 + _31789;
        #line 52 "gpu_device_shm.impala"
        double _31791;
        _31791 = *index_31790;
        #line 52 "gpu_device_shm.impala"
        double _31794;
        _31794 = _31791;
        #line 43 "gaussian.impala"
        double _31795;
        _31795 = _31793 * _31794;
        #line 43 "gaussian.impala"
        double _31796;
        _31796 = sum_31773 + _31795;
        #line 20 "gpu_device_shm.impala"
        p_31771 = _31777;
        psum_31773 = _31796;
        goto l31770;
    l31700: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31703 = blockDim_x();
        p_31703 = _31703;
    l31701: ;
        _31703 = p_31703;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31706 = blockDim_x();
        p_31706 = _31706;
    l31704: ;
        _31706 = p_31706;
        #line 164 "gpu_device_shm.impala"
        int _31710;
        _31710 = _31703 + _31645;
        #line 166 "gpu_device_shm.impala"
        int img_index_y_31718;
        img_index_y_31718 = _31660 + _31697;
        #line 168 "gpu_device_shm.impala"
        bool _31721;
        _31721 = img_index_y_31718 < _31663;
        #line 168 "gpu_device_shm.impala"
        bool _31719;
        _31719 = 0 <= img_index_y_31718;
        #line 169 "gpu_device_shm.impala"
        int _31735;
        _31735 = img_index_y_31718 * _31655;
        #line 31 "gpu_device_shm.impala"
        p_31708 = 0;
        goto l31707;
    l31707: ;
        _31708 = p_31708;
        #line 31 "gpu_device_shm.impala"
        bool _31711;
        _31711 = _31708 < _31710;
        #line 31 "gpu_device_shm.impala"
        if (_31711) goto l31712; else goto l31753;
    l31753: ;
        #line 35 "gpu_device_shm.impala"
        int _31754;
        _31754 = _31697 + _31629;
        #line 31 "gpu_device_shm.impala"
        p_31697 = _31754;
        goto l31696;
    l31712: ;
        #line 165 "gpu_device_shm.impala"
        int img_index_x_31713;
        img_index_x_31713 = _31651 + _31708;
        #line 168 "gpu_device_shm.impala"
        bool _31714;
        _31714 = 0 <= img_index_x_31713;
        #line 168 "gpu_device_shm.impala"
        if (_31714) goto l31715; else goto l31752;
    l31752: ;
        #line 170 "gpu_device_shm.impala"
        goto l31749;
    l31715: ;
        #line 168 "gpu_device_shm.impala"
        bool _31716;
        _31716 = img_index_x_31713 < _31655;
        #line 168 "gpu_device_shm.impala"
        if (_31716) goto l31717; else goto l31751;
    l31751: ;
        #line 170 "gpu_device_shm.impala"
        goto l31749;
    l31717: ;
        #line 168 "gpu_device_shm.impala"
        if (_31719) goto l31720; else goto l31750;
    l31750: ;
        #line 170 "gpu_device_shm.impala"
        goto l31749;
    l31720: ;
        #line 168 "gpu_device_shm.impala"
        if (_31721) goto l31722; else goto l31748;
    l31748: ;
        #line 170 "gpu_device_shm.impala"
        goto l31749;
    l31749: ;
        #line 33 "gpu_device_shm.impala"
        goto l31732;
    l31722: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31725 = threadIdx_y();
        p_31725 = _31725;
    l31723: ;
        _31725 = p_31725;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31728 = blockDim_y();
        p_31728 = _31728;
    l31726: ;
        _31728 = p_31728;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31731 = threadIdx_x();
        p_31731 = _31731;
    l31729: ;
        _31731 = p_31731;
        #line 169 "gpu_device_shm.impala"
        int _31736;
        _31736 = _31735 + img_index_x_31713;
        #line 169 "gpu_device_shm.impala"
        int _31740;
        _31740 = _31725 + _31697;
        #line 169 "gpu_device_shm.impala"
        int _31741;
        _31741 = _31728 + _31631;
        #line 169 "gpu_device_shm.impala"
        double* _31737;
        _31737 = _26510_31598 + _31736;
        #line 169 "gpu_device_shm.impala"
        int _31742;
        _31742 = _31740 * _31741;
        #line 169 "gpu_device_shm.impala"
        double _31738;
        _31738 = *_31737;
        #line 169 "gpu_device_shm.impala"
        int _31743;
        _31743 = _31742 + _31731;
        #line 169 "gpu_device_shm.impala"
        double _31746;
        _31746 = _31738;
        #line 169 "gpu_device_shm.impala"
        int _31744;
        _31744 = _31743 + _31708;
        #line 169 "gpu_device_shm.impala"
        double* _31745;
        _31745 = shared_31623 + _31744;
        #line 169 "gpu_device_shm.impala"
        *_31745 = _31746;
        #line 33 "gpu_device_shm.impala"
        goto l31732;
    l31732: ;
        #line 35 "gpu_device_shm.impala"
        int _31734;
        _31734 = _31708 + _31706;
        #line 31 "gpu_device_shm.impala"
        p_31708 = _31734;
        goto l31707;
    l31648: ;
        #line 165 "gpu_device_shm.impala"
        int img_index_x_31652;
        img_index_x_31652 = _31651 + _31642;
        #line 168 "gpu_device_shm.impala"
        bool _31653;
        _31653 = 0 <= img_index_x_31652;
        #line 168 "gpu_device_shm.impala"
        if (_31653) goto l31654; else goto l31694;
    l31694: ;
        #line 170 "gpu_device_shm.impala"
        goto l31691;
    l31654: ;
        #line 168 "gpu_device_shm.impala"
        bool _31656;
        _31656 = img_index_x_31652 < _31655;
        #line 168 "gpu_device_shm.impala"
        if (_31656) goto l31657; else goto l31693;
    l31693: ;
        #line 170 "gpu_device_shm.impala"
        goto l31691;
    l31657: ;
        #line 168 "gpu_device_shm.impala"
        if (_31661) goto l31662; else goto l31692;
    l31692: ;
        #line 170 "gpu_device_shm.impala"
        goto l31691;
    l31662: ;
        #line 168 "gpu_device_shm.impala"
        if (_31664) goto l31665; else goto l31690;
    l31690: ;
        #line 170 "gpu_device_shm.impala"
        goto l31691;
    l31691: ;
        #line 33 "gpu_device_shm.impala"
        goto l31675;
    l31665: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31668 = threadIdx_y();
        p_31668 = _31668;
    l31666: ;
        _31668 = p_31668;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31671 = blockDim_y();
        p_31671 = _31671;
    l31669: ;
        _31671 = p_31671;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _31674 = threadIdx_x();
        p_31674 = _31674;
    l31672: ;
        _31674 = p_31674;
        #line 169 "gpu_device_shm.impala"
        int _31679;
        _31679 = _31678 + img_index_x_31652;
        #line 169 "gpu_device_shm.impala"
        double* _31680;
        _31680 = _26510_31598 + _31679;
        #line 169 "gpu_device_shm.impala"
        int _31683;
        _31683 = _31671 + _31631;
        #line 169 "gpu_device_shm.impala"
        double _31681;
        _31681 = *_31680;
        #line 169 "gpu_device_shm.impala"
        int _31684;
        _31684 = _31668 * _31683;
        #line 169 "gpu_device_shm.impala"
        double _31688;
        _31688 = _31681;
        #line 169 "gpu_device_shm.impala"
        int _31685;
        _31685 = _31684 + _31674;
        #line 169 "gpu_device_shm.impala"
        int _31686;
        _31686 = _31685 + _31642;
        #line 169 "gpu_device_shm.impala"
        double* _31687;
        _31687 = shared_31623 + _31686;
        #line 169 "gpu_device_shm.impala"
        *_31687 = _31688;
        #line 33 "gpu_device_shm.impala"
        goto l31675;
    l31675: ;
        #line 35 "gpu_device_shm.impala"
        int _31677;
        _31677 = _31642 + _31640;
        #line 31 "gpu_device_shm.impala"
        p_31642 = _31677;
        goto l31641;
}

}
