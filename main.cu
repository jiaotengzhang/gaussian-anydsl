#include "hip/hip_runtime.h"
extern "C" {
typedef struct {
    int e0;
    char* e1;
} struct_Buffer_5003;
typedef struct {
    struct_Buffer_5003 e0;
    struct_Buffer_5003 e1;
    int e2;
    int e3;
} struct_image_5002;
typedef struct {
    struct_Buffer_5003 e0;
    int e1;
} struct_filter_5006;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_17247(double*, struct_image_5002, struct_filter_5006);
__global__ void lambda_17383(double*, struct_filter_5006, struct_image_5002);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_17247(double* _17250_19529, struct_image_5002 _17251_19530, struct_filter_5006 _17252_19531) {
    int  _19537;
    int p_19537;
    int  _19543;
    int p_19543;
    int  _19549;
    int p_19549;
    int  _19555;
    int p_19555;
    int  _19561;
    int p_19561;
    int  _19567;
    int p_19567;
    int  _19590;
    int p_19590;
    double  sum_19592;
    double psum_19592;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _19537 = blockIdx_x();
    p_19537 = _19537;
    l19535: ;
        _19537 = p_19537;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _19543 = blockDim_x();
        p_19543 = _19543;
    l19541: ;
        _19543 = p_19543;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _19549 = threadIdx_x();
        p_19549 = _19549;
    l19547: ;
        _19549 = p_19549;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _19555 = blockIdx_y();
        p_19555 = _19555;
    l19553: ;
        _19555 = p_19555;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _19561 = blockDim_y();
        p_19561 = _19561;
    l19559: ;
        _19561 = p_19561;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _19567 = threadIdx_y();
        p_19567 = _19567;
    l19565: ;
        _19567 = p_19567;
        #line 50 "gpu_device.impala"
        int _19568;
        _19568 = _19537 * _19543;
        #line 6 "main.impala"
        int _19571;
        _19571 = _17251_19530.e2;
        #line 50 "gpu_device.impala"
        int gid_x_19569;
        gid_x_19569 = _19568 + _19549;
        #line 53 "gpu_device.impala"
        bool _19572;
        _19572 = gid_x_19569 < _19571;
        #line 53 "gpu_device.impala"
        if (_19572) goto l19573; else goto l19638;
    l19638: ;
        #line 55 "gpu_device.impala"
        goto l19637;
    l19573: ;
        #line 51 "gpu_device.impala"
        int _19574;
        _19574 = _19555 * _19561;
        #line 51 "gpu_device.impala"
        int gid_y_19575;
        gid_y_19575 = _19574 + _19567;
        #line 6 "main.impala"
        int _19577;
        _19577 = _17251_19530.e3;
        #line 53 "gpu_device.impala"
        bool _19578;
        _19578 = gid_y_19575 < _19577;
        #line 53 "gpu_device.impala"
        if (_19578) goto l19579; else goto l19636;
    l19636: ;
        #line 55 "gpu_device.impala"
        goto l19637;
    l19637: ;
        return ;
    l19579: ;
        #line 3 "gaussian.impala"
        int _19581;
        _19581 = _17252_19531.e1;
        #line 31 "gpu_device.impala"
        struct_Buffer_5003 _19606;
        _19606 = _17251_19530.e1;
        #line 31 "gpu_device.impala"
        char* _19607;
        _19607 = _19606.e1;
        #line 31 "gpu_device.impala"
        int _19609;
        _19609 = gid_y_19575 * _19571;
        #line 3 "gaussian.impala"
        int anchor_19583;
        anchor_19583 = _19581 / 2;
        #line 31 "gpu_device.impala"
        double* _19608;
        union { double* dst; char* src; } u_19608;
        u_19608.src = _19607;
        _19608 = u_19608.dst;
        #line 31 "gpu_device.impala"
        int _19622;
        _19622 = _19609 + gid_x_19569;
        #line 10 "gaussian.impala"
        bool _19584;
        _19584 = anchor_19583 <= gid_x_19569;
        #line 26 "gpu_device.impala"
        double* _19623;
        _19623 = double* _19621_6;
        union { double* dst; char* src; } u_19621_6;
        u_19621_6.src = // bottom: char* _19620_7;;
        _19621_6 = u_19621_6.dst; + _19622;
        #line 10 "gaussian.impala"
        if (_19584) goto l19585; else goto l19635;
    l19635: ;
        #line 16 "gaussian.impala"
        goto l19629;
    l19585: ;
        #line 10 "gaussian.impala"
        int _19586;
        _19586 = _19571 - anchor_19583;
        #line 10 "gaussian.impala"
        bool _19587;
        _19587 = gid_x_19569 < _19586;
        #line 10 "gaussian.impala"
        if (_19587) goto l19588; else goto l19628;
    l19628: ;
        #line 16 "gaussian.impala"
        goto l19629;
    l19629: ;
        #line 31 "gpu_device.impala"
        double* _19630;
        _19630 = _19608 + _19622;
        #line 31 "gpu_device.impala"
        double _19631;
        _19631 = *_19630;
        #line 31 "gpu_device.impala"
        double _19633;
        _19633 = _19631;
        #line 26 "gpu_device.impala"
        *_19623 = _19633;
        return ;
    l19588: ;
        #line 11 "gaussian.impala"
        int _19626;
        _19626 = 0 - anchor_19583;
        #line 11 "gaussian.impala"
        int _19594;
        _19594 = 1 + anchor_19583;
        #line 36 "gpu_device.impala"
        struct_Buffer_5003 _19599;
        _19599 = _17252_19531.e0;
        #line 36 "gpu_device.impala"
        char* _19600;
        _19600 = _19599.e1;
        #line 36 "gpu_device.impala"
        double* _19601;
        union { double* dst; char* src; } u_19601;
        u_19601.src = _19600;
        _19601 = u_19601.dst;
        #line 14 "gpu_device.impala"
        p_19590 = _19626;
        psum_19592 = 0.000000e+00;
        goto l19589;
    l19589: ;
        _19590 = p_19590;
        sum_19592 = psum_19592;
        #line 14 "gpu_device.impala"
        bool _19595;
        _19595 = _19590 < _19594;
        #line 14 "gpu_device.impala"
        if (_19595) goto l19596; else goto l19619;
    l19619: ;
        #line 26 "gpu_device.impala"
        *_19623 = sum_19592;
        return ;
    l19596: ;
        #line 12 "gaussian.impala"
        int _19602;
        _19602 = _19590 + anchor_19583;
        #line 16 "gpu_device.impala"
        int _19597;
        _19597 = 1 + _19590;
        #line 12 "gaussian.impala"
        int _19610;
        _19610 = gid_x_19569 + _19590;
        #line 35 "gpu_device.impala"
        double* i_19603;
        i_19603 = _19601 + _19602;
        #line 31 "gpu_device.impala"
        int _19611;
        _19611 = _19609 + _19610;
        #line 36 "gpu_device.impala"
        double _19604;
        _19604 = *i_19603;
        #line 31 "gpu_device.impala"
        double* _19612;
        _19612 = _19608 + _19611;
        #line 36 "gpu_device.impala"
        double _19615;
        _19615 = _19604;
        #line 31 "gpu_device.impala"
        double _19613;
        _19613 = *_19612;
        #line 31 "gpu_device.impala"
        double _19616;
        _19616 = _19613;
        #line 12 "gaussian.impala"
        double _19617;
        _19617 = _19615 * _19616;
        #line 12 "gaussian.impala"
        double _19618;
        _19618 = sum_19592 + _19617;
        #line 14 "gpu_device.impala"
        p_19590 = _19597;
        psum_19592 = _19618;
        goto l19589;
}

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_17383(double* _17386_19642, struct_filter_5006 _17387_19643, struct_image_5002 _17388_19644) {
    int  _19647;
    int p_19647;
    int  _19650;
    int p_19650;
    int  _19653;
    int p_19653;
    int  _19656;
    int p_19656;
    int  _19659;
    int p_19659;
    int  _19662;
    int p_19662;
    int  _19681;
    int p_19681;
    double  sum_19683;
    double psum_19683;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _19647 = blockIdx_x();
    p_19647 = _19647;
    l19645: ;
        _19647 = p_19647;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _19650 = blockDim_x();
        p_19650 = _19650;
    l19648: ;
        _19650 = p_19650;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _19653 = threadIdx_x();
        p_19653 = _19653;
    l19651: ;
        _19653 = p_19653;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _19656 = blockIdx_y();
        p_19656 = _19656;
    l19654: ;
        _19656 = p_19656;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _19659 = blockDim_y();
        p_19659 = _19659;
    l19657: ;
        _19659 = p_19659;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _19662 = threadIdx_y();
        p_19662 = _19662;
    l19660: ;
        _19662 = p_19662;
        #line 6 "main.impala"
        int _19665;
        _19665 = _17388_19644.e2;
        #line 50 "gpu_device.impala"
        int _19663;
        _19663 = _19647 * _19650;
        #line 50 "gpu_device.impala"
        int gid_x_19664;
        gid_x_19664 = _19663 + _19653;
        #line 53 "gpu_device.impala"
        bool _19666;
        _19666 = gid_x_19664 < _19665;
        #line 53 "gpu_device.impala"
        if (_19666) goto l19667; else goto l19724;
    l19724: ;
        #line 55 "gpu_device.impala"
        goto l19723;
    l19667: ;
        #line 6 "main.impala"
        int _19670;
        _19670 = _17388_19644.e3;
        #line 51 "gpu_device.impala"
        int _19668;
        _19668 = _19656 * _19659;
        #line 51 "gpu_device.impala"
        int gid_y_19669;
        gid_y_19669 = _19668 + _19662;
        #line 53 "gpu_device.impala"
        bool _19671;
        _19671 = gid_y_19669 < _19670;
        #line 53 "gpu_device.impala"
        if (_19671) goto l19672; else goto l19722;
    l19722: ;
        #line 55 "gpu_device.impala"
        goto l19723;
    l19723: ;
        return ;
    l19672: ;
        #line 24 "gaussian.impala"
        int _19673;
        _19673 = _17387_19643.e1;
        #line 31 "gpu_device.impala"
        int _19709;
        _19709 = gid_y_19669 * _19665;
        #line 31 "gpu_device.impala"
        int _19710;
        _19710 = _19709 + gid_x_19664;
        #line 31 "gpu_device.impala"
        struct_Buffer_5003 _19695;
        _19695 = _17388_19644.e1;
        #line 24 "gaussian.impala"
        int anchor_19674;
        anchor_19674 = _19673 / 2;
        #line 26 "gpu_device.impala"
        double* _19711;
        _19711 = double* _19621_19;
        union { double* dst; char* src; } u_19621_19;
        u_19621_19.src = // bottom: char* _19620_20;;
        _19621_19 = u_19621_19.dst; + _19710;
        #line 31 "gpu_device.impala"
        char* _19696;
        _19696 = _19695.e1;
        #line 31 "gaussian.impala"
        bool _19675;
        _19675 = anchor_19674 <= gid_y_19669;
        #line 31 "gpu_device.impala"
        double* _19697;
        union { double* dst; char* src; } u_19697;
        u_19697.src = _19696;
        _19697 = u_19697.dst;
        #line 31 "gaussian.impala"
        if (_19675) goto l19676; else goto l19721;
    l19721: ;
        #line 37 "gaussian.impala"
        goto l19715;
    l19676: ;
        #line 31 "gaussian.impala"
        int _19677;
        _19677 = _19670 - anchor_19674;
        #line 31 "gaussian.impala"
        bool _19678;
        _19678 = gid_y_19669 < _19677;
        #line 31 "gaussian.impala"
        if (_19678) goto l19679; else goto l19714;
    l19714: ;
        #line 37 "gaussian.impala"
        goto l19715;
    l19715: ;
        #line 31 "gpu_device.impala"
        double* _19716;
        _19716 = _19697 + _19710;
        #line 31 "gpu_device.impala"
        double _19717;
        _19717 = *_19716;
        #line 31 "gpu_device.impala"
        double _19719;
        _19719 = _19717;
        #line 26 "gpu_device.impala"
        *_19711 = _19719;
        return ;
    l19679: ;
        #line 36 "gpu_device.impala"
        struct_Buffer_5003 _19688;
        _19688 = _17387_19643.e0;
        #line 32 "gaussian.impala"
        int _19713;
        _19713 = 0 - anchor_19674;
        #line 32 "gaussian.impala"
        int _19684;
        _19684 = 1 + anchor_19674;
        #line 36 "gpu_device.impala"
        char* _19689;
        _19689 = _19688.e1;
        #line 36 "gpu_device.impala"
        double* _19690;
        union { double* dst; char* src; } u_19690;
        u_19690.src = _19689;
        _19690 = u_19690.dst;
        #line 14 "gpu_device.impala"
        p_19681 = _19713;
        psum_19683 = 0.000000e+00;
        goto l19680;
    l19680: ;
        _19681 = p_19681;
        sum_19683 = psum_19683;
        #line 14 "gpu_device.impala"
        bool _19685;
        _19685 = _19681 < _19684;
        #line 14 "gpu_device.impala"
        if (_19685) goto l19686; else goto l19708;
    l19708: ;
        #line 26 "gpu_device.impala"
        *_19711 = sum_19683;
        return ;
    l19686: ;
        #line 33 "gaussian.impala"
        int _19691;
        _19691 = _19681 + anchor_19674;
        #line 16 "gpu_device.impala"
        int _19687;
        _19687 = 1 + _19681;
        #line 35 "gpu_device.impala"
        double* i_19692;
        i_19692 = _19690 + _19691;
        #line 33 "gaussian.impala"
        int _19698;
        _19698 = gid_y_19669 + _19681;
        #line 31 "gpu_device.impala"
        int _19699;
        _19699 = _19698 * _19665;
        #line 36 "gpu_device.impala"
        double _19693;
        _19693 = *i_19692;
        #line 31 "gpu_device.impala"
        int _19700;
        _19700 = _19699 + gid_x_19664;
        #line 36 "gpu_device.impala"
        double _19704;
        _19704 = _19693;
        #line 31 "gpu_device.impala"
        double* _19701;
        _19701 = _19697 + _19700;
        #line 31 "gpu_device.impala"
        double _19702;
        _19702 = *_19701;
        #line 31 "gpu_device.impala"
        double _19705;
        _19705 = _19702;
        #line 33 "gaussian.impala"
        double _19706;
        _19706 = _19704 * _19705;
        #line 33 "gaussian.impala"
        double _19707;
        _19707 = sum_19683 + _19706;
        #line 14 "gpu_device.impala"
        p_19681 = _19687;
        psum_19683 = _19707;
        goto l19680;
}

}