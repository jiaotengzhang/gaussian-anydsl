
#include <hip/hip_runtime.h>
extern "C" {
typedef struct {
    int e0;
    char* e1;
} struct_Buffer_5489;
typedef struct {
    struct_Buffer_5489 e0;
    struct_Buffer_5489 e1;
    int e2;
    int e3;
} struct_image_5488;
typedef struct {
    struct_Buffer_5489 e0;
    int e1;
} struct_filter_5492;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_20698(struct_image_5488, struct_Buffer_5489, struct_filter_5492);
__global__ void lambda_20830(struct_image_5488, struct_Buffer_5489, double*, struct_Buffer_5489, struct_filter_5492);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_20698(struct_image_5488 _20701_23037, struct_Buffer_5489 _20702_23038, struct_filter_5492 _20703_23039) {
    int  _23045;
    int p_23045;
    int  _23051;
    int p_23051;
    int  _23057;
    int p_23057;
    int  _23063;
    int p_23063;
    int  _23069;
    int p_23069;
    int  _23075;
    int p_23075;
    int  _23098;
    int p_23098;
    double  sum_23100;
    double psum_23100;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _23045 = blockIdx_x();
    p_23045 = _23045;
    l23043: ;
        _23045 = p_23045;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23051 = blockDim_x();
        p_23051 = _23051;
    l23049: ;
        _23051 = p_23051;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23057 = threadIdx_x();
        p_23057 = _23057;
    l23055: ;
        _23057 = p_23057;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23063 = blockIdx_y();
        p_23063 = _23063;
    l23061: ;
        _23063 = p_23063;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23069 = blockDim_y();
        p_23069 = _23069;
    l23067: ;
        _23069 = p_23069;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23075 = threadIdx_y();
        p_23075 = _23075;
    l23073: ;
        _23075 = p_23075;
        #line 87 "gpu_device.impala"
        int _23076;
        _23076 = _23045 * _23051;
        #line 6 "main.impala"
        int _23079;
        _23079 = _20701_23037.e2;
        #line 87 "gpu_device.impala"
        int gid_x_23077;
        gid_x_23077 = _23076 + _23057;
        #line 90 "gpu_device.impala"
        bool _23080;
        _23080 = gid_x_23077 < _23079;
        #line 90 "gpu_device.impala"
        if (_23080) goto l23081; else goto l23146;
    l23146: ;
        #line 92 "gpu_device.impala"
        goto l23145;
    l23081: ;
        #line 88 "gpu_device.impala"
        int _23082;
        _23082 = _23063 * _23069;
        #line 6 "main.impala"
        int _23085;
        _23085 = _20701_23037.e3;
        #line 88 "gpu_device.impala"
        int gid_y_23083;
        gid_y_23083 = _23082 + _23075;
        #line 90 "gpu_device.impala"
        bool _23086;
        _23086 = gid_y_23083 < _23085;
        #line 90 "gpu_device.impala"
        if (_23086) goto l23087; else goto l23144;
    l23144: ;
        #line 92 "gpu_device.impala"
        goto l23145;
    l23145: ;
        return ;
    l23087: ;
        #line 18 "gaussian.impala"
        int _23117;
        _23117 = gid_y_23083 * _23079;
        #line 3 "gaussian.impala"
        int _23089;
        _23089 = _20703_23039.e1;
        #line 39 "gpu_device.impala"
        struct_Buffer_5489 _23114;
        _23114 = _20701_23037.e1;
        #line 25 "gpu_device.impala"
        char* _23128;
        _23128 = _20702_23038.e1;
        #line 3 "gaussian.impala"
        int h_anchor_23091;
        h_anchor_23091 = _23089 / 2;
        #line 18 "gaussian.impala"
        int _23130;
        _23130 = _23117 + gid_x_23077;
        #line 39 "gpu_device.impala"
        char* _23115;
        _23115 = _23114.e1;
        #line 25 "gpu_device.impala"
        double* _23129;
        union { double* dst; char* src; } u_23129;
        u_23129.src = _23128;
        _23129 = u_23129.dst;
        #line 13 "gaussian.impala"
        bool _23092;
        _23092 = h_anchor_23091 <= gid_x_23077;
        #line 24 "gpu_device.impala"
        double* index_23131;
        index_23131 = _23129 + _23130;
        #line 39 "gpu_device.impala"
        double* _23116;
        union { double* dst; char* src; } u_23116;
        u_23116.src = _23115;
        _23116 = u_23116.dst;
        #line 13 "gaussian.impala"
        if (_23092) goto l23093; else goto l23143;
    l23143: ;
        #line 19 "gaussian.impala"
        goto l23137;
    l23093: ;
        #line 13 "gaussian.impala"
        int _23094;
        _23094 = _23079 - h_anchor_23091;
        #line 13 "gaussian.impala"
        bool _23095;
        _23095 = gid_x_23077 < _23094;
        #line 13 "gaussian.impala"
        if (_23095) goto l23096; else goto l23136;
    l23136: ;
        #line 19 "gaussian.impala"
        goto l23137;
    l23137: ;
        #line 39 "gpu_device.impala"
        double* _23138;
        _23138 = _23116 + _23130;
        #line 39 "gpu_device.impala"
        double _23139;
        _23139 = *_23138;
        #line 39 "gpu_device.impala"
        double _23141;
        _23141 = _23139;
        #line 25 "gpu_device.impala"
        *index_23131 = _23141;
        return ;
    l23096: ;
        #line 14 "gaussian.impala"
        int _23134;
        _23134 = 0 - h_anchor_23091;
        #line 14 "gaussian.impala"
        int _23102;
        _23102 = 1 + h_anchor_23091;
        #line 44 "gpu_device.impala"
        struct_Buffer_5489 _23107;
        _23107 = _20703_23039.e0;
        #line 44 "gpu_device.impala"
        char* _23108;
        _23108 = _23107.e1;
        #line 44 "gpu_device.impala"
        double* _23109;
        union { double* dst; char* src; } u_23109;
        u_23109.src = _23108;
        _23109 = u_23109.dst;
        #line 14 "gpu_device.impala"
        p_23098 = _23134;
        psum_23100 = 0.000000e+00;
        goto l23097;
    l23097: ;
        _23098 = p_23098;
        sum_23100 = psum_23100;
        #line 14 "gpu_device.impala"
        bool _23103;
        _23103 = _23098 < _23102;
        #line 14 "gpu_device.impala"
        if (_23103) goto l23104; else goto l23127;
    l23127: ;
        #line 25 "gpu_device.impala"
        *index_23131 = sum_23100;
        return ;
    l23104: ;
        #line 15 "gaussian.impala"
        int _23118;
        _23118 = gid_x_23077 + _23098;
        #line 16 "gpu_device.impala"
        int _23105;
        _23105 = 1 + _23098;
        #line 15 "gaussian.impala"
        int _23110;
        _23110 = _23098 + h_anchor_23091;
        #line 39 "gpu_device.impala"
        int _23119;
        _23119 = _23117 + _23118;
        #line 39 "gpu_device.impala"
        double* _23120;
        _23120 = _23116 + _23119;
        #line 43 "gpu_device.impala"
        double* i_23111;
        i_23111 = _23109 + _23110;
        #line 44 "gpu_device.impala"
        double _23112;
        _23112 = *i_23111;
        #line 44 "gpu_device.impala"
        double _23123;
        _23123 = _23112;
        #line 39 "gpu_device.impala"
        double _23121;
        _23121 = *_23120;
        #line 39 "gpu_device.impala"
        double _23124;
        _23124 = _23121;
        #line 15 "gaussian.impala"
        double _23125;
        _23125 = _23123 * _23124;
        #line 15 "gaussian.impala"
        double _23126;
        _23126 = sum_23100 + _23125;
        #line 14 "gpu_device.impala"
        p_23098 = _23105;
        psum_23100 = _23126;
        goto l23097;
}

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_20830(struct_image_5488 _20833_23150, struct_Buffer_5489 _20834_23151, double* _20835_23152, struct_Buffer_5489 _20836_23153, struct_filter_5492 _20837_23154) {
    int  _23157;
    int p_23157;
    int  _23160;
    int p_23160;
    int  _23163;
    int p_23163;
    int  _23166;
    int p_23166;
    int  _23169;
    int p_23169;
    int  _23172;
    int p_23172;
    int  _23191;
    int p_23191;
    double  sum_23193;
    double psum_23193;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _23157 = blockIdx_x();
    p_23157 = _23157;
    l23155: ;
        _23157 = p_23157;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23160 = blockDim_x();
        p_23160 = _23160;
    l23158: ;
        _23160 = p_23160;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23163 = threadIdx_x();
        p_23163 = _23163;
    l23161: ;
        _23163 = p_23163;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23166 = blockIdx_y();
        p_23166 = _23166;
    l23164: ;
        _23166 = p_23166;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23169 = blockDim_y();
        p_23169 = _23169;
    l23167: ;
        _23169 = p_23169;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23172 = threadIdx_y();
        p_23172 = _23172;
    l23170: ;
        _23172 = p_23172;
        #line 6 "main.impala"
        int _23175;
        _23175 = _20833_23150.e2;
        #line 87 "gpu_device.impala"
        int _23173;
        _23173 = _23157 * _23160;
        #line 87 "gpu_device.impala"
        int gid_x_23174;
        gid_x_23174 = _23173 + _23163;
        #line 90 "gpu_device.impala"
        bool _23176;
        _23176 = gid_x_23174 < _23175;
        #line 90 "gpu_device.impala"
        if (_23176) goto l23177; else goto l23235;
    l23235: ;
        #line 92 "gpu_device.impala"
        goto l23234;
    l23177: ;
        #line 6 "main.impala"
        int _23180;
        _23180 = _20833_23150.e3;
        #line 88 "gpu_device.impala"
        int _23178;
        _23178 = _23166 * _23169;
        #line 88 "gpu_device.impala"
        int gid_y_23179;
        gid_y_23179 = _23178 + _23172;
        #line 90 "gpu_device.impala"
        bool _23181;
        _23181 = gid_y_23179 < _23180;
        #line 90 "gpu_device.impala"
        if (_23181) goto l23182; else goto l23233;
    l23233: ;
        #line 92 "gpu_device.impala"
        goto l23234;
    l23234: ;
        return ;
    l23182: ;
        #line 34 "gpu_device.impala"
        char* _23218;
        _23218 = _20836_23153.e1;
        #line 4 "gaussian.impala"
        int _23183;
        _23183 = _20837_23154.e1;
        #line 34 "gpu_device.impala"
        double* _23219;
        union { double* dst; char* src; } u_23219;
        u_23219.src = _23218;
        _23219 = u_23219.dst;
        #line 4 "gaussian.impala"
        int v_anchor_23184;
        v_anchor_23184 = _23183 / 2;
        #line 29 "gpu_device.impala"
        char* _23205;
        _23205 = _20834_23151.e1;
        #line 36 "gaussian.impala"
        int _23220;
        _23220 = gid_y_23179 * _23175;
        #line 29 "gaussian.impala"
        bool _23185;
        _23185 = v_anchor_23184 <= gid_y_23179;
        #line 29 "gpu_device.impala"
        double* _23206;
        union { double* dst; char* src; } u_23206;
        u_23206.src = _23205;
        _23206 = u_23206.dst;
        #line 36 "gaussian.impala"
        int _23221;
        _23221 = _23220 + gid_x_23174;
        #line 34 "gpu_device.impala"
        double* _23222;
        _23222 = _23219 + _23221;
        #line 29 "gaussian.impala"
        if (_23185) goto l23186; else goto l23232;
    l23232: ;
        #line 35 "gaussian.impala"
        goto l23226;
    l23186: ;
        #line 29 "gaussian.impala"
        int _23187;
        _23187 = _23180 - v_anchor_23184;
        #line 29 "gaussian.impala"
        bool _23188;
        _23188 = gid_y_23179 < _23187;
        #line 29 "gaussian.impala"
        if (_23188) goto l23189; else goto l23225;
    l23225: ;
        #line 35 "gaussian.impala"
        goto l23226;
    l23226: ;
        #line 28 "gpu_device.impala"
        double* index_23227;
        index_23227 = _23206 + _23221;
        #line 29 "gpu_device.impala"
        double _23228;
        _23228 = *index_23227;
        #line 29 "gpu_device.impala"
        double _23230;
        _23230 = _23228;
        #line 34 "gpu_device.impala"
        *_23222 = _23230;
        return ;
    l23189: ;
        #line 44 "gpu_device.impala"
        struct_Buffer_5489 _23198;
        _23198 = _20837_23154.e0;
        #line 30 "gaussian.impala"
        int _23224;
        _23224 = 0 - v_anchor_23184;
        #line 44 "gpu_device.impala"
        char* _23199;
        _23199 = _23198.e1;
        #line 30 "gaussian.impala"
        int _23194;
        _23194 = 1 + v_anchor_23184;
        #line 44 "gpu_device.impala"
        double* _23200;
        union { double* dst; char* src; } u_23200;
        u_23200.src = _23199;
        _23200 = u_23200.dst;
        #line 14 "gpu_device.impala"
        p_23191 = _23224;
        psum_23193 = 0.000000e+00;
        goto l23190;
    l23190: ;
        _23191 = p_23191;
        sum_23193 = psum_23193;
        #line 14 "gpu_device.impala"
        bool _23195;
        _23195 = _23191 < _23194;
        #line 14 "gpu_device.impala"
        if (_23195) goto l23196; else goto l23217;
    l23217: ;
        #line 34 "gpu_device.impala"
        *_23222 = sum_23193;
        return ;
    l23196: ;
        #line 31 "gaussian.impala"
        int _23201;
        _23201 = _23191 + v_anchor_23184;
        #line 43 "gpu_device.impala"
        double* i_23202;
        i_23202 = _23200 + _23201;
        #line 31 "gaussian.impala"
        int _23207;
        _23207 = gid_y_23179 + _23191;
        #line 16 "gpu_device.impala"
        int _23197;
        _23197 = 1 + _23191;
        #line 44 "gpu_device.impala"
        double _23203;
        _23203 = *i_23202;
        #line 31 "gaussian.impala"
        int _23208;
        _23208 = _23207 * _23175;
        #line 44 "gpu_device.impala"
        double _23213;
        _23213 = _23203;
        #line 31 "gaussian.impala"
        int _23209;
        _23209 = _23208 + gid_x_23174;
        #line 28 "gpu_device.impala"
        double* index_23210;
        index_23210 = _23206 + _23209;
        #line 29 "gpu_device.impala"
        double _23211;
        _23211 = *index_23210;
        #line 29 "gpu_device.impala"
        double _23214;
        _23214 = _23211;
        #line 31 "gaussian.impala"
        double _23215;
        _23215 = _23213 * _23214;
        #line 31 "gaussian.impala"
        double _23216;
        _23216 = sum_23193 + _23215;
        #line 14 "gpu_device.impala"
        p_23191 = _23197;
        psum_23193 = _23216;
        goto l23190;
}

}