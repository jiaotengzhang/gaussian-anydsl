
#include <hip/hip_runtime.h>
extern "C" {
typedef struct {
    int e0;
    char* e1;
} struct_Buffer_5474;
typedef struct {
    struct_Buffer_5474 e0;
    struct_Buffer_5474 e1;
    int e2;
    int e3;
} struct_image_5473;
typedef struct {
    struct_Buffer_5474 e0;
    int e1;
} struct_filter_5477;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_20598(struct_image_5473, struct_filter_5477, struct_Buffer_5474);
__global__ void lambda_20730(struct_image_5473, struct_Buffer_5474, struct_filter_5477, struct_Buffer_5474, double*);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_20598(struct_image_5473 _20601_22937, struct_filter_5477 _20602_22938, struct_Buffer_5474 _20603_22939) {
    int  _22945;
    int p_22945;
    int  _22951;
    int p_22951;
    int  _22957;
    int p_22957;
    int  _22963;
    int p_22963;
    int  _22969;
    int p_22969;
    int  _22975;
    int p_22975;
    int  _22998;
    int p_22998;
    double  sum_23000;
    double psum_23000;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _22945 = blockIdx_x();
    p_22945 = _22945;
    l22943: ;
        _22945 = p_22945;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22951 = blockDim_x();
        p_22951 = _22951;
    l22949: ;
        _22951 = p_22951;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22957 = threadIdx_x();
        p_22957 = _22957;
    l22955: ;
        _22957 = p_22957;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22963 = blockIdx_y();
        p_22963 = _22963;
    l22961: ;
        _22963 = p_22963;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22969 = blockDim_y();
        p_22969 = _22969;
    l22967: ;
        _22969 = p_22969;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22975 = threadIdx_y();
        p_22975 = _22975;
    l22973: ;
        _22975 = p_22975;
        #line 11 "main.impala"
        int _22979;
        _22979 = _20601_22937.e2;
        #line 119 "gpu_device.impala"
        int _22976;
        _22976 = _22945 * _22951;
        #line 119 "gpu_device.impala"
        int gid_x_22977;
        gid_x_22977 = _22976 + _22957;
        #line 124 "gpu_device.impala"
        bool _22980;
        _22980 = gid_x_22977 < _22979;
        #line 124 "gpu_device.impala"
        if (_22980) goto l22981; else goto l23046;
    l23046: ;
        #line 127 "gpu_device.impala"
        goto l23045;
    l22981: ;
        #line 121 "gpu_device.impala"
        int _22982;
        _22982 = _22963 * _22969;
        #line 11 "main.impala"
        int _22985;
        _22985 = _20601_22937.e3;
        #line 121 "gpu_device.impala"
        int gid_y_22983;
        gid_y_22983 = _22982 + _22975;
        #line 124 "gpu_device.impala"
        bool _22986;
        _22986 = gid_y_22983 < _22985;
        #line 124 "gpu_device.impala"
        if (_22986) goto l22987; else goto l23044;
    l23044: ;
        #line 127 "gpu_device.impala"
        goto l23045;
    l23045: ;
        return ;
    l22987: ;
        #line 25 "gaussian.impala"
        int _23017;
        _23017 = gid_y_22983 * _22979;
        #line 49 "gpu_device.impala"
        struct_Buffer_5474 _23014;
        _23014 = _20601_22937.e1;
        #line 34 "gpu_device.impala"
        char* _23028;
        _23028 = _20603_22939.e1;
        #line 4 "gaussian.impala"
        int _22989;
        _22989 = _20602_22938.e1;
        #line 25 "gaussian.impala"
        int _23030;
        _23030 = _23017 + gid_x_22977;
        #line 49 "gpu_device.impala"
        char* _23015;
        _23015 = _23014.e1;
        #line 34 "gpu_device.impala"
        double* _23029;
        union { double* dst; char* src; } u_23029;
        u_23029.src = _23028;
        _23029 = u_23029.dst;
        #line 4 "gaussian.impala"
        int h_anchor_22991;
        h_anchor_22991 = _22989 / 2;
        #line 33 "gpu_device.impala"
        double* index_23031;
        index_23031 = _23029 + _23030;
        #line 49 "gpu_device.impala"
        double* _23016;
        union { double* dst; char* src; } u_23016;
        u_23016.src = _23015;
        _23016 = u_23016.dst;
        #line 17 "gaussian.impala"
        bool _22992;
        _22992 = h_anchor_22991 <= gid_x_22977;
        #line 17 "gaussian.impala"
        if (_22992) goto l22993; else goto l23043;
    l23043: ;
        #line 27 "gaussian.impala"
        goto l23037;
    l22993: ;
        #line 17 "gaussian.impala"
        int _22994;
        _22994 = _22979 - h_anchor_22991;
        #line 17 "gaussian.impala"
        bool _22995;
        _22995 = gid_x_22977 < _22994;
        #line 17 "gaussian.impala"
        if (_22995) goto l22996; else goto l23036;
    l23036: ;
        #line 27 "gaussian.impala"
        goto l23037;
    l23037: ;
        #line 49 "gpu_device.impala"
        double* _23038;
        _23038 = _23016 + _23030;
        #line 49 "gpu_device.impala"
        double _23039;
        _23039 = *_23038;
        #line 49 "gpu_device.impala"
        double _23041;
        _23041 = _23039;
        #line 34 "gpu_device.impala"
        *index_23031 = _23041;
        return ;
    l22996: ;
        #line 19 "gaussian.impala"
        int _23002;
        _23002 = 1 + h_anchor_22991;
        #line 54 "gpu_device.impala"
        struct_Buffer_5474 _23007;
        _23007 = _20602_22938.e0;
        #line 19 "gaussian.impala"
        int _23034;
        _23034 = 0 - h_anchor_22991;
        #line 54 "gpu_device.impala"
        char* _23008;
        _23008 = _23007.e1;
        #line 54 "gpu_device.impala"
        double* _23009;
        union { double* dst; char* src; } u_23009;
        u_23009.src = _23008;
        _23009 = u_23009.dst;
        #line 18 "gpu_device.impala"
        p_22998 = _23034;
        psum_23000 = 0.000000e+00;
        goto l22997;
    l22997: ;
        _22998 = p_22998;
        sum_23000 = psum_23000;
        #line 18 "gpu_device.impala"
        bool _23003;
        _23003 = _22998 < _23002;
        #line 18 "gpu_device.impala"
        if (_23003) goto l23004; else goto l23027;
    l23027: ;
        #line 34 "gpu_device.impala"
        *index_23031 = sum_23000;
        return ;
    l23004: ;
        #line 21 "gaussian.impala"
        int _23010;
        _23010 = _22998 + h_anchor_22991;
        #line 22 "gpu_device.impala"
        int _23005;
        _23005 = 1 + _22998;
        #line 53 "gpu_device.impala"
        double* i_23011;
        i_23011 = _23009 + _23010;
        #line 21 "gaussian.impala"
        int _23018;
        _23018 = gid_x_22977 + _22998;
        #line 54 "gpu_device.impala"
        double _23012;
        _23012 = *i_23011;
        #line 49 "gpu_device.impala"
        int _23019;
        _23019 = _23017 + _23018;
        #line 54 "gpu_device.impala"
        double _23023;
        _23023 = _23012;
        #line 49 "gpu_device.impala"
        double* _23020;
        _23020 = _23016 + _23019;
        #line 49 "gpu_device.impala"
        double _23021;
        _23021 = *_23020;
        #line 49 "gpu_device.impala"
        double _23024;
        _23024 = _23021;
        #line 21 "gaussian.impala"
        double _23025;
        _23025 = _23023 * _23024;
        #line 21 "gaussian.impala"
        double _23026;
        _23026 = sum_23000 + _23025;
        #line 18 "gpu_device.impala"
        p_22998 = _23005;
        psum_23000 = _23026;
        goto l22997;
}

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_20730(struct_image_5473 _20733_23050, struct_Buffer_5474 _20734_23051, struct_filter_5477 _20735_23052, struct_Buffer_5474 _20736_23053, double* _20737_23054) {
    int  _23057;
    int p_23057;
    int  _23060;
    int p_23060;
    int  _23063;
    int p_23063;
    int  _23066;
    int p_23066;
    int  _23069;
    int p_23069;
    int  _23072;
    int p_23072;
    int  _23091;
    int p_23091;
    double  sum_23093;
    double psum_23093;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _23057 = blockIdx_x();
    p_23057 = _23057;
    l23055: ;
        _23057 = p_23057;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23060 = blockDim_x();
        p_23060 = _23060;
    l23058: ;
        _23060 = p_23060;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23063 = threadIdx_x();
        p_23063 = _23063;
    l23061: ;
        _23063 = p_23063;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23066 = blockIdx_y();
        p_23066 = _23066;
    l23064: ;
        _23066 = p_23066;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23069 = blockDim_y();
        p_23069 = _23069;
    l23067: ;
        _23069 = p_23069;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _23072 = threadIdx_y();
        p_23072 = _23072;
    l23070: ;
        _23072 = p_23072;
        #line 119 "gpu_device.impala"
        int _23073;
        _23073 = _23057 * _23060;
        #line 119 "gpu_device.impala"
        int gid_x_23074;
        gid_x_23074 = _23073 + _23063;
        #line 11 "main.impala"
        int _23075;
        _23075 = _20733_23050.e2;
        #line 124 "gpu_device.impala"
        bool _23076;
        _23076 = gid_x_23074 < _23075;
        #line 124 "gpu_device.impala"
        if (_23076) goto l23077; else goto l23135;
    l23135: ;
        #line 127 "gpu_device.impala"
        goto l23134;
    l23077: ;
        #line 11 "main.impala"
        int _23080;
        _23080 = _20733_23050.e3;
        #line 121 "gpu_device.impala"
        int _23078;
        _23078 = _23066 * _23069;
        #line 121 "gpu_device.impala"
        int gid_y_23079;
        gid_y_23079 = _23078 + _23072;
        #line 124 "gpu_device.impala"
        bool _23081;
        _23081 = gid_y_23079 < _23080;
        #line 124 "gpu_device.impala"
        if (_23081) goto l23082; else goto l23133;
    l23133: ;
        #line 127 "gpu_device.impala"
        goto l23134;
    l23134: ;
        return ;
    l23082: ;
        #line 44 "gpu_device.impala"
        char* _23118;
        _23118 = _20736_23053.e1;
        #line 39 "gpu_device.impala"
        char* _23105;
        _23105 = _20734_23051.e1;
        #line 44 "gpu_device.impala"
        double* _23119;
        union { double* dst; char* src; } u_23119;
        u_23119.src = _23118;
        _23119 = u_23119.dst;
        #line 6 "gaussian.impala"
        int _23083;
        _23083 = _20735_23052.e1;
        #line 51 "gaussian.impala"
        int _23120;
        _23120 = gid_y_23079 * _23075;
        #line 39 "gpu_device.impala"
        double* _23106;
        union { double* dst; char* src; } u_23106;
        u_23106.src = _23105;
        _23106 = u_23106.dst;
        #line 6 "gaussian.impala"
        int v_anchor_23084;
        v_anchor_23084 = _23083 / 2;
        #line 51 "gaussian.impala"
        int _23121;
        _23121 = _23120 + gid_x_23074;
        #line 39 "gaussian.impala"
        bool _23085;
        _23085 = v_anchor_23084 <= gid_y_23079;
        #line 44 "gpu_device.impala"
        double* _23122;
        _23122 = _23119 + _23121;
        #line 39 "gaussian.impala"
        if (_23085) goto l23086; else goto l23132;
    l23132: ;
        #line 49 "gaussian.impala"
        goto l23126;
    l23086: ;
        #line 39 "gaussian.impala"
        int _23087;
        _23087 = _23080 - v_anchor_23084;
        #line 39 "gaussian.impala"
        bool _23088;
        _23088 = gid_y_23079 < _23087;
        #line 39 "gaussian.impala"
        if (_23088) goto l23089; else goto l23125;
    l23125: ;
        #line 49 "gaussian.impala"
        goto l23126;
    l23126: ;
        #line 38 "gpu_device.impala"
        double* index_23127;
        index_23127 = _23106 + _23121;
        #line 39 "gpu_device.impala"
        double _23128;
        _23128 = *index_23127;
        #line 39 "gpu_device.impala"
        double _23130;
        _23130 = _23128;
        #line 44 "gpu_device.impala"
        *_23122 = _23130;
        return ;
    l23089: ;
        #line 41 "gaussian.impala"
        int _23094;
        _23094 = 1 + v_anchor_23084;
        #line 41 "gaussian.impala"
        int _23124;
        _23124 = 0 - v_anchor_23084;
        #line 54 "gpu_device.impala"
        struct_Buffer_5474 _23098;
        _23098 = _20735_23052.e0;
        #line 54 "gpu_device.impala"
        char* _23099;
        _23099 = _23098.e1;
        #line 54 "gpu_device.impala"
        double* _23100;
        union { double* dst; char* src; } u_23100;
        u_23100.src = _23099;
        _23100 = u_23100.dst;
        #line 18 "gpu_device.impala"
        p_23091 = _23124;
        psum_23093 = 0.000000e+00;
        goto l23090;
    l23090: ;
        _23091 = p_23091;
        sum_23093 = psum_23093;
        #line 18 "gpu_device.impala"
        bool _23095;
        _23095 = _23091 < _23094;
        #line 18 "gpu_device.impala"
        if (_23095) goto l23096; else goto l23117;
    l23117: ;
        #line 44 "gpu_device.impala"
        *_23122 = sum_23093;
        return ;
    l23096: ;
        #line 22 "gpu_device.impala"
        int _23097;
        _23097 = 1 + _23091;
        #line 43 "gaussian.impala"
        int _23101;
        _23101 = _23091 + v_anchor_23084;
        #line 53 "gpu_device.impala"
        double* i_23102;
        i_23102 = _23100 + _23101;
        #line 43 "gaussian.impala"
        int _23107;
        _23107 = gid_y_23079 + _23091;
        #line 54 "gpu_device.impala"
        double _23103;
        _23103 = *i_23102;
        #line 43 "gaussian.impala"
        int _23108;
        _23108 = _23107 * _23075;
        #line 54 "gpu_device.impala"
        double _23113;
        _23113 = _23103;
        #line 43 "gaussian.impala"
        int _23109;
        _23109 = _23108 + gid_x_23074;
        #line 38 "gpu_device.impala"
        double* index_23110;
        index_23110 = _23106 + _23109;
        #line 39 "gpu_device.impala"
        double _23111;
        _23111 = *index_23110;
        #line 39 "gpu_device.impala"
        double _23114;
        _23114 = _23111;
        #line 43 "gaussian.impala"
        double _23115;
        _23115 = _23113 * _23114;
        #line 43 "gaussian.impala"
        double _23116;
        _23116 = sum_23093 + _23115;
        #line 18 "gpu_device.impala"
        p_23091 = _23097;
        psum_23093 = _23116;
        goto l23090;
}

}