
#include <hip/hip_runtime.h>
extern "C" {
typedef struct {
    int e0;
    char* e1;
} struct_Buffer_5307;
typedef struct {
    struct_Buffer_5307 e0;
    int e1;
} struct_filter_5306;
typedef struct {
    struct_Buffer_5307 e0;
    struct_Buffer_5307 e1;
    int e2;
    int e3;
} struct_image_5312;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_19830(struct_filter_5306, struct_Buffer_5307, double*, struct_image_5312);
__global__ void lambda_19967(struct_Buffer_5307, double*, struct_image_5312, struct_filter_5306);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_19830(struct_filter_5306 _19833_22179, struct_Buffer_5307 _19834_22180, double* _19835_22181, struct_image_5312 _19836_22182) {
    int  _22188;
    int p_22188;
    int  _22194;
    int p_22194;
    int  _22200;
    int p_22200;
    int  _22206;
    int p_22206;
    int  _22212;
    int p_22212;
    int  _22218;
    int p_22218;
    int  _22241;
    int p_22241;
    double  sum_22243;
    double psum_22243;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _22188 = blockIdx_x();
    p_22188 = _22188;
    l22186: ;
        _22188 = p_22188;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22194 = blockDim_x();
        p_22194 = _22194;
    l22192: ;
        _22194 = p_22194;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22200 = threadIdx_x();
        p_22200 = _22200;
    l22198: ;
        _22200 = p_22200;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22206 = blockIdx_y();
        p_22206 = _22206;
    l22204: ;
        _22206 = p_22206;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22212 = blockDim_y();
        p_22212 = _22212;
    l22210: ;
        _22212 = p_22212;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22218 = threadIdx_y();
        p_22218 = _22218;
    l22216: ;
        _22218 = p_22218;
        #line 6 "main.impala"
        int _22222;
        _22222 = _19836_22182.e2;
        #line 66 "gpu_device.impala"
        int _22219;
        _22219 = _22188 * _22194;
        #line 66 "gpu_device.impala"
        int gid_x_22220;
        gid_x_22220 = _22219 + _22200;
        #line 69 "gpu_device.impala"
        bool _22223;
        _22223 = gid_x_22220 < _22222;
        #line 69 "gpu_device.impala"
        if (_22223) goto l22224; else goto l22289;
    l22289: ;
        #line 71 "gpu_device.impala"
        goto l22288;
    l22224: ;
        #line 67 "gpu_device.impala"
        int _22225;
        _22225 = _22206 * _22212;
        #line 67 "gpu_device.impala"
        int gid_y_22226;
        gid_y_22226 = _22225 + _22218;
        #line 6 "main.impala"
        int _22228;
        _22228 = _19836_22182.e3;
        #line 69 "gpu_device.impala"
        bool _22229;
        _22229 = gid_y_22226 < _22228;
        #line 69 "gpu_device.impala"
        if (_22229) goto l22230; else goto l22287;
    l22287: ;
        #line 71 "gpu_device.impala"
        goto l22288;
    l22288: ;
        return ;
    l22230: ;
        #line 31 "gpu_device.impala"
        struct_Buffer_5307 _22257;
        _22257 = _19836_22182.e1;
        #line 26 "gpu_device.impala"
        char* _22271;
        _22271 = _19834_22180.e1;
        #line 31 "gpu_device.impala"
        int _22260;
        _22260 = gid_y_22226 * _22222;
        #line 31 "gpu_device.impala"
        char* _22258;
        _22258 = _22257.e1;
        #line 3 "gaussian.impala"
        int _22232;
        _22232 = _19833_22179.e1;
        #line 31 "gpu_device.impala"
        double* _22259;
        union { double* dst; char* src; } u_22259;
        u_22259.src = _22258;
        _22259 = u_22259.dst;
        #line 3 "gaussian.impala"
        int anchor_22234;
        anchor_22234 = _22232 / 2;
        #line 26 "gpu_device.impala"
        double* _22272;
        union { double* dst; char* src; } u_22272;
        u_22272.src = _22271;
        _22272 = u_22272.dst;
        #line 31 "gpu_device.impala"
        int _22273;
        _22273 = _22260 + gid_x_22220;
        #line 10 "gaussian.impala"
        bool _22235;
        _22235 = anchor_22234 <= gid_x_22220;
        #line 26 "gpu_device.impala"
        double* _22274;
        _22274 = _22272 + _22273;
        #line 10 "gaussian.impala"
        if (_22235) goto l22236; else goto l22286;
    l22286: ;
        #line 16 "gaussian.impala"
        goto l22280;
    l22236: ;
        #line 10 "gaussian.impala"
        int _22237;
        _22237 = _22222 - anchor_22234;
        #line 10 "gaussian.impala"
        bool _22238;
        _22238 = gid_x_22220 < _22237;
        #line 10 "gaussian.impala"
        if (_22238) goto l22239; else goto l22279;
    l22279: ;
        #line 16 "gaussian.impala"
        goto l22280;
    l22280: ;
        #line 31 "gpu_device.impala"
        double* _22281;
        _22281 = _22259 + _22273;
        #line 31 "gpu_device.impala"
        double _22282;
        _22282 = *_22281;
        #line 31 "gpu_device.impala"
        double _22284;
        _22284 = _22282;
        #line 26 "gpu_device.impala"
        *_22274 = _22284;
        return ;
    l22239: ;
        #line 11 "gaussian.impala"
        int _22245;
        _22245 = 1 + anchor_22234;
        #line 36 "gpu_device.impala"
        struct_Buffer_5307 _22250;
        _22250 = _19833_22179.e0;
        #line 11 "gaussian.impala"
        int _22277;
        _22277 = 0 - anchor_22234;
        #line 36 "gpu_device.impala"
        char* _22251;
        _22251 = _22250.e1;
        #line 36 "gpu_device.impala"
        double* _22252;
        union { double* dst; char* src; } u_22252;
        u_22252.src = _22251;
        _22252 = u_22252.dst;
        #line 14 "gpu_device.impala"
        p_22241 = _22277;
        psum_22243 = 0.000000e+00;
        goto l22240;
    l22240: ;
        _22241 = p_22241;
        sum_22243 = psum_22243;
        #line 14 "gpu_device.impala"
        bool _22246;
        _22246 = _22241 < _22245;
        #line 14 "gpu_device.impala"
        if (_22246) goto l22247; else goto l22270;
    l22270: ;
        #line 26 "gpu_device.impala"
        *_22274 = sum_22243;
        return ;
    l22247: ;
        #line 12 "gaussian.impala"
        int _22253;
        _22253 = _22241 + anchor_22234;
        #line 35 "gpu_device.impala"
        double* i_22254;
        i_22254 = _22252 + _22253;
        #line 36 "gpu_device.impala"
        double _22255;
        _22255 = *i_22254;
        #line 12 "gaussian.impala"
        int _22261;
        _22261 = gid_x_22220 + _22241;
        #line 36 "gpu_device.impala"
        double _22266;
        _22266 = _22255;
        #line 16 "gpu_device.impala"
        int _22248;
        _22248 = 1 + _22241;
        #line 31 "gpu_device.impala"
        int _22262;
        _22262 = _22260 + _22261;
        #line 31 "gpu_device.impala"
        double* _22263;
        _22263 = _22259 + _22262;
        #line 31 "gpu_device.impala"
        double _22264;
        _22264 = *_22263;
        #line 31 "gpu_device.impala"
        double _22267;
        _22267 = _22264;
        #line 12 "gaussian.impala"
        double _22268;
        _22268 = _22266 * _22267;
        #line 12 "gaussian.impala"
        double _22269;
        _22269 = sum_22243 + _22268;
        #line 14 "gpu_device.impala"
        p_22241 = _22248;
        psum_22243 = _22269;
        goto l22240;
}

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_19967(struct_Buffer_5307 _19970_22293, double* _19971_22294, struct_image_5312 _19972_22295, struct_filter_5306 _19973_22296) {
    int  _22299;
    int p_22299;
    int  _22302;
    int p_22302;
    int  _22305;
    int p_22305;
    int  _22308;
    int p_22308;
    int  _22311;
    int p_22311;
    int  _22314;
    int p_22314;
    int  _22333;
    int p_22333;
    double  sum_22335;
    double psum_22335;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _22299 = blockIdx_x();
    p_22299 = _22299;
    l22297: ;
        _22299 = p_22299;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22302 = blockDim_x();
        p_22302 = _22302;
    l22300: ;
        _22302 = p_22302;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22305 = threadIdx_x();
        p_22305 = _22305;
    l22303: ;
        _22305 = p_22305;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22308 = blockIdx_y();
        p_22308 = _22308;
    l22306: ;
        _22308 = p_22308;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22311 = blockDim_y();
        p_22311 = _22311;
    l22309: ;
        _22311 = p_22311;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _22314 = threadIdx_y();
        p_22314 = _22314;
    l22312: ;
        _22314 = p_22314;
        #line 66 "gpu_device.impala"
        int _22315;
        _22315 = _22299 * _22302;
        #line 6 "main.impala"
        int _22317;
        _22317 = _19972_22295.e2;
        #line 66 "gpu_device.impala"
        int gid_x_22316;
        gid_x_22316 = _22315 + _22305;
        #line 69 "gpu_device.impala"
        bool _22318;
        _22318 = gid_x_22316 < _22317;
        #line 69 "gpu_device.impala"
        if (_22318) goto l22319; else goto l22378;
    l22378: ;
        #line 71 "gpu_device.impala"
        goto l22377;
    l22319: ;
        #line 6 "main.impala"
        int _22322;
        _22322 = _19972_22295.e3;
        #line 67 "gpu_device.impala"
        int _22320;
        _22320 = _22308 * _22311;
        #line 67 "gpu_device.impala"
        int gid_y_22321;
        gid_y_22321 = _22320 + _22314;
        #line 69 "gpu_device.impala"
        bool _22323;
        _22323 = gid_y_22321 < _22322;
        #line 69 "gpu_device.impala"
        if (_22323) goto l22324; else goto l22376;
    l22376: ;
        #line 71 "gpu_device.impala"
        goto l22377;
    l22377: ;
        return ;
    l22324: ;
        #line 31 "gpu_device.impala"
        struct_Buffer_5307 _22347;
        _22347 = _19972_22295.e1;
        #line 26 "gpu_device.impala"
        char* _22361;
        _22361 = _19970_22293.e1;
        #line 31 "gpu_device.impala"
        char* _22348;
        _22348 = _22347.e1;
        #line 24 "gaussian.impala"
        int _22325;
        _22325 = _19973_22296.e1;
        #line 31 "gpu_device.impala"
        double* _22349;
        union { double* dst; char* src; } u_22349;
        u_22349.src = _22348;
        _22349 = u_22349.dst;
        #line 31 "gpu_device.impala"
        int _22363;
        _22363 = gid_y_22321 * _22317;
        #line 26 "gpu_device.impala"
        double* _22362;
        union { double* dst; char* src; } u_22362;
        u_22362.src = _22361;
        _22362 = u_22362.dst;
        #line 24 "gaussian.impala"
        int anchor_22326;
        anchor_22326 = _22325 / 2;
        #line 31 "gpu_device.impala"
        int _22364;
        _22364 = _22363 + gid_x_22316;
        #line 26 "gpu_device.impala"
        double* _22365;
        _22365 = _22362 + _22364;
        #line 31 "gaussian.impala"
        bool _22327;
        _22327 = anchor_22326 <= gid_y_22321;
        #line 31 "gaussian.impala"
        if (_22327) goto l22328; else goto l22375;
    l22375: ;
        #line 37 "gaussian.impala"
        goto l22369;
    l22328: ;
        #line 31 "gaussian.impala"
        int _22329;
        _22329 = _22322 - anchor_22326;
        #line 31 "gaussian.impala"
        bool _22330;
        _22330 = gid_y_22321 < _22329;
        #line 31 "gaussian.impala"
        if (_22330) goto l22331; else goto l22368;
    l22368: ;
        #line 37 "gaussian.impala"
        goto l22369;
    l22369: ;
        #line 31 "gpu_device.impala"
        double* _22370;
        _22370 = _22349 + _22364;
        #line 31 "gpu_device.impala"
        double _22371;
        _22371 = *_22370;
        #line 31 "gpu_device.impala"
        double _22373;
        _22373 = _22371;
        #line 26 "gpu_device.impala"
        *_22365 = _22373;
        return ;
    l22331: ;
        #line 36 "gpu_device.impala"
        struct_Buffer_5307 _22340;
        _22340 = _19973_22296.e0;
        #line 32 "gaussian.impala"
        int _22367;
        _22367 = 0 - anchor_22326;
        #line 32 "gaussian.impala"
        int _22336;
        _22336 = 1 + anchor_22326;
        #line 36 "gpu_device.impala"
        char* _22341;
        _22341 = _22340.e1;
        #line 36 "gpu_device.impala"
        double* _22342;
        union { double* dst; char* src; } u_22342;
        u_22342.src = _22341;
        _22342 = u_22342.dst;
        #line 14 "gpu_device.impala"
        p_22333 = _22367;
        psum_22335 = 0.000000e+00;
        goto l22332;
    l22332: ;
        _22333 = p_22333;
        sum_22335 = psum_22335;
        #line 14 "gpu_device.impala"
        bool _22337;
        _22337 = _22333 < _22336;
        #line 14 "gpu_device.impala"
        if (_22337) goto l22338; else goto l22360;
    l22360: ;
        #line 26 "gpu_device.impala"
        *_22365 = sum_22335;
        return ;
    l22338: ;
        #line 33 "gaussian.impala"
        int _22350;
        _22350 = gid_y_22321 + _22333;
        #line 16 "gpu_device.impala"
        int _22339;
        _22339 = 1 + _22333;
        #line 31 "gpu_device.impala"
        int _22351;
        _22351 = _22350 * _22317;
        #line 33 "gaussian.impala"
        int _22343;
        _22343 = _22333 + anchor_22326;
        #line 31 "gpu_device.impala"
        int _22352;
        _22352 = _22351 + gid_x_22316;
        #line 35 "gpu_device.impala"
        double* i_22344;
        i_22344 = _22342 + _22343;
        #line 31 "gpu_device.impala"
        double* _22353;
        _22353 = _22349 + _22352;
        #line 36 "gpu_device.impala"
        double _22345;
        _22345 = *i_22344;
        #line 36 "gpu_device.impala"
        double _22356;
        _22356 = _22345;
        #line 31 "gpu_device.impala"
        double _22354;
        _22354 = *_22353;
        #line 31 "gpu_device.impala"
        double _22357;
        _22357 = _22354;
        #line 33 "gaussian.impala"
        double _22358;
        _22358 = _22356 * _22357;
        #line 33 "gaussian.impala"
        double _22359;
        _22359 = sum_22335 + _22358;
        #line 14 "gpu_device.impala"
        p_22333 = _22339;
        psum_22335 = _22359;
        goto l22332;
}

}