
#include <hip/hip_runtime.h>
extern "C" {
typedef struct {
    int e0;
    char* e1;
} struct_Buffer_6341;
typedef struct {
    struct_Buffer_6341 e0;
    int e1;
    int e2;
} struct_filter_6340;
typedef struct {
    struct_Buffer_6341 e0;
    struct_Buffer_6341 e1;
    int e2;
    int e3;
} struct_image_6346;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_25604(struct_filter_6340, double*, struct_Buffer_6341, struct_image_6346);
__global__ void lambda_25866(double*, double*, struct_filter_6340, struct_Buffer_6341, struct_image_6346);

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_25604(struct_filter_6340 _25607_30475, double* _25608_30476, struct_Buffer_6341 _25609_30477, struct_image_6346 _25610_30478) {
    int  bdimx_30484;
    int pbdimx_30484;
    int  bdimy_30490;
    int pbdimy_30490;
    int  bidx_30496;
    int pbidx_30496;
    int  bidy_30502;
    int pbidy_30502;
    int  tidx_30508;
    int ptidx_30508;
    int  tidy_30514;
    int ptidy_30514;
    double*  reserve_shared_30522;
    double* preserve_shared_30522;
    int  _30525;
    int p_30525;
    int  _30528;
    int p_30528;
    int  _30539;
    int p_30539;
    int  _30595;
    int p_30595;
    int  _30661;
    int p_30661;
    double  sum_30663;
    double psum_30663;
    int  _30600;
    int p_30600;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    bdimx_30484 = blockDim_x();
    pbdimx_30484 = bdimx_30484;
    l30482: ;
        bdimx_30484 = pbdimx_30484;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        bdimy_30490 = blockDim_y();
        pbdimy_30490 = bdimy_30490;
    l30488: ;
        bdimy_30490 = pbdimy_30490;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        bidx_30496 = blockIdx_x();
        pbidx_30496 = bidx_30496;
    l30494: ;
        bidx_30496 = pbidx_30496;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        bidy_30502 = blockIdx_y();
        pbidy_30502 = bidy_30502;
    l30500: ;
        bidy_30502 = pbidy_30502;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        tidx_30508 = threadIdx_x();
        ptidx_30508 = tidx_30508;
    l30506: ;
        tidx_30508 = ptidx_30508;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        tidy_30514 = threadIdx_y();
        ptidy_30514 = tidy_30514;
    l30512: ;
        tidy_30514 = ptidy_30514;
        #line 201 "gpu_device_shm.impala"
        __shared__ double reserver_reserve_shared_30522[938];
        preserve_shared_30522 = reserver_reserve_shared_30522;
    l30520: ;
        reserve_shared_30522 = preserve_shared_30522;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _30525 = blockDim_x();
        p_30525 = _30525;
    l30523: ;
        _30525 = p_30525;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _30528 = blockDim_y();
        p_30528 = _30528;
    l30526: ;
        _30528 = p_30528;
        #line 4 "gaussian.impala"
        int _30542;
        _30542 = _25607_30475.e1;
        #line 189 "gpu_device_shm.impala"
        int _30562;
        _30562 = bidy_30502 * bdimy_30490;
        #line 187 "gpu_device_shm.impala"
        int _30553;
        _30553 = bidx_30496 * bdimx_30484;
        #line 193 "gpu_device_shm.impala"
        int _30532;
        _30532 = _25607_30475.e2;
        #line 189 "gpu_device_shm.impala"
        int gid_y_30563;
        gid_y_30563 = _30562 + tidy_30514;
        #line 4 "gaussian.impala"
        int h_anchor_30543;
        h_anchor_30543 = _30542 / 2;
        #line 11 "main.impala"
        int _30568;
        _30568 = _25610_30478.e3;
        #line 193 "gpu_device_shm.impala"
        int extend_height_30533;
        extend_height_30533 = _30532 / 2;
        #line 187 "gpu_device_shm.impala"
        int gid_x_30554;
        gid_x_30554 = _30553 + tidx_30508;
        #line 195 "gpu_device_shm.impala"
        int _30544;
        _30544 = 2 * h_anchor_30543;
        #line 197 "gpu_device_shm.impala"
        int _30534;
        _30534 = 2 * extend_height_30533;
        #line 11 "main.impala"
        int _30559;
        _30559 = _25610_30478.e2;
        #line 195 "gpu_device_shm.impala"
        int shm_dimx_30545;
        shm_dimx_30545 = bdimx_30484 + _30544;
        #line 197 "gpu_device_shm.impala"
        int shm_dimy_30535;
        shm_dimy_30535 = bdimy_30490 + _30534;
        #line 38 "gpu_device_shm.impala"
        bool _30536;
        _30536 = 0 < shm_dimy_30535;
        #line 38 "gpu_device_shm.impala"
        if (_30536) goto l30537; else goto l30714;
    l30714: ;
        #line 231 "gpu_device_shm.impala"
        goto l30644;
    l30537: ;
        #line 219 "gpu_device_shm.impala"
        int _30555;
        _30555 = gid_x_30554 - h_anchor_30543;
        #line 226 "gpu_device_shm.impala"
        int _30580;
        _30580 = tidy_30514 * shm_dimx_30545;
        #line 221 "gpu_device_shm.impala"
        int _30564;
        _30564 = gid_y_30563 - extend_height_30533;
        #line 224 "gpu_device_shm.impala"
        bool _30569;
        _30569 = _30564 < _30568;
        #line 224 "gpu_device_shm.impala"
        bool _30565;
        _30565 = 0 <= _30564;
        #line 217 "gpu_device_shm.impala"
        bool _30551;
        _30551 = tidy_30514 < shm_dimy_30535;
        #line 227 "gpu_device_shm.impala"
        int _30574;
        _30574 = _30564 * _30559;
        #line 38 "gpu_device_shm.impala"
        p_30539 = 0;
        goto l30538;
    l30538: ;
        _30539 = p_30539;
        #line 38 "gpu_device_shm.impala"
        bool _30546;
        _30546 = _30539 < shm_dimx_30545;
        #line 38 "gpu_device_shm.impala"
        if (_30546) goto l30547; else goto l30593;
    l30593: ;
        #line 38 "gpu_device_shm.impala"
        p_30595 = bdimy_30490;
        goto l30594;
    l30594: ;
        _30595 = p_30595;
        #line 38 "gpu_device_shm.impala"
        bool _30597;
        _30597 = _30595 < shm_dimy_30535;
        #line 38 "gpu_device_shm.impala"
        if (_30597) goto l30598; else goto l30643;
    l30643: ;
        #line 231 "gpu_device_shm.impala"
        goto l30644;
    l30644: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        __syncthreads();
    l30649: ;
        #line 237 "gpu_device_shm.impala"
        bool _30651;
        _30651 = gid_x_30554 < _30559;
        #line 237 "gpu_device_shm.impala"
        if (_30651) goto l30652; else goto l30713;
    l30713: ;
        #line 240 "gpu_device_shm.impala"
        goto l30712;
    l30652: ;
        #line 237 "gpu_device_shm.impala"
        bool _30653;
        _30653 = gid_y_30563 < _30568;
        #line 237 "gpu_device_shm.impala"
        if (_30653) goto l30654; else goto l30711;
    l30711: ;
        #line 240 "gpu_device_shm.impala"
        goto l30712;
    l30712: ;
        return ;
    l30654: ;
        #line 204 "gpu_device_shm.impala"
        int _30676;
        _30676 = bidy_30502 * _30528;
        #line 64 "gpu_device_shm.impala"
        char* _30693;
        _30693 = _25609_30477.e1;
        #line 17 "gaussian.impala"
        bool _30655;
        _30655 = h_anchor_30543 <= gid_x_30554;
        #line 64 "gpu_device_shm.impala"
        double* _30694;
        union { double* dst; char* src; } u_30694;
        u_30694.src = _30693;
        _30694 = u_30694.dst;
        #line 204 "gpu_device_shm.impala"
        int _30677;
        _30677 = extend_height_30533 - _30676;
        #line 64 "gpu_device_shm.impala"
        int _30695;
        _30695 = gid_y_30563 * _30559;
        #line 203 "gpu_device_shm.impala"
        int _30682;
        _30682 = bidx_30496 * _30525;
        #line 203 "gpu_device_shm.impala"
        int _30683;
        _30683 = h_anchor_30543 - _30682;
        #line 72 "gpu_device_shm.impala"
        int _30678;
        _30678 = gid_y_30563 + _30677;
        #line 64 "gpu_device_shm.impala"
        int _30696;
        _30696 = _30695 + gid_x_30554;
        #line 72 "gpu_device_shm.impala"
        int _30679;
        _30679 = _30678 * shm_dimx_30545;
        #line 64 "gpu_device_shm.impala"
        double* _30697;
        _30697 = _30694 + _30696;
        #line 17 "gaussian.impala"
        if (_30655) goto l30656; else goto l30710;
    l30710: ;
        #line 27 "gaussian.impala"
        goto l30702;
    l30656: ;
        #line 17 "gaussian.impala"
        int _30657;
        _30657 = _30559 - h_anchor_30543;
        #line 17 "gaussian.impala"
        bool _30658;
        _30658 = gid_x_30554 < _30657;
        #line 17 "gaussian.impala"
        if (_30658) goto l30659; else goto l30701;
    l30701: ;
        #line 27 "gaussian.impala"
        goto l30702;
    l30702: ;
        #line 72 "gpu_device_shm.impala"
        int _30703;
        _30703 = _30679 + gid_x_30554;
        #line 72 "gpu_device_shm.impala"
        int _30704;
        _30704 = _30703 + _30683;
        #line 72 "gpu_device_shm.impala"
        double* _30705;
        _30705 = reserve_shared_30522 + _30704;
        #line 72 "gpu_device_shm.impala"
        double _30706;
        _30706 = *_30705;
        #line 72 "gpu_device_shm.impala"
        double _30708;
        _30708 = _30706;
        #line 64 "gpu_device_shm.impala"
        *_30697 = _30708;
        return ;
    l30659: ;
        #line 19 "gaussian.impala"
        int _30665;
        _30665 = 1 + h_anchor_30543;
        #line 77 "gpu_device_shm.impala"
        struct_Buffer_6341 _30669;
        _30669 = _25607_30475.e0;
        #line 19 "gaussian.impala"
        int _30699;
        _30699 = 0 - h_anchor_30543;
        #line 77 "gpu_device_shm.impala"
        char* _30670;
        _30670 = _30669.e1;
        #line 77 "gpu_device_shm.impala"
        double* _30671;
        union { double* dst; char* src; } u_30671;
        u_30671.src = _30670;
        _30671 = u_30671.dst;
        #line 27 "gpu_device_shm.impala"
        p_30661 = _30699;
        psum_30663 = 0.000000e+00;
        goto l30660;
    l30660: ;
        _30661 = p_30661;
        sum_30663 = psum_30663;
        #line 27 "gpu_device_shm.impala"
        bool _30666;
        _30666 = _30661 < _30665;
        #line 27 "gpu_device_shm.impala"
        if (_30666) goto l30667; else goto l30692;
    l30692: ;
        #line 64 "gpu_device_shm.impala"
        *_30697 = sum_30663;
        return ;
    l30667: ;
        #line 21 "gaussian.impala"
        int _30672;
        _30672 = _30661 + h_anchor_30543;
        #line 21 "gaussian.impala"
        int _30680;
        _30680 = gid_x_30554 + _30661;
        #line 31 "gpu_device_shm.impala"
        int _30668;
        _30668 = 1 + _30661;
        #line 76 "gpu_device_shm.impala"
        double* i_30673;
        i_30673 = _30671 + _30672;
        #line 72 "gpu_device_shm.impala"
        int _30681;
        _30681 = _30679 + _30680;
        #line 77 "gpu_device_shm.impala"
        double _30674;
        _30674 = *i_30673;
        #line 72 "gpu_device_shm.impala"
        int _30684;
        _30684 = _30681 + _30683;
        #line 77 "gpu_device_shm.impala"
        double _30688;
        _30688 = _30674;
        #line 72 "gpu_device_shm.impala"
        double* _30685;
        _30685 = reserve_shared_30522 + _30684;
        #line 72 "gpu_device_shm.impala"
        double _30686;
        _30686 = *_30685;
        #line 72 "gpu_device_shm.impala"
        double _30689;
        _30689 = _30686;
        #line 21 "gaussian.impala"
        double _30690;
        _30690 = _30688 * _30689;
        #line 21 "gaussian.impala"
        double _30691;
        _30691 = sum_30663 + _30690;
        #line 27 "gpu_device_shm.impala"
        p_30661 = _30668;
        psum_30663 = _30691;
        goto l30660;
    l30598: ;
        #line 221 "gpu_device_shm.impala"
        int img_index_y_30615;
        img_index_y_30615 = _30564 + _30595;
        #line 214 "gpu_device_shm.impala"
        int shm_index_y_30607;
        shm_index_y_30607 = tidy_30514 + _30595;
        #line 226 "gpu_device_shm.impala"
        int _30628;
        _30628 = shm_index_y_30607 * shm_dimx_30545;
        #line 224 "gpu_device_shm.impala"
        bool _30616;
        _30616 = 0 <= img_index_y_30615;
        #line 227 "gpu_device_shm.impala"
        int _30623;
        _30623 = img_index_y_30615 * _30559;
        #line 224 "gpu_device_shm.impala"
        bool _30618;
        _30618 = img_index_y_30615 < _30568;
        #line 217 "gpu_device_shm.impala"
        bool _30608;
        _30608 = shm_index_y_30607 < shm_dimy_30535;
        #line 38 "gpu_device_shm.impala"
        p_30600 = 0;
        goto l30599;
    l30599: ;
        _30600 = p_30600;
        #line 38 "gpu_device_shm.impala"
        bool _30602;
        _30602 = _30600 < shm_dimx_30545;
        #line 38 "gpu_device_shm.impala"
        if (_30602) goto l30603; else goto l30641;
    l30641: ;
        #line 42 "gpu_device_shm.impala"
        int _30642;
        _30642 = _30595 + bdimy_30490;
        #line 38 "gpu_device_shm.impala"
        p_30595 = _30642;
        goto l30594;
    l30603: ;
        #line 212 "gpu_device_shm.impala"
        int shm_index_x_30604;
        shm_index_x_30604 = tidx_30508 + _30600;
        #line 217 "gpu_device_shm.impala"
        bool _30605;
        _30605 = shm_index_x_30604 < shm_dimx_30545;
        #line 217 "gpu_device_shm.impala"
        if (_30605) goto l30606; else goto l30640;
    l30640: ;
        #line 229 "gpu_device_shm.impala"
        goto l30639;
    l30606: ;
        #line 217 "gpu_device_shm.impala"
        if (_30608) goto l30609; else goto l30638;
    l30638: ;
        #line 229 "gpu_device_shm.impala"
        goto l30639;
    l30639: ;
        #line 40 "gpu_device_shm.impala"
        goto l30620;
    l30609: ;
        #line 219 "gpu_device_shm.impala"
        int img_index_x_30610;
        img_index_x_30610 = _30555 + _30600;
        #line 224 "gpu_device_shm.impala"
        bool _30611;
        _30611 = 0 <= img_index_x_30610;
        #line 224 "gpu_device_shm.impala"
        if (_30611) goto l30612; else goto l30637;
    l30637: ;
        #line 228 "gpu_device_shm.impala"
        goto l30634;
    l30612: ;
        #line 224 "gpu_device_shm.impala"
        bool _30613;
        _30613 = img_index_x_30610 < _30559;
        #line 224 "gpu_device_shm.impala"
        if (_30613) goto l30614; else goto l30636;
    l30636: ;
        #line 228 "gpu_device_shm.impala"
        goto l30634;
    l30614: ;
        #line 224 "gpu_device_shm.impala"
        if (_30616) goto l30617; else goto l30635;
    l30635: ;
        #line 228 "gpu_device_shm.impala"
        goto l30634;
    l30617: ;
        #line 224 "gpu_device_shm.impala"
        if (_30618) goto l30619; else goto l30633;
    l30633: ;
        #line 228 "gpu_device_shm.impala"
        goto l30634;
    l30634: ;
        #line 40 "gpu_device_shm.impala"
        goto l30620;
    l30619: ;
        #line 226 "gpu_device_shm.impala"
        int _30629;
        _30629 = _30628 + shm_index_x_30604;
        #line 227 "gpu_device_shm.impala"
        int _30624;
        _30624 = _30623 + img_index_x_30610;
        #line 226 "gpu_device_shm.impala"
        double* _30630;
        _30630 = reserve_shared_30522 + _30629;
        #line 227 "gpu_device_shm.impala"
        double* _30625;
        _30625 = _25608_30476 + _30624;
        #line 227 "gpu_device_shm.impala"
        double _30626;
        _30626 = *_30625;
        #line 227 "gpu_device_shm.impala"
        double _30631;
        _30631 = _30626;
        #line 226 "gpu_device_shm.impala"
        *_30630 = _30631;
        #line 40 "gpu_device_shm.impala"
        goto l30620;
    l30620: ;
        #line 42 "gpu_device_shm.impala"
        int _30622;
        _30622 = _30600 + bdimx_30484;
        #line 38 "gpu_device_shm.impala"
        p_30600 = _30622;
        goto l30599;
    l30547: ;
        #line 212 "gpu_device_shm.impala"
        int shm_index_x_30548;
        shm_index_x_30548 = tidx_30508 + _30539;
        #line 217 "gpu_device_shm.impala"
        bool _30549;
        _30549 = shm_index_x_30548 < shm_dimx_30545;
        #line 217 "gpu_device_shm.impala"
        if (_30549) goto l30550; else goto l30592;
    l30592: ;
        #line 229 "gpu_device_shm.impala"
        goto l30591;
    l30550: ;
        #line 217 "gpu_device_shm.impala"
        if (_30551) goto l30552; else goto l30590;
    l30590: ;
        #line 229 "gpu_device_shm.impala"
        goto l30591;
    l30591: ;
        #line 40 "gpu_device_shm.impala"
        goto l30571;
    l30552: ;
        #line 219 "gpu_device_shm.impala"
        int img_index_x_30556;
        img_index_x_30556 = _30555 + _30539;
        #line 224 "gpu_device_shm.impala"
        bool _30557;
        _30557 = 0 <= img_index_x_30556;
        #line 224 "gpu_device_shm.impala"
        if (_30557) goto l30558; else goto l30589;
    l30589: ;
        #line 228 "gpu_device_shm.impala"
        goto l30586;
    l30558: ;
        #line 224 "gpu_device_shm.impala"
        bool _30560;
        _30560 = img_index_x_30556 < _30559;
        #line 224 "gpu_device_shm.impala"
        if (_30560) goto l30561; else goto l30588;
    l30588: ;
        #line 228 "gpu_device_shm.impala"
        goto l30586;
    l30561: ;
        #line 224 "gpu_device_shm.impala"
        if (_30565) goto l30566; else goto l30587;
    l30587: ;
        #line 228 "gpu_device_shm.impala"
        goto l30586;
    l30566: ;
        #line 224 "gpu_device_shm.impala"
        if (_30569) goto l30570; else goto l30585;
    l30585: ;
        #line 228 "gpu_device_shm.impala"
        goto l30586;
    l30586: ;
        #line 40 "gpu_device_shm.impala"
        goto l30571;
    l30570: ;
        #line 227 "gpu_device_shm.impala"
        int _30575;
        _30575 = _30574 + img_index_x_30556;
        #line 227 "gpu_device_shm.impala"
        double* _30576;
        _30576 = _25608_30476 + _30575;
        #line 227 "gpu_device_shm.impala"
        double _30577;
        _30577 = *_30576;
        #line 226 "gpu_device_shm.impala"
        int _30581;
        _30581 = _30580 + shm_index_x_30548;
        #line 227 "gpu_device_shm.impala"
        double _30583;
        _30583 = _30577;
        #line 226 "gpu_device_shm.impala"
        double* _30582;
        _30582 = reserve_shared_30522 + _30581;
        #line 226 "gpu_device_shm.impala"
        *_30582 = _30583;
        #line 40 "gpu_device_shm.impala"
        goto l30571;
    l30571: ;
        #line 42 "gpu_device_shm.impala"
        int _30573;
        _30573 = _30539 + bdimx_30484;
        #line 38 "gpu_device_shm.impala"
        p_30539 = _30573;
        goto l30538;
}

__global__ __launch_bounds__ (128 * 1 * 1) void lambda_25866(double* _25869_30718, double* _25870_30719, struct_filter_6340 _25871_30720, struct_Buffer_6341 _25872_30721, struct_image_6346 _25873_30722) {
    int  bdimx_30725;
    int pbdimx_30725;
    int  bdimy_30728;
    int pbdimy_30728;
    int  bidx_30731;
    int pbidx_30731;
    int  bidy_30734;
    int pbidy_30734;
    int  tidx_30737;
    int ptidx_30737;
    int  tidy_30740;
    int ptidy_30740;
    double*  reserve_shared_30743;
    double* preserve_shared_30743;
    int  _30746;
    int p_30746;
    int  _30749;
    int p_30749;
    int  _30757;
    int p_30757;
    int  _30810;
    int p_30810;
    int  _30873;
    int p_30873;
    double  sum_30875;
    double psum_30875;
    int  _30815;
    int p_30815;
    #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
    bdimx_30725 = blockDim_x();
    pbdimx_30725 = bdimx_30725;
    l30723: ;
        bdimx_30725 = pbdimx_30725;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        bdimy_30728 = blockDim_y();
        pbdimy_30728 = bdimy_30728;
    l30726: ;
        bdimy_30728 = pbdimy_30728;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        bidx_30731 = blockIdx_x();
        pbidx_30731 = bidx_30731;
    l30729: ;
        bidx_30731 = pbidx_30731;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        bidy_30734 = blockIdx_y();
        pbidy_30734 = bidy_30734;
    l30732: ;
        bidy_30734 = pbidy_30734;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        tidx_30737 = threadIdx_x();
        ptidx_30737 = tidx_30737;
    l30735: ;
        tidx_30737 = ptidx_30737;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        tidy_30740 = threadIdx_y();
        ptidy_30740 = tidy_30740;
    l30738: ;
        tidy_30740 = ptidy_30740;
        #line 201 "gpu_device_shm.impala"
        __shared__ double reserver_reserve_shared_30743[938];
        preserve_shared_30743 = reserver_reserve_shared_30743;
    l30741: ;
        reserve_shared_30743 = preserve_shared_30743;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _30746 = blockDim_x();
        p_30746 = _30746;
    l30744: ;
        _30746 = p_30746;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _30749 = blockDim_y();
        p_30749 = _30749;
    l30747: ;
        _30749 = p_30749;
        #line 189 "gpu_device_shm.impala"
        int _30779;
        _30779 = bidy_30734 * bdimy_30728;
        #line 11 "main.impala"
        int _30784;
        _30784 = _25873_30722.e3;
        #line 191 "gpu_device_shm.impala"
        int _30759;
        _30759 = _25871_30720.e1;
        #line 187 "gpu_device_shm.impala"
        int _30770;
        _30770 = bidx_30731 * bdimx_30725;
        #line 6 "gaussian.impala"
        int _30750;
        _30750 = _25871_30720.e2;
        #line 6 "gaussian.impala"
        int v_anchor_30751;
        v_anchor_30751 = _30750 / 2;
        #line 187 "gpu_device_shm.impala"
        int gid_x_30771;
        gid_x_30771 = _30770 + tidx_30737;
        #line 189 "gpu_device_shm.impala"
        int gid_y_30780;
        gid_y_30780 = _30779 + tidy_30740;
        #line 191 "gpu_device_shm.impala"
        int extend_width_30760;
        extend_width_30760 = _30759 / 2;
        #line 197 "gpu_device_shm.impala"
        int _30752;
        _30752 = 2 * v_anchor_30751;
        #line 197 "gpu_device_shm.impala"
        int shm_dimy_30753;
        shm_dimy_30753 = bdimy_30728 + _30752;
        #line 11 "main.impala"
        int _30776;
        _30776 = _25873_30722.e2;
        #line 195 "gpu_device_shm.impala"
        int _30761;
        _30761 = 2 * extend_width_30760;
        #line 38 "gpu_device_shm.impala"
        bool _30754;
        _30754 = 0 < shm_dimy_30753;
        #line 195 "gpu_device_shm.impala"
        int shm_dimx_30762;
        shm_dimx_30762 = bdimx_30725 + _30761;
        #line 38 "gpu_device_shm.impala"
        if (_30754) goto l30755; else goto l30926;
    l30926: ;
        #line 231 "gpu_device_shm.impala"
        goto l30859;
    l30755: ;
        #line 219 "gpu_device_shm.impala"
        int _30772;
        _30772 = gid_x_30771 - extend_width_30760;
        #line 217 "gpu_device_shm.impala"
        bool _30768;
        _30768 = tidy_30740 < shm_dimy_30753;
        #line 226 "gpu_device_shm.impala"
        int _30795;
        _30795 = tidy_30740 * shm_dimx_30762;
        #line 221 "gpu_device_shm.impala"
        int _30781;
        _30781 = gid_y_30780 - v_anchor_30751;
        #line 224 "gpu_device_shm.impala"
        bool _30782;
        _30782 = 0 <= _30781;
        #line 224 "gpu_device_shm.impala"
        bool _30785;
        _30785 = _30781 < _30784;
        #line 227 "gpu_device_shm.impala"
        int _30790;
        _30790 = _30781 * _30776;
        #line 38 "gpu_device_shm.impala"
        p_30757 = 0;
        goto l30756;
    l30756: ;
        _30757 = p_30757;
        #line 38 "gpu_device_shm.impala"
        bool _30763;
        _30763 = _30757 < shm_dimx_30762;
        #line 38 "gpu_device_shm.impala"
        if (_30763) goto l30764; else goto l30808;
    l30808: ;
        #line 38 "gpu_device_shm.impala"
        p_30810 = bdimy_30728;
        goto l30809;
    l30809: ;
        _30810 = p_30810;
        #line 38 "gpu_device_shm.impala"
        bool _30812;
        _30812 = _30810 < shm_dimy_30753;
        #line 38 "gpu_device_shm.impala"
        if (_30812) goto l30813; else goto l30858;
    l30858: ;
        #line 231 "gpu_device_shm.impala"
        goto l30859;
    l30859: ;
        #line 1 "/home/rafael/repositories/anydsl/runtime/platforms/intrinsics_cuda.impala"
        __syncthreads();
    l30861: ;
        #line 237 "gpu_device_shm.impala"
        bool _30863;
        _30863 = gid_x_30771 < _30776;
        #line 237 "gpu_device_shm.impala"
        if (_30863) goto l30864; else goto l30925;
    l30925: ;
        #line 240 "gpu_device_shm.impala"
        goto l30924;
    l30864: ;
        #line 237 "gpu_device_shm.impala"
        bool _30865;
        _30865 = gid_y_30780 < _30784;
        #line 237 "gpu_device_shm.impala"
        if (_30865) goto l30866; else goto l30923;
    l30923: ;
        #line 240 "gpu_device_shm.impala"
        goto l30924;
    l30924: ;
        return ;
    l30866: ;
        #line 203 "gpu_device_shm.impala"
        int _30893;
        _30893 = bidx_30731 * _30746;
        #line 39 "gaussian.impala"
        bool _30867;
        _30867 = v_anchor_30751 <= gid_y_30780;
        #line 64 "gpu_device_shm.impala"
        char* _30904;
        _30904 = _25872_30721.e1;
        #line 203 "gpu_device_shm.impala"
        int _30894;
        _30894 = extend_width_30760 - _30893;
        #line 64 "gpu_device_shm.impala"
        int _30906;
        _30906 = gid_y_30780 * _30776;
        #line 204 "gpu_device_shm.impala"
        int _30888;
        _30888 = bidy_30734 * _30749;
        #line 64 "gpu_device_shm.impala"
        double* _30905;
        union { double* dst; char* src; } u_30905;
        u_30905.src = _30904;
        _30905 = u_30905.dst;
        #line 64 "gpu_device_shm.impala"
        int _30907;
        _30907 = _30906 + gid_x_30771;
        #line 204 "gpu_device_shm.impala"
        int _30889;
        _30889 = v_anchor_30751 - _30888;
        #line 64 "gpu_device_shm.impala"
        double* _30908;
        _30908 = _30905 + _30907;
        #line 39 "gaussian.impala"
        if (_30867) goto l30868; else goto l30922;
    l30922: ;
        #line 49 "gaussian.impala"
        goto l30912;
    l30868: ;
        #line 39 "gaussian.impala"
        int _30869;
        _30869 = _30784 - v_anchor_30751;
        #line 39 "gaussian.impala"
        bool _30870;
        _30870 = gid_y_30780 < _30869;
        #line 39 "gaussian.impala"
        if (_30870) goto l30871; else goto l30911;
    l30911: ;
        #line 49 "gaussian.impala"
        goto l30912;
    l30912: ;
        #line 72 "gpu_device_shm.impala"
        int _30913;
        _30913 = gid_y_30780 + _30889;
        #line 72 "gpu_device_shm.impala"
        int _30914;
        _30914 = _30913 * shm_dimx_30762;
        #line 72 "gpu_device_shm.impala"
        int _30915;
        _30915 = _30914 + gid_x_30771;
        #line 72 "gpu_device_shm.impala"
        int _30916;
        _30916 = _30915 + _30894;
        #line 72 "gpu_device_shm.impala"
        double* _30917;
        _30917 = reserve_shared_30743 + _30916;
        #line 72 "gpu_device_shm.impala"
        double _30918;
        _30918 = *_30917;
        #line 72 "gpu_device_shm.impala"
        double _30920;
        _30920 = _30918;
        #line 64 "gpu_device_shm.impala"
        *_30908 = _30920;
        return ;
    l30871: ;
        #line 77 "gpu_device_shm.impala"
        struct_Buffer_6341 _30880;
        _30880 = _25871_30720.e0;
        #line 41 "gaussian.impala"
        int _30910;
        _30910 = 0 - v_anchor_30751;
        #line 41 "gaussian.impala"
        int _30876;
        _30876 = 1 + v_anchor_30751;
        #line 77 "gpu_device_shm.impala"
        char* _30881;
        _30881 = _30880.e1;
        #line 77 "gpu_device_shm.impala"
        double* _30882;
        union { double* dst; char* src; } u_30882;
        u_30882.src = _30881;
        _30882 = u_30882.dst;
        #line 27 "gpu_device_shm.impala"
        p_30873 = _30910;
        psum_30875 = 0.000000e+00;
        goto l30872;
    l30872: ;
        _30873 = p_30873;
        sum_30875 = psum_30875;
        #line 27 "gpu_device_shm.impala"
        bool _30877;
        _30877 = _30873 < _30876;
        #line 27 "gpu_device_shm.impala"
        if (_30877) goto l30878; else goto l30903;
    l30903: ;
        #line 64 "gpu_device_shm.impala"
        *_30908 = sum_30875;
        return ;
    l30878: ;
        #line 31 "gpu_device_shm.impala"
        int _30879;
        _30879 = 1 + _30873;
        #line 43 "gaussian.impala"
        int _30887;
        _30887 = gid_y_30780 + _30873;
        #line 72 "gpu_device_shm.impala"
        int _30890;
        _30890 = _30887 + _30889;
        #line 43 "gaussian.impala"
        int _30883;
        _30883 = _30873 + v_anchor_30751;
        #line 72 "gpu_device_shm.impala"
        int _30891;
        _30891 = _30890 * shm_dimx_30762;
        #line 76 "gpu_device_shm.impala"
        double* i_30884;
        i_30884 = _30882 + _30883;
        #line 72 "gpu_device_shm.impala"
        int _30892;
        _30892 = _30891 + gid_x_30771;
        #line 77 "gpu_device_shm.impala"
        double _30885;
        _30885 = *i_30884;
        #line 72 "gpu_device_shm.impala"
        int _30895;
        _30895 = _30892 + _30894;
        #line 77 "gpu_device_shm.impala"
        double _30899;
        _30899 = _30885;
        #line 72 "gpu_device_shm.impala"
        double* _30896;
        _30896 = reserve_shared_30743 + _30895;
        #line 72 "gpu_device_shm.impala"
        double _30897;
        _30897 = *_30896;
        #line 72 "gpu_device_shm.impala"
        double _30900;
        _30900 = _30897;
        #line 43 "gaussian.impala"
        double _30901;
        _30901 = _30899 * _30900;
        #line 43 "gaussian.impala"
        double _30902;
        _30902 = sum_30875 + _30901;
        #line 27 "gpu_device_shm.impala"
        p_30873 = _30879;
        psum_30875 = _30902;
        goto l30872;
    l30813: ;
        #line 221 "gpu_device_shm.impala"
        int img_index_y_30830;
        img_index_y_30830 = _30781 + _30810;
        #line 227 "gpu_device_shm.impala"
        int _30838;
        _30838 = img_index_y_30830 * _30776;
        #line 214 "gpu_device_shm.impala"
        int shm_index_y_30822;
        shm_index_y_30822 = tidy_30740 + _30810;
        #line 224 "gpu_device_shm.impala"
        bool _30833;
        _30833 = img_index_y_30830 < _30784;
        #line 224 "gpu_device_shm.impala"
        bool _30831;
        _30831 = 0 <= img_index_y_30830;
        #line 217 "gpu_device_shm.impala"
        bool _30823;
        _30823 = shm_index_y_30822 < shm_dimy_30753;
        #line 226 "gpu_device_shm.impala"
        int _30843;
        _30843 = shm_index_y_30822 * shm_dimx_30762;
        #line 38 "gpu_device_shm.impala"
        p_30815 = 0;
        goto l30814;
    l30814: ;
        _30815 = p_30815;
        #line 38 "gpu_device_shm.impala"
        bool _30817;
        _30817 = _30815 < shm_dimx_30762;
        #line 38 "gpu_device_shm.impala"
        if (_30817) goto l30818; else goto l30856;
    l30856: ;
        #line 42 "gpu_device_shm.impala"
        int _30857;
        _30857 = _30810 + bdimy_30728;
        #line 38 "gpu_device_shm.impala"
        p_30810 = _30857;
        goto l30809;
    l30818: ;
        #line 212 "gpu_device_shm.impala"
        int shm_index_x_30819;
        shm_index_x_30819 = tidx_30737 + _30815;
        #line 217 "gpu_device_shm.impala"
        bool _30820;
        _30820 = shm_index_x_30819 < shm_dimx_30762;
        #line 217 "gpu_device_shm.impala"
        if (_30820) goto l30821; else goto l30855;
    l30855: ;
        #line 229 "gpu_device_shm.impala"
        goto l30854;
    l30821: ;
        #line 217 "gpu_device_shm.impala"
        if (_30823) goto l30824; else goto l30853;
    l30853: ;
        #line 229 "gpu_device_shm.impala"
        goto l30854;
    l30854: ;
        #line 40 "gpu_device_shm.impala"
        goto l30835;
    l30824: ;
        #line 219 "gpu_device_shm.impala"
        int img_index_x_30825;
        img_index_x_30825 = _30772 + _30815;
        #line 224 "gpu_device_shm.impala"
        bool _30826;
        _30826 = 0 <= img_index_x_30825;
        #line 224 "gpu_device_shm.impala"
        if (_30826) goto l30827; else goto l30852;
    l30852: ;
        #line 228 "gpu_device_shm.impala"
        goto l30849;
    l30827: ;
        #line 224 "gpu_device_shm.impala"
        bool _30828;
        _30828 = img_index_x_30825 < _30776;
        #line 224 "gpu_device_shm.impala"
        if (_30828) goto l30829; else goto l30851;
    l30851: ;
        #line 228 "gpu_device_shm.impala"
        goto l30849;
    l30829: ;
        #line 224 "gpu_device_shm.impala"
        if (_30831) goto l30832; else goto l30850;
    l30850: ;
        #line 228 "gpu_device_shm.impala"
        goto l30849;
    l30832: ;
        #line 224 "gpu_device_shm.impala"
        if (_30833) goto l30834; else goto l30848;
    l30848: ;
        #line 228 "gpu_device_shm.impala"
        goto l30849;
    l30849: ;
        #line 40 "gpu_device_shm.impala"
        goto l30835;
    l30834: ;
        #line 227 "gpu_device_shm.impala"
        int _30839;
        _30839 = _30838 + img_index_x_30825;
        #line 226 "gpu_device_shm.impala"
        int _30844;
        _30844 = _30843 + shm_index_x_30819;
        #line 226 "gpu_device_shm.impala"
        double* _30845;
        _30845 = reserve_shared_30743 + _30844;
        #line 227 "gpu_device_shm.impala"
        double* _30840;
        _30840 = _25870_30719 + _30839;
        #line 227 "gpu_device_shm.impala"
        double _30841;
        _30841 = *_30840;
        #line 227 "gpu_device_shm.impala"
        double _30846;
        _30846 = _30841;
        #line 226 "gpu_device_shm.impala"
        *_30845 = _30846;
        #line 40 "gpu_device_shm.impala"
        goto l30835;
    l30835: ;
        #line 42 "gpu_device_shm.impala"
        int _30837;
        _30837 = _30815 + bdimx_30725;
        #line 38 "gpu_device_shm.impala"
        p_30815 = _30837;
        goto l30814;
    l30764: ;
        #line 212 "gpu_device_shm.impala"
        int shm_index_x_30765;
        shm_index_x_30765 = tidx_30737 + _30757;
        #line 217 "gpu_device_shm.impala"
        bool _30766;
        _30766 = shm_index_x_30765 < shm_dimx_30762;
        #line 217 "gpu_device_shm.impala"
        if (_30766) goto l30767; else goto l30807;
    l30807: ;
        #line 229 "gpu_device_shm.impala"
        goto l30806;
    l30767: ;
        #line 217 "gpu_device_shm.impala"
        if (_30768) goto l30769; else goto l30805;
    l30805: ;
        #line 229 "gpu_device_shm.impala"
        goto l30806;
    l30806: ;
        #line 40 "gpu_device_shm.impala"
        goto l30787;
    l30769: ;
        #line 219 "gpu_device_shm.impala"
        int img_index_x_30773;
        img_index_x_30773 = _30772 + _30757;
        #line 224 "gpu_device_shm.impala"
        bool _30774;
        _30774 = 0 <= img_index_x_30773;
        #line 224 "gpu_device_shm.impala"
        if (_30774) goto l30775; else goto l30804;
    l30804: ;
        #line 228 "gpu_device_shm.impala"
        goto l30801;
    l30775: ;
        #line 224 "gpu_device_shm.impala"
        bool _30777;
        _30777 = img_index_x_30773 < _30776;
        #line 224 "gpu_device_shm.impala"
        if (_30777) goto l30778; else goto l30803;
    l30803: ;
        #line 228 "gpu_device_shm.impala"
        goto l30801;
    l30778: ;
        #line 224 "gpu_device_shm.impala"
        if (_30782) goto l30783; else goto l30802;
    l30802: ;
        #line 228 "gpu_device_shm.impala"
        goto l30801;
    l30783: ;
        #line 224 "gpu_device_shm.impala"
        if (_30785) goto l30786; else goto l30800;
    l30800: ;
        #line 228 "gpu_device_shm.impala"
        goto l30801;
    l30801: ;
        #line 40 "gpu_device_shm.impala"
        goto l30787;
    l30786: ;
        #line 226 "gpu_device_shm.impala"
        int _30796;
        _30796 = _30795 + shm_index_x_30765;
        #line 227 "gpu_device_shm.impala"
        int _30791;
        _30791 = _30790 + img_index_x_30773;
        #line 227 "gpu_device_shm.impala"
        double* _30792;
        _30792 = _25870_30719 + _30791;
        #line 226 "gpu_device_shm.impala"
        double* _30797;
        _30797 = reserve_shared_30743 + _30796;
        #line 227 "gpu_device_shm.impala"
        double _30793;
        _30793 = *_30792;
        #line 227 "gpu_device_shm.impala"
        double _30798;
        _30798 = _30793;
        #line 226 "gpu_device_shm.impala"
        *_30797 = _30798;
        #line 40 "gpu_device_shm.impala"
        goto l30787;
    l30787: ;
        #line 42 "gpu_device_shm.impala"
        int _30789;
        _30789 = _30757 + bdimx_30725;
        #line 38 "gpu_device_shm.impala"
        p_30757 = _30789;
        goto l30756;
}

}